/*
 * Copyright 2011 University of Sheffield.
 * Author: Dr Paul Richmond 
 * Contact: p.richmond@sheffield.ac.uk (http://www.paulrichmond.staff.shef.ac.uk)
 *
 * University of Sheffield retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * University of Sheffield is strictly prohibited.
 *
 * For terms of licence agreement please attached licence or view licence 
 * on www.flamegpu.com website.
 * 
 */
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <cmath>
#include <GL/glew.h>
#include <GL/glut.h>
#include <cuda_gl_interop.h>

#include "header.h"

extern void stepFLAMESimulation()
{
	singleIteration();
}


extern void setCudaDevice()
{
	//set the CUDA GL device: Will cause an error without this since CUDA 3.0
    cudaGLSetGLDevice(0);
}