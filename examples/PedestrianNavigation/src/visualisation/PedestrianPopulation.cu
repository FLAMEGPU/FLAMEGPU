#include "hip/hip_runtime.h"
/*
 * Copyright 2011 University of Sheffield.
 * Author: Dr Paul Richmond 
 * Contact: p.richmond@sheffield.ac.uk (http://www.paulrichmond.staff.shef.ac.uk)
 *
 * University of Sheffield retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * University of Sheffield is strictly prohibited.
 *
 * For terms of licence agreement please attached licence or view licence 
 * on www.flamegpu.com website.
 * 
 */
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <cmath>
#include <GL/glew.h>
#include <GL/glut.h>
#include <cuda_gl_interop.h>
#include <glm/glm.hpp>
#include "header.h"

/* Error check function for safe CUDA API calling */
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/* Error check function for post CUDA Kernel calling */
#define gpuErrchkLaunch() { gpuLaunchAssert(__FILE__, __LINE__); }
inline void gpuLaunchAssert(const char *file, int line, bool abort=true)
{
	gpuAssert( hipPeekAtLastError(), file, line );
#ifdef _DEBUG
	gpuAssert( hipDeviceSynchronize(), file, line );
#endif
}
//KERNEL DEFINITIONS
/** output_navmaps_to_TBO
 * Outputs navmap agent data from FLAME GPU to a 4 component vector used for instancing
 * @param	agents	pedestrian agent list from FLAME GPU
 * @param	data1 four component vector used to output instance data 
 * @param	data2 four component vector used to output instance data 
 */
__global__ void output_pedestrians_to_TBO(xmachine_memory_agent_list* agents, glm::vec4* data1, glm::vec4* data2){

	//global thread index
	int index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;

	data1[index].x = agents->x[index];
	data1[index].y = agents->y[index];
	data1[index].z = agents->animate[index];
	data1[index].w = agents->height[index];

	data2[index].x = agents->velx[index];
	data2[index].y = agents->vely[index];
	data2[index].z = (float)agents->exit_no[index];
	data2[index].w = 0.0;
}


void generate_pedestrian_instances(GLuint* instances_data1_tbo, GLuint* instances_data2_tbo)
{
	//kernals sizes
	int threads_per_tile = 128;
	int tile_size;
	dim3 grid;
    dim3 threads;

	//pointer
	glm::vec4 *dptr_1;
	glm::vec4 *dptr_2;
	
	if (get_agent_agent_default_count() > 0)
	{
		// map OpenGL buffer object for writing from CUDA
		gpuErrchk(cudaGLMapBufferObject( (void**)&dptr_1, *instances_data1_tbo));
		gpuErrchk(cudaGLMapBufferObject( (void**)&dptr_2, *instances_data2_tbo));
		//cuda block size
		tile_size = (int) ceil((float)get_agent_agent_default_count()/threads_per_tile);
		grid = dim3(tile_size, 1, 1);
		threads = dim3(threads_per_tile, 1, 1);
		//kernel
		output_pedestrians_to_TBO<<< grid, threads>>>(get_device_agent_default_agents(), dptr_1, dptr_2);
		gpuErrchkLaunch();
		// unmap buffer object
		gpuErrchk(cudaGLUnmapBufferObject(*instances_data1_tbo));
		gpuErrchk(cudaGLUnmapBufferObject(*instances_data2_tbo));
	}
}


int getPedestrianCount()
{
	return get_agent_agent_default_count();
}