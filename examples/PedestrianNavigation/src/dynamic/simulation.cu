#include "hip/hip_runtime.h"

/*
 * FLAME GPU v 1.4.0 for CUDA 6
 * Copyright 2015 University of Sheffield.
 * Author: Dr Paul Richmond 
 * Contact: p.richmond@sheffield.ac.uk (http://www.paulrichmond.staff.shef.ac.uk)
 *
 * University of Sheffield retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * University of Sheffield is strictly prohibited.
 *
 * For terms of licence agreement please attached licence or view licence 
 * on www.flamegpu.com website.
 * 
 */

//Disable internal thrust warnings about conversions
#pragma warning(push)
#pragma warning (disable : 4267)
#pragma warning (disable : 4244)

// includes
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_vector_types.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/system/cuda/execution_policy.h>
#include <vector_operators.h>

// include FLAME kernels
#include "FLAMEGPU_kernals.cu"

#pragma warning(pop)

/* Error check function for safe CUDA API calling */
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/* Error check function for post CUDA Kernel calling */
#define gpuErrchkLaunch() { gpuLaunchAssert(__FILE__, __LINE__); }
inline void gpuLaunchAssert(const char *file, int line, bool abort=true)
{
	gpuAssert( hipPeekAtLastError(), file, line );
#ifdef _DEBUG
	gpuAssert( hipDeviceSynchronize(), file, line );
#endif
   
}

/* SM padding and offset variables */
int SM_START;
int PADDING;

/* Agent Memory */

/* agent Agent variables these lists are used in the agent function where as the other lists are used only outside the agent functions*/
xmachine_memory_agent_list* d_agents;      /**< Pointer to agent list (population) on the device*/
xmachine_memory_agent_list* d_agents_swap; /**< Pointer to agent list swap on the device (used when killing agents)*/
xmachine_memory_agent_list* d_agents_new;  /**< Pointer to new agent list on the device (used to hold new agents bfore they are appended to the population)*/
int h_xmachine_memory_agent_count;   /**< Agent population size counter */ 
uint * d_xmachine_memory_agent_keys;	  /**< Agent sort identifiers keys*/
uint * d_xmachine_memory_agent_values;  /**< Agent sort identifiers value */
    
/* agent state variables */
xmachine_memory_agent_list* h_agents_default;      /**< Pointer to agent list (population) on host*/
xmachine_memory_agent_list* d_agents_default;      /**< Pointer to agent list (population) on the device*/
int h_xmachine_memory_agent_default_count;   /**< Agent population size counter */ 

/* navmap Agent variables these lists are used in the agent function where as the other lists are used only outside the agent functions*/
xmachine_memory_navmap_list* d_navmaps;      /**< Pointer to agent list (population) on the device*/
xmachine_memory_navmap_list* d_navmaps_swap; /**< Pointer to agent list swap on the device (used when killing agents)*/
xmachine_memory_navmap_list* d_navmaps_new;  /**< Pointer to new agent list on the device (used to hold new agents bfore they are appended to the population)*/
int h_xmachine_memory_navmap_count;   /**< Agent population size counter */ 
int h_xmachine_memory_navmap_pop_width;   /**< Agent population width */
uint * d_xmachine_memory_navmap_keys;	  /**< Agent sort identifiers keys*/
uint * d_xmachine_memory_navmap_values;  /**< Agent sort identifiers value */
    
/* navmap state variables */
xmachine_memory_navmap_list* h_navmaps_static;      /**< Pointer to agent list (population) on host*/
xmachine_memory_navmap_list* d_navmaps_static;      /**< Pointer to agent list (population) on the device*/
int h_xmachine_memory_navmap_static_count;   /**< Agent population size counter */ 


/* Message Memory */

/* pedestrian_location Message variables */
xmachine_message_pedestrian_location_list* h_pedestrian_locations;         /**< Pointer to message list on host*/
xmachine_message_pedestrian_location_list* d_pedestrian_locations;         /**< Pointer to message list on device*/
xmachine_message_pedestrian_location_list* d_pedestrian_locations_swap;    /**< Pointer to message swap list on device (used for holding optional messages)*/
/* Non partitioned and spatial partitioned message variables  */
int h_message_pedestrian_location_count;         /**< message list counter*/
int h_message_pedestrian_location_output_type;   /**< message output type (single or optional)*/
/* Spatial Partitioning Variables*/
#ifdef FAST_ATOMIC_SORTING
	uint * d_xmachine_message_pedestrian_location_local_bin_index;	  /**< index offset within the assigned bin */
	uint * d_xmachine_message_pedestrian_location_unsorted_index;		/**< unsorted index (hash) value for message */
#else
	uint * d_xmachine_message_pedestrian_location_keys;	  /**< message sort identifier keys*/
	uint * d_xmachine_message_pedestrian_location_values;  /**< message sort identifier values */
#endif
xmachine_message_pedestrian_location_PBM * d_pedestrian_location_partition_matrix;  /**< Pointer to PCB matrix */
float3 h_message_pedestrian_location_min_bounds;           /**< min bounds (x,y,z) of partitioning environment */
float3 h_message_pedestrian_location_max_bounds;           /**< max bounds (x,y,z) of partitioning environment */
int3 h_message_pedestrian_location_partitionDim;           /**< partition dimensions (x,y,z) of partitioning environment */
float h_message_pedestrian_location_radius;                 /**< partition radius (used to determin the size of the partitions) */
/* Texture offset values for host */
int h_tex_xmachine_message_pedestrian_location_x_offset;
int h_tex_xmachine_message_pedestrian_location_y_offset;
int h_tex_xmachine_message_pedestrian_location_z_offset;
int h_tex_xmachine_message_pedestrian_location_pbm_start_offset;
int h_tex_xmachine_message_pedestrian_location_pbm_end_or_count_offset;

/* navmap_cell Message variables */
xmachine_message_navmap_cell_list* h_navmap_cells;         /**< Pointer to message list on host*/
xmachine_message_navmap_cell_list* d_navmap_cells;         /**< Pointer to message list on device*/
xmachine_message_navmap_cell_list* d_navmap_cells_swap;    /**< Pointer to message swap list on device (used for holding optional messages)*/
/* Discrete Partitioning Variables*/
int h_message_navmap_cell_range;     /**< range of the discrete message*/
int h_message_navmap_cell_width;     /**< with of the message grid*/
/* Texture offset values for host */
int h_tex_xmachine_message_navmap_cell_x_offset;
int h_tex_xmachine_message_navmap_cell_y_offset;
int h_tex_xmachine_message_navmap_cell_exit_no_offset;
int h_tex_xmachine_message_navmap_cell_height_offset;
int h_tex_xmachine_message_navmap_cell_collision_x_offset;
int h_tex_xmachine_message_navmap_cell_collision_y_offset;
int h_tex_xmachine_message_navmap_cell_exit0_x_offset;
int h_tex_xmachine_message_navmap_cell_exit0_y_offset;
int h_tex_xmachine_message_navmap_cell_exit1_x_offset;
int h_tex_xmachine_message_navmap_cell_exit1_y_offset;
int h_tex_xmachine_message_navmap_cell_exit2_x_offset;
int h_tex_xmachine_message_navmap_cell_exit2_y_offset;
int h_tex_xmachine_message_navmap_cell_exit3_x_offset;
int h_tex_xmachine_message_navmap_cell_exit3_y_offset;
int h_tex_xmachine_message_navmap_cell_exit4_x_offset;
int h_tex_xmachine_message_navmap_cell_exit4_y_offset;
int h_tex_xmachine_message_navmap_cell_exit5_x_offset;
int h_tex_xmachine_message_navmap_cell_exit5_y_offset;
int h_tex_xmachine_message_navmap_cell_exit6_x_offset;
int h_tex_xmachine_message_navmap_cell_exit6_y_offset;
  
/* CUDA Streams for function layers */
hipStream_t stream1;
hipStream_t stream2;

/*Global condition counts*/

/* RNG rand48 */
RNG_rand48* h_rand48;    /**< Pointer to RNG_rand48 seed list on host*/
RNG_rand48* d_rand48;    /**< Pointer to RNG_rand48 seed list on device*/

/* CUDA Parallel Primatives variables */
int scan_last_sum;           /**< Indicates if the position (in message list) of last message*/
int scan_last_included;      /**< Indicates if last sum value is included in the total sum count*/

/* Agent function prototypes */

/** agent_output_pedestrian_location
 * Agent function prototype for output_pedestrian_location function of agent agent
 */
void agent_output_pedestrian_location(hipStream_t &stream);

/** agent_avoid_pedestrians
 * Agent function prototype for avoid_pedestrians function of agent agent
 */
void agent_avoid_pedestrians(hipStream_t &stream);

/** agent_force_flow
 * Agent function prototype for force_flow function of agent agent
 */
void agent_force_flow(hipStream_t &stream);

/** agent_move
 * Agent function prototype for move function of agent agent
 */
void agent_move(hipStream_t &stream);

/** navmap_output_navmap_cells
 * Agent function prototype for output_navmap_cells function of navmap agent
 */
void navmap_output_navmap_cells(hipStream_t &stream);

/** navmap_generate_pedestrians
 * Agent function prototype for generate_pedestrians function of navmap agent
 */
void navmap_generate_pedestrians(hipStream_t &stream);

  
void setPaddingAndOffset()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	int x64_sys = 0;

	// This function call returns 9999 for both major & minor fields, if no CUDA capable devices are present
	if (deviceProp.major == 9999 && deviceProp.minor == 9999){
		printf("Error: There is no device supporting CUDA.\n");
		exit(0);
	}
    
    //check if double is used and supported
#ifdef _DOUBLE_SUPPORT_REQUIRED_
	printf("Simulation requires full precision double values\n");
	if ((deviceProp.major < 2)&&(deviceProp.minor < 3)){
		printf("Error: Hardware does not support full precision double values!\n");
		exit(0);
	}
    
#endif

	//check 32 or 64bit
	x64_sys = (sizeof(void*)==8);
	if (x64_sys)
	{
		printf("64Bit System Detected\n");
	}
	else
	{
		printf("32Bit System Detected\n");
	}

	SM_START = 0;
	PADDING = 0;
  
	//copy padding and offset to GPU
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_SM_START), &SM_START, sizeof(int)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_PADDING), &PADDING, sizeof(int)));     
}

int closest_sqr_pow2(int x){
	int h, h_d;
	int l, l_d;
	
	//higher bound
	h = (int)pow(4, ceil(log(x)/log(4)));
	h_d = h-x;
	
	//escape early if x is square power of 2
	if (h_d == x)
		return x;
	
	//lower bound		
	l = (int)pow(4, floor(log(x)/log(4)));
	l_d = x-l;
	
	//closest bound
	if(h_d < l_d)
		return h;
	else 
		return l;
}

int is_sqr_pow2(int x){
	int r = (int)pow(4, ceil(log(x)/log(4)));
	return (r == x);
}

/* Unary function required for hipOccupancyMaxPotentialBlockSizeVariableSMem to avoid warnings */
int no_sm(int b){
	return 0;
}

/* Unary function to return shared memory size for reorder message kernels */
int reorder_messages_sm_size(int blockSize)
{
	return sizeof(unsigned int)*(blockSize+1);
}


void initialise(char * inputfile){

	//set the padding and offset values depending on architecture and OS
	setPaddingAndOffset();
  

	printf("Allocating Host and Device memeory\n");
  
	/* Agent memory allocation (CPU) */
	int xmachine_agent_SoA_size = sizeof(xmachine_memory_agent_list);
	h_agents_default = (xmachine_memory_agent_list*)malloc(xmachine_agent_SoA_size);
	int xmachine_navmap_SoA_size = sizeof(xmachine_memory_navmap_list);
	h_navmaps_static = (xmachine_memory_navmap_list*)malloc(xmachine_navmap_SoA_size);

	/* Message memory allocation (CPU) */
	int message_pedestrian_location_SoA_size = sizeof(xmachine_message_pedestrian_location_list);
	h_pedestrian_locations = (xmachine_message_pedestrian_location_list*)malloc(message_pedestrian_location_SoA_size);
	int message_navmap_cell_SoA_size = sizeof(xmachine_message_navmap_cell_list);
	h_navmap_cells = (xmachine_message_navmap_cell_list*)malloc(message_navmap_cell_SoA_size);

	//Exit if agent or message buffer sizes are to small for function outpus
			
	/* Set spatial partitioning pedestrian_location message variables (min_bounds, max_bounds)*/
	h_message_pedestrian_location_radius = (float)0.025;
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_message_pedestrian_location_radius), &h_message_pedestrian_location_radius, sizeof(float)));	
	    h_message_pedestrian_location_min_bounds = make_float3((float)-1.0, (float)-1.0, (float)0.0);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_message_pedestrian_location_min_bounds), &h_message_pedestrian_location_min_bounds, sizeof(float3)));	
	h_message_pedestrian_location_max_bounds = make_float3((float)1.0, (float)1.0, (float)0.025);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_message_pedestrian_location_max_bounds), &h_message_pedestrian_location_max_bounds, sizeof(float3)));	
	h_message_pedestrian_location_partitionDim.x = (int)ceil((h_message_pedestrian_location_max_bounds.x - h_message_pedestrian_location_min_bounds.x)/h_message_pedestrian_location_radius);
	h_message_pedestrian_location_partitionDim.y = (int)ceil((h_message_pedestrian_location_max_bounds.y - h_message_pedestrian_location_min_bounds.y)/h_message_pedestrian_location_radius);
	h_message_pedestrian_location_partitionDim.z = (int)ceil((h_message_pedestrian_location_max_bounds.z - h_message_pedestrian_location_min_bounds.z)/h_message_pedestrian_location_radius);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_message_pedestrian_location_partitionDim), &h_message_pedestrian_location_partitionDim, sizeof(int3)));	
	
	
	/* Set discrete navmap_cell message variables (range, width)*/
	h_message_navmap_cell_range = 0; //from xml
	h_message_navmap_cell_width = (int)floor(sqrt((float)xmachine_message_navmap_cell_MAX));
	//check the width
	if (!is_sqr_pow2(xmachine_message_navmap_cell_MAX)){
		printf("ERROR: navmap_cell message max must be a square power of 2 for a 2D discrete message grid!\n");
		exit(0);
	}
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_message_navmap_cell_range), &h_message_navmap_cell_range, sizeof(int)));	
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_message_navmap_cell_width), &h_message_navmap_cell_width, sizeof(int)));
	
	/* Check that population size is a square power of 2*/
	if (!is_sqr_pow2(xmachine_memory_navmap_MAX)){
		printf("ERROR: navmaps agent count must be a square power of 2!\n");
		exit(0);
	}
	h_xmachine_memory_navmap_pop_width = (int)sqrt(xmachine_memory_navmap_MAX);
	

	//read initial states
	readInitialStates(inputfile, h_agents_default, &h_xmachine_memory_agent_default_count, h_navmaps_static, &h_xmachine_memory_navmap_static_count);
	
	
	/* agent Agent memory allocation (GPU) */
	gpuErrchk( hipMalloc( (void**) &d_agents, xmachine_agent_SoA_size));
	gpuErrchk( hipMalloc( (void**) &d_agents_swap, xmachine_agent_SoA_size));
	gpuErrchk( hipMalloc( (void**) &d_agents_new, xmachine_agent_SoA_size));
    //continuous agent sort identifiers
  gpuErrchk( hipMalloc( (void**) &d_xmachine_memory_agent_keys, xmachine_memory_agent_MAX* sizeof(uint)));
	gpuErrchk( hipMalloc( (void**) &d_xmachine_memory_agent_values, xmachine_memory_agent_MAX* sizeof(uint)));
	/* default memory allocation (GPU) */
	gpuErrchk( hipMalloc( (void**) &d_agents_default, xmachine_agent_SoA_size));
	gpuErrchk( hipMemcpy( d_agents_default, h_agents_default, xmachine_agent_SoA_size, hipMemcpyHostToDevice));
    
	/* navmap Agent memory allocation (GPU) */
	gpuErrchk( hipMalloc( (void**) &d_navmaps, xmachine_navmap_SoA_size));
	gpuErrchk( hipMalloc( (void**) &d_navmaps_swap, xmachine_navmap_SoA_size));
	gpuErrchk( hipMalloc( (void**) &d_navmaps_new, xmachine_navmap_SoA_size));
    
	/* static memory allocation (GPU) */
	gpuErrchk( hipMalloc( (void**) &d_navmaps_static, xmachine_navmap_SoA_size));
	gpuErrchk( hipMemcpy( d_navmaps_static, h_navmaps_static, xmachine_navmap_SoA_size, hipMemcpyHostToDevice));
    
	/* pedestrian_location Message memory allocation (GPU) */
	gpuErrchk( hipMalloc( (void**) &d_pedestrian_locations, message_pedestrian_location_SoA_size));
	gpuErrchk( hipMalloc( (void**) &d_pedestrian_locations_swap, message_pedestrian_location_SoA_size));
	gpuErrchk( hipMemcpy( d_pedestrian_locations, h_pedestrian_locations, message_pedestrian_location_SoA_size, hipMemcpyHostToDevice));
	gpuErrchk( hipMalloc( (void**) &d_pedestrian_location_partition_matrix, sizeof(xmachine_message_pedestrian_location_PBM)));
#ifdef FAST_ATOMIC_SORTING
	gpuErrchk( hipMalloc( (void**) &d_xmachine_message_pedestrian_location_local_bin_index, xmachine_message_pedestrian_location_MAX* sizeof(uint)));
	gpuErrchk( hipMalloc( (void**) &d_xmachine_message_pedestrian_location_unsorted_index, xmachine_message_pedestrian_location_MAX* sizeof(uint)));
#else
	gpuErrchk( hipMalloc( (void**) &d_xmachine_message_pedestrian_location_keys, xmachine_message_pedestrian_location_MAX* sizeof(uint)));
	gpuErrchk( hipMalloc( (void**) &d_xmachine_message_pedestrian_location_values, xmachine_message_pedestrian_location_MAX* sizeof(uint)));
#endif
	
	/* navmap_cell Message memory allocation (GPU) */
	gpuErrchk( hipMalloc( (void**) &d_navmap_cells, message_navmap_cell_SoA_size));
	gpuErrchk( hipMalloc( (void**) &d_navmap_cells_swap, message_navmap_cell_SoA_size));
	gpuErrchk( hipMemcpy( d_navmap_cells, h_navmap_cells, message_navmap_cell_SoA_size, hipMemcpyHostToDevice));
		

	/*Set global condition counts*/

	/* RNG rand48 */
	int h_rand48_SoA_size = sizeof(RNG_rand48);
	h_rand48 = (RNG_rand48*)malloc(h_rand48_SoA_size);
	//allocate on GPU
	gpuErrchk( hipMalloc( (void**) &d_rand48, h_rand48_SoA_size));
	// calculate strided iteration constants
	static const unsigned long long a = 0x5DEECE66DLL, c = 0xB;
	int seed = 123;
	unsigned long long A, C;
	A = 1LL; C = 0LL;
	for (unsigned int i = 0; i < buffer_size_MAX; ++i) {
		C += A*c;
		A *= a;
	}
	h_rand48->A.x = A & 0xFFFFFFLL;
	h_rand48->A.y = (A >> 24) & 0xFFFFFFLL;
	h_rand48->C.x = C & 0xFFFFFFLL;
	h_rand48->C.y = (C >> 24) & 0xFFFFFFLL;
	// prepare first nThreads random numbers from seed
	unsigned long long x = (((unsigned long long)seed) << 16) | 0x330E;
	for (unsigned int i = 0; i < buffer_size_MAX; ++i) {
		x = a*x + c;
		h_rand48->seeds[i].x = x & 0xFFFFFFLL;
		h_rand48->seeds[i].y = (x >> 24) & 0xFFFFFFLL;
	}
	//copy to device
	gpuErrchk( hipMemcpy( d_rand48, h_rand48, h_rand48_SoA_size, hipMemcpyHostToDevice));

	/* Call all init functions */
	
  
  /* Init CUDA Streams for function layers */
  
  gpuErrchk(hipStreamCreate(&stream1));
  gpuErrchk(hipStreamCreate(&stream2));
} 


void sort_agents_default(void (*generate_key_value_pairs)(unsigned int* keys, unsigned int* values, xmachine_memory_agent_list* agents))
{
	int blockSize;
	int minGridSize;
	int gridSize;

	//generate sort keys
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, generate_key_value_pairs, no_sm, h_xmachine_memory_agent_default_count); 
	gridSize = (h_xmachine_memory_agent_default_count + blockSize - 1) / blockSize;    // Round up according to array size 
	generate_key_value_pairs<<<gridSize, blockSize>>>(d_xmachine_memory_agent_keys, d_xmachine_memory_agent_values, d_agents_default);
	gpuErrchkLaunch();

	//updated Thrust sort
	thrust::sort_by_key( thrust::device_pointer_cast(d_xmachine_memory_agent_keys),  thrust::device_pointer_cast(d_xmachine_memory_agent_keys) + h_xmachine_memory_agent_default_count,  thrust::device_pointer_cast(d_xmachine_memory_agent_values));
	gpuErrchkLaunch();

	//reorder agents
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, reorder_agent_agents, no_sm, h_xmachine_memory_agent_default_count); 
	gridSize = (h_xmachine_memory_agent_default_count + blockSize - 1) / blockSize;    // Round up according to array size 
	reorder_agent_agents<<<gridSize, blockSize>>>(d_xmachine_memory_agent_values, d_agents_default, d_agents_swap);
	gpuErrchkLaunch();

	//swap
	xmachine_memory_agent_list* d_agents_temp = d_agents_default;
	d_agents_default = d_agents_swap;
	d_agents_swap = d_agents_temp;	
}


void cleanup(){

	/* Agent data free*/
	
	/* agent Agent variables */
	gpuErrchk(hipFree(d_agents));
	gpuErrchk(hipFree(d_agents_swap));
	gpuErrchk(hipFree(d_agents_new));
	
	free( h_agents_default);
	gpuErrchk(hipFree(d_agents_default));
	
	/* navmap Agent variables */
	gpuErrchk(hipFree(d_navmaps));
	gpuErrchk(hipFree(d_navmaps_swap));
	gpuErrchk(hipFree(d_navmaps_new));
	
	free( h_navmaps_static);
	gpuErrchk(hipFree(d_navmaps_static));
	

	/* Message data free */
	
	/* pedestrian_location Message variables */
	free( h_pedestrian_locations);
	gpuErrchk(hipFree(d_pedestrian_locations));
	gpuErrchk(hipFree(d_pedestrian_locations_swap));
	gpuErrchk(hipFree(d_pedestrian_location_partition_matrix));
#ifdef FAST_ATOMIC_SORTING
	gpuErrchk(hipFree(d_xmachine_message_pedestrian_location_local_bin_index));
	gpuErrchk(hipFree(d_xmachine_message_pedestrian_location_unsorted_index));
#else
	gpuErrchk(hipFree(d_xmachine_message_pedestrian_location_keys));
	gpuErrchk(hipFree(d_xmachine_message_pedestrian_location_values));
#endif
	
	/* navmap_cell Message variables */
	free( h_navmap_cells);
	gpuErrchk(hipFree(d_navmap_cells));
	gpuErrchk(hipFree(d_navmap_cells_swap));
	
  
  /* CUDA Streams for function layers */
  
  gpuErrchk(hipStreamDestroy(stream1));
  gpuErrchk(hipStreamDestroy(stream2));
}

void singleIteration(){

	/* set all non partitioned and spatial partitionded message counts to 0*/
	h_message_pedestrian_location_count = 0;
	//upload to device constant
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_message_pedestrian_location_count), &h_message_pedestrian_location_count, sizeof(int)));
	

	/* Call agent functions in order itterating through the layer functions */
	
	/* Layer 1*/
	navmap_generate_pedestrians(stream1);
	hipDeviceSynchronize();
  
	/* Layer 2*/
	agent_output_pedestrian_location(stream1);
	navmap_output_navmap_cells(stream2);
	hipDeviceSynchronize();
  
	/* Layer 3*/
	agent_avoid_pedestrians(stream1);
	hipDeviceSynchronize();
  
	/* Layer 4*/
	agent_force_flow(stream1);
	hipDeviceSynchronize();
  
	/* Layer 5*/
	agent_move(stream1);
	hipDeviceSynchronize();
  
}

/* Environment functions */


void set_EMMISION_RATE_EXIT1(float* h_EMMISION_RATE_EXIT1){
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(EMMISION_RATE_EXIT1), h_EMMISION_RATE_EXIT1, sizeof(float)));
}

void set_EMMISION_RATE_EXIT2(float* h_EMMISION_RATE_EXIT2){
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(EMMISION_RATE_EXIT2), h_EMMISION_RATE_EXIT2, sizeof(float)));
}

void set_EMMISION_RATE_EXIT3(float* h_EMMISION_RATE_EXIT3){
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(EMMISION_RATE_EXIT3), h_EMMISION_RATE_EXIT3, sizeof(float)));
}

void set_EMMISION_RATE_EXIT4(float* h_EMMISION_RATE_EXIT4){
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(EMMISION_RATE_EXIT4), h_EMMISION_RATE_EXIT4, sizeof(float)));
}

void set_EMMISION_RATE_EXIT5(float* h_EMMISION_RATE_EXIT5){
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(EMMISION_RATE_EXIT5), h_EMMISION_RATE_EXIT5, sizeof(float)));
}

void set_EMMISION_RATE_EXIT6(float* h_EMMISION_RATE_EXIT6){
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(EMMISION_RATE_EXIT6), h_EMMISION_RATE_EXIT6, sizeof(float)));
}

void set_EMMISION_RATE_EXIT7(float* h_EMMISION_RATE_EXIT7){
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(EMMISION_RATE_EXIT7), h_EMMISION_RATE_EXIT7, sizeof(float)));
}

void set_EXIT1_PROBABILITY(int* h_EXIT1_PROBABILITY){
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(EXIT1_PROBABILITY), h_EXIT1_PROBABILITY, sizeof(int)));
}

void set_EXIT2_PROBABILITY(int* h_EXIT2_PROBABILITY){
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(EXIT2_PROBABILITY), h_EXIT2_PROBABILITY, sizeof(int)));
}

void set_EXIT3_PROBABILITY(int* h_EXIT3_PROBABILITY){
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(EXIT3_PROBABILITY), h_EXIT3_PROBABILITY, sizeof(int)));
}

void set_EXIT4_PROBABILITY(int* h_EXIT4_PROBABILITY){
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(EXIT4_PROBABILITY), h_EXIT4_PROBABILITY, sizeof(int)));
}

void set_EXIT5_PROBABILITY(int* h_EXIT5_PROBABILITY){
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(EXIT5_PROBABILITY), h_EXIT5_PROBABILITY, sizeof(int)));
}

void set_EXIT6_PROBABILITY(int* h_EXIT6_PROBABILITY){
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(EXIT6_PROBABILITY), h_EXIT6_PROBABILITY, sizeof(int)));
}

void set_EXIT7_PROBABILITY(int* h_EXIT7_PROBABILITY){
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(EXIT7_PROBABILITY), h_EXIT7_PROBABILITY, sizeof(int)));
}

void set_EXIT1_STATE(int* h_EXIT1_STATE){
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(EXIT1_STATE), h_EXIT1_STATE, sizeof(int)));
}

void set_EXIT2_STATE(int* h_EXIT2_STATE){
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(EXIT2_STATE), h_EXIT2_STATE, sizeof(int)));
}

void set_EXIT3_STATE(int* h_EXIT3_STATE){
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(EXIT3_STATE), h_EXIT3_STATE, sizeof(int)));
}

void set_EXIT4_STATE(int* h_EXIT4_STATE){
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(EXIT4_STATE), h_EXIT4_STATE, sizeof(int)));
}

void set_EXIT5_STATE(int* h_EXIT5_STATE){
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(EXIT5_STATE), h_EXIT5_STATE, sizeof(int)));
}

void set_EXIT6_STATE(int* h_EXIT6_STATE){
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(EXIT6_STATE), h_EXIT6_STATE, sizeof(int)));
}

void set_EXIT7_STATE(int* h_EXIT7_STATE){
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(EXIT7_STATE), h_EXIT7_STATE, sizeof(int)));
}

void set_TIME_SCALER(float* h_TIME_SCALER){
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(TIME_SCALER), h_TIME_SCALER, sizeof(float)));
}

void set_STEER_WEIGHT(float* h_STEER_WEIGHT){
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(STEER_WEIGHT), h_STEER_WEIGHT, sizeof(float)));
}

void set_AVOID_WEIGHT(float* h_AVOID_WEIGHT){
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(AVOID_WEIGHT), h_AVOID_WEIGHT, sizeof(float)));
}

void set_COLLISION_WEIGHT(float* h_COLLISION_WEIGHT){
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(COLLISION_WEIGHT), h_COLLISION_WEIGHT, sizeof(float)));
}

void set_GOAL_WEIGHT(float* h_GOAL_WEIGHT){
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(GOAL_WEIGHT), h_GOAL_WEIGHT, sizeof(float)));
}


/* Agent data access functions*/

    
int get_agent_agent_MAX_count(){
    return xmachine_memory_agent_MAX;
}


int get_agent_agent_default_count(){
	//continuous agent
	return h_xmachine_memory_agent_default_count;
	
}

xmachine_memory_agent_list* get_device_agent_default_agents(){
	return d_agents_default;
}

xmachine_memory_agent_list* get_host_agent_default_agents(){
	return h_agents_default;
}

    
int get_agent_navmap_MAX_count(){
    return xmachine_memory_navmap_MAX;
}


int get_agent_navmap_static_count(){
	//discrete agent 
	return xmachine_memory_navmap_MAX;
}

xmachine_memory_navmap_list* get_device_navmap_static_agents(){
	return d_navmaps_static;
}

xmachine_memory_navmap_list* get_host_navmap_static_agents(){
	return h_navmaps_static;
}

int get_navmap_population_width(){
  return h_xmachine_memory_navmap_pop_width;
}


/* Agent functions */


	
/* Shared memory size calculator for agent function */
int agent_output_pedestrian_location_sm_size(int blockSize){
	int sm_size;
	sm_size = SM_START;
  
	return sm_size;
}

/** agent_output_pedestrian_location
 * Agent function prototype for output_pedestrian_location function of agent agent
 */
void agent_output_pedestrian_location(hipStream_t &stream){

	int sm_size;
	int blockSize;
	int minGridSize;
	int gridSize;
	int state_list_size;
	dim3 g; //grid for agent func
	dim3 b; //block for agent func

	
	//CHECK THE CURRENT STATE LIST COUNT IS NOT EQUAL TO 0
	
	if (h_xmachine_memory_agent_default_count == 0)
	{
		return;
	}
	
	
	//SET SM size to 0 and save state list size for occupancy calculations
	sm_size = SM_START;
	state_list_size = h_xmachine_memory_agent_default_count;

	

	//******************************** AGENT FUNCTION CONDITION *********************
	//THERE IS NOT A FUNCTION CONDITION
	//currentState maps to working list
	xmachine_memory_agent_list* agents_default_temp = d_agents;
	d_agents = d_agents_default;
	d_agents_default = agents_default_temp;
	//set working count to current state count
	h_xmachine_memory_agent_count = h_xmachine_memory_agent_default_count;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_agent_count), &h_xmachine_memory_agent_count, sizeof(int)));	
	//set current state count to 0
	h_xmachine_memory_agent_default_count = 0;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_agent_default_count), &h_xmachine_memory_agent_default_count, sizeof(int)));	
	
 

	//******************************** AGENT FUNCTION *******************************

	
	//CONTINUOUS AGENT CHECK FUNCTION OUTPUT BUFFERS FOR OUT OF BOUNDS
	if (h_message_pedestrian_location_count + h_xmachine_memory_agent_count > xmachine_message_pedestrian_location_MAX){
		printf("Error: Buffer size of pedestrian_location message will be exceeded in function output_pedestrian_location\n");
		exit(0);
	}
	
	
	//calculate the grid block size for main agent function
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, GPUFLAME_output_pedestrian_location, agent_output_pedestrian_location_sm_size, state_list_size);
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	b.x = blockSize;
	g.x = gridSize;
	
	sm_size = agent_output_pedestrian_location_sm_size(blockSize);
	
	
	
	//SET THE OUTPUT MESSAGE TYPE FOR CONTINUOUS AGENTS
	//Set the message_type for non partitioned and spatially partitioned message outputs
	h_message_pedestrian_location_output_type = single_message;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_message_pedestrian_location_output_type), &h_message_pedestrian_location_output_type, sizeof(int)));
	
	
	//MAIN XMACHINE FUNCTION CALL (output_pedestrian_location)
	//Reallocate   : false
	//Input        : 
	//Output       : pedestrian_location
	//Agent Output : 
	GPUFLAME_output_pedestrian_location<<<g, b, sm_size, stream>>>(d_agents, d_pedestrian_locations);
	gpuErrchkLaunch();
	
	
	//CONTINUOUS AGENTS SCATTER NON PARTITIONED OPTIONAL OUTPUT MESSAGES
	
	//UPDATE MESSAGE COUNTS FOR CONTINUOUS AGENTS WITH NON PARTITIONED MESSAGE OUTPUT 
	h_message_pedestrian_location_count += h_xmachine_memory_agent_count;	
	//Copy count to device
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_message_pedestrian_location_count), &h_message_pedestrian_location_count, sizeof(int)));	
	
#ifdef FAST_ATOMIC_SORTING
  //USE ATOMICS TO BUILD PARTITION BOUNDARY
	//reset partition matrix
	gpuErrchk( hipMemset( (void*) d_pedestrian_location_partition_matrix, 0, sizeof(xmachine_message_pedestrian_location_PBM)));
  //
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, hist_pedestrian_location_messages, no_sm, state_list_size); 
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	hist_pedestrian_location_messages<<<gridSize, blockSize, 0, stream>>>(d_xmachine_message_pedestrian_location_local_bin_index, d_xmachine_message_pedestrian_location_unsorted_index, d_pedestrian_location_partition_matrix->end_or_count, d_pedestrian_locations);
	gpuErrchkLaunch();
	
	thrust::device_ptr<int> ptr_count = thrust::device_pointer_cast(d_pedestrian_location_partition_matrix->end_or_count);
	thrust::device_ptr<int> ptr_index = thrust::device_pointer_cast(d_pedestrian_location_partition_matrix->start);
	thrust::exclusive_scan(thrust::cuda::par.on(stream), ptr_count, ptr_count + xmachine_message_pedestrian_location_grid_size, ptr_index); // scan
	
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, reorder_pedestrian_location_messages, no_sm, state_list_size); 
	gridSize = (state_list_size + blockSize - 1) / blockSize; 	// Round up according to array size 
	reorder_pedestrian_location_messages <<<gridSize, blockSize, 0, stream>>>(d_xmachine_message_pedestrian_location_local_bin_index, d_xmachine_message_pedestrian_location_unsorted_index, d_pedestrian_location_partition_matrix->start, d_pedestrian_locations, d_pedestrian_locations_swap);
	gpuErrchkLaunch();
#else
	//HASH, SORT, REORDER AND BUILD PMB FOR SPATIAL PARTITIONING MESSAGE OUTPUTS
	//Get message hash values for sorting
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, hash_pedestrian_location_messages, no_sm, state_list_size); 
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	hash_pedestrian_location_messages<<<gridSize, blockSize, 0, stream>>>(d_xmachine_message_pedestrian_location_keys, d_xmachine_message_pedestrian_location_values, d_pedestrian_locations);
	gpuErrchkLaunch();
	//Sort
	thrust::sort_by_key(thrust::cuda::par.on(stream), thrust::device_pointer_cast(d_xmachine_message_pedestrian_location_keys),  thrust::device_pointer_cast(d_xmachine_message_pedestrian_location_keys) + h_message_pedestrian_location_count,  thrust::device_pointer_cast(d_xmachine_message_pedestrian_location_values));
	gpuErrchkLaunch();
	//reorder and build pcb
	gpuErrchk(hipMemset(d_pedestrian_location_partition_matrix->start, 0xffffffff, xmachine_message_pedestrian_location_grid_size* sizeof(int)));
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, reorder_pedestrian_location_messages, reorder_messages_sm_size, state_list_size); 
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	int reorder_sm_size = reorder_messages_sm_size(blockSize);
	reorder_pedestrian_location_messages<<<gridSize, blockSize, reorder_sm_size, stream>>>(d_xmachine_message_pedestrian_location_keys, d_xmachine_message_pedestrian_location_values, d_pedestrian_location_partition_matrix, d_pedestrian_locations, d_pedestrian_locations_swap);
	gpuErrchkLaunch();
#endif
	//swap ordered list
	xmachine_message_pedestrian_location_list* d_pedestrian_locations_temp = d_pedestrian_locations;
	d_pedestrian_locations = d_pedestrian_locations_swap;
	d_pedestrian_locations_swap = d_pedestrian_locations_temp;
	
	
	//************************ MOVE AGENTS TO NEXT STATE ****************************
    
	//check the working agents wont exceed the buffer size in the new state list
	if (h_xmachine_memory_agent_default_count+h_xmachine_memory_agent_count > xmachine_memory_agent_MAX){
		printf("Error: Buffer size of output_pedestrian_location agents in state default will be exceeded moving working agents to next state in function output_pedestrian_location\n");
		exit(0);
	}
	//append agents to next state list
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, append_agent_Agents, no_sm, state_list_size); 
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	append_agent_Agents<<<gridSize, blockSize, 0, stream>>>(d_agents_default, d_agents, h_xmachine_memory_agent_default_count, h_xmachine_memory_agent_count);
	gpuErrchkLaunch();
	//update new state agent size
	h_xmachine_memory_agent_default_count += h_xmachine_memory_agent_count;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_agent_default_count), &h_xmachine_memory_agent_default_count, sizeof(int)));	
	
	
}



	
/* Shared memory size calculator for agent function */
int agent_avoid_pedestrians_sm_size(int blockSize){
	int sm_size;
	sm_size = SM_START;
  //Continuous agent and message input is spatially partitioned
	sm_size += (blockSize * sizeof(xmachine_message_pedestrian_location));
	
	//all continuous agent types require single 32bit word per thread offset (to avoid sm bank conflicts)
	sm_size += (blockSize * PADDING);
	
	return sm_size;
}

/** agent_avoid_pedestrians
 * Agent function prototype for avoid_pedestrians function of agent agent
 */
void agent_avoid_pedestrians(hipStream_t &stream){

	int sm_size;
	int blockSize;
	int minGridSize;
	int gridSize;
	int state_list_size;
	dim3 g; //grid for agent func
	dim3 b; //block for agent func

	
	//CHECK THE CURRENT STATE LIST COUNT IS NOT EQUAL TO 0
	
	if (h_xmachine_memory_agent_default_count == 0)
	{
		return;
	}
	
	
	//SET SM size to 0 and save state list size for occupancy calculations
	sm_size = SM_START;
	state_list_size = h_xmachine_memory_agent_default_count;

	

	//******************************** AGENT FUNCTION CONDITION *********************
	//THERE IS NOT A FUNCTION CONDITION
	//currentState maps to working list
	xmachine_memory_agent_list* agents_default_temp = d_agents;
	d_agents = d_agents_default;
	d_agents_default = agents_default_temp;
	//set working count to current state count
	h_xmachine_memory_agent_count = h_xmachine_memory_agent_default_count;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_agent_count), &h_xmachine_memory_agent_count, sizeof(int)));	
	//set current state count to 0
	h_xmachine_memory_agent_default_count = 0;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_agent_default_count), &h_xmachine_memory_agent_default_count, sizeof(int)));	
	
 

	//******************************** AGENT FUNCTION *******************************

	
	
	//calculate the grid block size for main agent function
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, GPUFLAME_avoid_pedestrians, agent_avoid_pedestrians_sm_size, state_list_size);
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	b.x = blockSize;
	g.x = gridSize;
	
	sm_size = agent_avoid_pedestrians_sm_size(blockSize);
	
	
	
	//BIND APPROPRIATE MESSAGE INPUT VARIABLES TO TEXTURES (to make use of the texture cache)
	//continuous agent with discrete or partitioned message input uses texture caching
	size_t tex_xmachine_message_pedestrian_location_x_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_pedestrian_location_x_byte_offset, tex_xmachine_message_pedestrian_location_x, d_pedestrian_locations->x, sizeof(int)*xmachine_message_pedestrian_location_MAX));
	h_tex_xmachine_message_pedestrian_location_x_offset = (int)tex_xmachine_message_pedestrian_location_x_byte_offset / sizeof(float);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_pedestrian_location_x_offset), &h_tex_xmachine_message_pedestrian_location_x_offset, sizeof(int)));
	size_t tex_xmachine_message_pedestrian_location_y_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_pedestrian_location_y_byte_offset, tex_xmachine_message_pedestrian_location_y, d_pedestrian_locations->y, sizeof(int)*xmachine_message_pedestrian_location_MAX));
	h_tex_xmachine_message_pedestrian_location_y_offset = (int)tex_xmachine_message_pedestrian_location_y_byte_offset / sizeof(float);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_pedestrian_location_y_offset), &h_tex_xmachine_message_pedestrian_location_y_offset, sizeof(int)));
	size_t tex_xmachine_message_pedestrian_location_z_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_pedestrian_location_z_byte_offset, tex_xmachine_message_pedestrian_location_z, d_pedestrian_locations->z, sizeof(int)*xmachine_message_pedestrian_location_MAX));
	h_tex_xmachine_message_pedestrian_location_z_offset = (int)tex_xmachine_message_pedestrian_location_z_byte_offset / sizeof(float);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_pedestrian_location_z_offset), &h_tex_xmachine_message_pedestrian_location_z_offset, sizeof(int)));
	//bind pbm start and end indices to textures
	size_t tex_xmachine_message_pedestrian_location_pbm_start_byte_offset;
	size_t tex_xmachine_message_pedestrian_location_pbm_end_or_count_byte_offset;
	gpuErrchk( hipBindTexture(&tex_xmachine_message_pedestrian_location_pbm_start_byte_offset, tex_xmachine_message_pedestrian_location_pbm_start, d_pedestrian_location_partition_matrix->start, sizeof(int)*xmachine_message_pedestrian_location_grid_size));
	h_tex_xmachine_message_pedestrian_location_pbm_start_offset = (int)tex_xmachine_message_pedestrian_location_pbm_start_byte_offset / sizeof(int);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_pedestrian_location_pbm_start_offset), &h_tex_xmachine_message_pedestrian_location_pbm_start_offset, sizeof(int)));
	gpuErrchk( hipBindTexture(&tex_xmachine_message_pedestrian_location_pbm_end_or_count_byte_offset, tex_xmachine_message_pedestrian_location_pbm_end_or_count, d_pedestrian_location_partition_matrix->end_or_count, sizeof(int)*xmachine_message_pedestrian_location_grid_size));
  h_tex_xmachine_message_pedestrian_location_pbm_end_or_count_offset = (int)tex_xmachine_message_pedestrian_location_pbm_end_or_count_byte_offset / sizeof(int);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_pedestrian_location_pbm_end_or_count_offset), &h_tex_xmachine_message_pedestrian_location_pbm_end_or_count_offset, sizeof(int)));

	
	
	//MAIN XMACHINE FUNCTION CALL (avoid_pedestrians)
	//Reallocate   : false
	//Input        : pedestrian_location
	//Output       : 
	//Agent Output : 
	GPUFLAME_avoid_pedestrians<<<g, b, sm_size, stream>>>(d_agents, d_pedestrian_locations, d_pedestrian_location_partition_matrix, d_rand48);
	gpuErrchkLaunch();
	
	
	//UNBIND MESSAGE INPUT VARIABLE TEXTURES
	//continuous agent with discrete or partitioned message input uses texture caching
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_pedestrian_location_x));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_pedestrian_location_y));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_pedestrian_location_z));
	//unbind pbm indices
    gpuErrchk( hipUnbindTexture(tex_xmachine_message_pedestrian_location_pbm_start));
    gpuErrchk( hipUnbindTexture(tex_xmachine_message_pedestrian_location_pbm_end_or_count));
    
	
	//************************ MOVE AGENTS TO NEXT STATE ****************************
    
	//check the working agents wont exceed the buffer size in the new state list
	if (h_xmachine_memory_agent_default_count+h_xmachine_memory_agent_count > xmachine_memory_agent_MAX){
		printf("Error: Buffer size of avoid_pedestrians agents in state default will be exceeded moving working agents to next state in function avoid_pedestrians\n");
		exit(0);
	}
	//append agents to next state list
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, append_agent_Agents, no_sm, state_list_size); 
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	append_agent_Agents<<<gridSize, blockSize, 0, stream>>>(d_agents_default, d_agents, h_xmachine_memory_agent_default_count, h_xmachine_memory_agent_count);
	gpuErrchkLaunch();
	//update new state agent size
	h_xmachine_memory_agent_default_count += h_xmachine_memory_agent_count;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_agent_default_count), &h_xmachine_memory_agent_default_count, sizeof(int)));	
	
	
}



	
/* Shared memory size calculator for agent function */
int agent_force_flow_sm_size(int blockSize){
	int sm_size;
	sm_size = SM_START;
  //Continuous agent and message input has discrete partitioning
	//Will be reading using texture lookups so sm size can stay the same but need to hold range and width
	sm_size += (blockSize * sizeof(xmachine_message_navmap_cell));
	
	//all continuous agent types require single 32bit word per thread offset (to avoid sm bank conflicts)
	sm_size += (blockSize * PADDING);
	
	return sm_size;
}

/** agent_force_flow
 * Agent function prototype for force_flow function of agent agent
 */
void agent_force_flow(hipStream_t &stream){

	int sm_size;
	int blockSize;
	int minGridSize;
	int gridSize;
	int state_list_size;
	dim3 g; //grid for agent func
	dim3 b; //block for agent func

	
	//CHECK THE CURRENT STATE LIST COUNT IS NOT EQUAL TO 0
	
	if (h_xmachine_memory_agent_default_count == 0)
	{
		return;
	}
	
	
	//SET SM size to 0 and save state list size for occupancy calculations
	sm_size = SM_START;
	state_list_size = h_xmachine_memory_agent_default_count;

	

	//******************************** AGENT FUNCTION CONDITION *********************
	//THERE IS NOT A FUNCTION CONDITION
	//currentState maps to working list
	xmachine_memory_agent_list* agents_default_temp = d_agents;
	d_agents = d_agents_default;
	d_agents_default = agents_default_temp;
	//set working count to current state count
	h_xmachine_memory_agent_count = h_xmachine_memory_agent_default_count;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_agent_count), &h_xmachine_memory_agent_count, sizeof(int)));	
	//set current state count to 0
	h_xmachine_memory_agent_default_count = 0;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_agent_default_count), &h_xmachine_memory_agent_default_count, sizeof(int)));	
	
 

	//******************************** AGENT FUNCTION *******************************

	
	
	//calculate the grid block size for main agent function
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, GPUFLAME_force_flow, agent_force_flow_sm_size, state_list_size);
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	b.x = blockSize;
	g.x = gridSize;
	
	sm_size = agent_force_flow_sm_size(blockSize);
	
	
	
	//BIND APPROPRIATE MESSAGE INPUT VARIABLES TO TEXTURES (to make use of the texture cache)
	//continuous agent with discrete or partitioned message input uses texture caching
	size_t tex_xmachine_message_navmap_cell_x_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_navmap_cell_x_byte_offset, tex_xmachine_message_navmap_cell_x, d_navmap_cells->x, sizeof(int)*xmachine_message_navmap_cell_MAX));
	h_tex_xmachine_message_navmap_cell_x_offset = (int)tex_xmachine_message_navmap_cell_x_byte_offset / sizeof(int);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_navmap_cell_x_offset), &h_tex_xmachine_message_navmap_cell_x_offset, sizeof(int)));
	size_t tex_xmachine_message_navmap_cell_y_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_navmap_cell_y_byte_offset, tex_xmachine_message_navmap_cell_y, d_navmap_cells->y, sizeof(int)*xmachine_message_navmap_cell_MAX));
	h_tex_xmachine_message_navmap_cell_y_offset = (int)tex_xmachine_message_navmap_cell_y_byte_offset / sizeof(int);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_navmap_cell_y_offset), &h_tex_xmachine_message_navmap_cell_y_offset, sizeof(int)));
	size_t tex_xmachine_message_navmap_cell_exit_no_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_navmap_cell_exit_no_byte_offset, tex_xmachine_message_navmap_cell_exit_no, d_navmap_cells->exit_no, sizeof(int)*xmachine_message_navmap_cell_MAX));
	h_tex_xmachine_message_navmap_cell_exit_no_offset = (int)tex_xmachine_message_navmap_cell_exit_no_byte_offset / sizeof(int);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_navmap_cell_exit_no_offset), &h_tex_xmachine_message_navmap_cell_exit_no_offset, sizeof(int)));
	size_t tex_xmachine_message_navmap_cell_height_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_navmap_cell_height_byte_offset, tex_xmachine_message_navmap_cell_height, d_navmap_cells->height, sizeof(int)*xmachine_message_navmap_cell_MAX));
	h_tex_xmachine_message_navmap_cell_height_offset = (int)tex_xmachine_message_navmap_cell_height_byte_offset / sizeof(float);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_navmap_cell_height_offset), &h_tex_xmachine_message_navmap_cell_height_offset, sizeof(int)));
	size_t tex_xmachine_message_navmap_cell_collision_x_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_navmap_cell_collision_x_byte_offset, tex_xmachine_message_navmap_cell_collision_x, d_navmap_cells->collision_x, sizeof(int)*xmachine_message_navmap_cell_MAX));
	h_tex_xmachine_message_navmap_cell_collision_x_offset = (int)tex_xmachine_message_navmap_cell_collision_x_byte_offset / sizeof(float);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_navmap_cell_collision_x_offset), &h_tex_xmachine_message_navmap_cell_collision_x_offset, sizeof(int)));
	size_t tex_xmachine_message_navmap_cell_collision_y_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_navmap_cell_collision_y_byte_offset, tex_xmachine_message_navmap_cell_collision_y, d_navmap_cells->collision_y, sizeof(int)*xmachine_message_navmap_cell_MAX));
	h_tex_xmachine_message_navmap_cell_collision_y_offset = (int)tex_xmachine_message_navmap_cell_collision_y_byte_offset / sizeof(float);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_navmap_cell_collision_y_offset), &h_tex_xmachine_message_navmap_cell_collision_y_offset, sizeof(int)));
	size_t tex_xmachine_message_navmap_cell_exit0_x_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_navmap_cell_exit0_x_byte_offset, tex_xmachine_message_navmap_cell_exit0_x, d_navmap_cells->exit0_x, sizeof(int)*xmachine_message_navmap_cell_MAX));
	h_tex_xmachine_message_navmap_cell_exit0_x_offset = (int)tex_xmachine_message_navmap_cell_exit0_x_byte_offset / sizeof(float);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_navmap_cell_exit0_x_offset), &h_tex_xmachine_message_navmap_cell_exit0_x_offset, sizeof(int)));
	size_t tex_xmachine_message_navmap_cell_exit0_y_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_navmap_cell_exit0_y_byte_offset, tex_xmachine_message_navmap_cell_exit0_y, d_navmap_cells->exit0_y, sizeof(int)*xmachine_message_navmap_cell_MAX));
	h_tex_xmachine_message_navmap_cell_exit0_y_offset = (int)tex_xmachine_message_navmap_cell_exit0_y_byte_offset / sizeof(float);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_navmap_cell_exit0_y_offset), &h_tex_xmachine_message_navmap_cell_exit0_y_offset, sizeof(int)));
	size_t tex_xmachine_message_navmap_cell_exit1_x_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_navmap_cell_exit1_x_byte_offset, tex_xmachine_message_navmap_cell_exit1_x, d_navmap_cells->exit1_x, sizeof(int)*xmachine_message_navmap_cell_MAX));
	h_tex_xmachine_message_navmap_cell_exit1_x_offset = (int)tex_xmachine_message_navmap_cell_exit1_x_byte_offset / sizeof(float);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_navmap_cell_exit1_x_offset), &h_tex_xmachine_message_navmap_cell_exit1_x_offset, sizeof(int)));
	size_t tex_xmachine_message_navmap_cell_exit1_y_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_navmap_cell_exit1_y_byte_offset, tex_xmachine_message_navmap_cell_exit1_y, d_navmap_cells->exit1_y, sizeof(int)*xmachine_message_navmap_cell_MAX));
	h_tex_xmachine_message_navmap_cell_exit1_y_offset = (int)tex_xmachine_message_navmap_cell_exit1_y_byte_offset / sizeof(float);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_navmap_cell_exit1_y_offset), &h_tex_xmachine_message_navmap_cell_exit1_y_offset, sizeof(int)));
	size_t tex_xmachine_message_navmap_cell_exit2_x_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_navmap_cell_exit2_x_byte_offset, tex_xmachine_message_navmap_cell_exit2_x, d_navmap_cells->exit2_x, sizeof(int)*xmachine_message_navmap_cell_MAX));
	h_tex_xmachine_message_navmap_cell_exit2_x_offset = (int)tex_xmachine_message_navmap_cell_exit2_x_byte_offset / sizeof(float);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_navmap_cell_exit2_x_offset), &h_tex_xmachine_message_navmap_cell_exit2_x_offset, sizeof(int)));
	size_t tex_xmachine_message_navmap_cell_exit2_y_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_navmap_cell_exit2_y_byte_offset, tex_xmachine_message_navmap_cell_exit2_y, d_navmap_cells->exit2_y, sizeof(int)*xmachine_message_navmap_cell_MAX));
	h_tex_xmachine_message_navmap_cell_exit2_y_offset = (int)tex_xmachine_message_navmap_cell_exit2_y_byte_offset / sizeof(float);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_navmap_cell_exit2_y_offset), &h_tex_xmachine_message_navmap_cell_exit2_y_offset, sizeof(int)));
	size_t tex_xmachine_message_navmap_cell_exit3_x_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_navmap_cell_exit3_x_byte_offset, tex_xmachine_message_navmap_cell_exit3_x, d_navmap_cells->exit3_x, sizeof(int)*xmachine_message_navmap_cell_MAX));
	h_tex_xmachine_message_navmap_cell_exit3_x_offset = (int)tex_xmachine_message_navmap_cell_exit3_x_byte_offset / sizeof(float);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_navmap_cell_exit3_x_offset), &h_tex_xmachine_message_navmap_cell_exit3_x_offset, sizeof(int)));
	size_t tex_xmachine_message_navmap_cell_exit3_y_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_navmap_cell_exit3_y_byte_offset, tex_xmachine_message_navmap_cell_exit3_y, d_navmap_cells->exit3_y, sizeof(int)*xmachine_message_navmap_cell_MAX));
	h_tex_xmachine_message_navmap_cell_exit3_y_offset = (int)tex_xmachine_message_navmap_cell_exit3_y_byte_offset / sizeof(float);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_navmap_cell_exit3_y_offset), &h_tex_xmachine_message_navmap_cell_exit3_y_offset, sizeof(int)));
	size_t tex_xmachine_message_navmap_cell_exit4_x_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_navmap_cell_exit4_x_byte_offset, tex_xmachine_message_navmap_cell_exit4_x, d_navmap_cells->exit4_x, sizeof(int)*xmachine_message_navmap_cell_MAX));
	h_tex_xmachine_message_navmap_cell_exit4_x_offset = (int)tex_xmachine_message_navmap_cell_exit4_x_byte_offset / sizeof(float);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_navmap_cell_exit4_x_offset), &h_tex_xmachine_message_navmap_cell_exit4_x_offset, sizeof(int)));
	size_t tex_xmachine_message_navmap_cell_exit4_y_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_navmap_cell_exit4_y_byte_offset, tex_xmachine_message_navmap_cell_exit4_y, d_navmap_cells->exit4_y, sizeof(int)*xmachine_message_navmap_cell_MAX));
	h_tex_xmachine_message_navmap_cell_exit4_y_offset = (int)tex_xmachine_message_navmap_cell_exit4_y_byte_offset / sizeof(float);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_navmap_cell_exit4_y_offset), &h_tex_xmachine_message_navmap_cell_exit4_y_offset, sizeof(int)));
	size_t tex_xmachine_message_navmap_cell_exit5_x_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_navmap_cell_exit5_x_byte_offset, tex_xmachine_message_navmap_cell_exit5_x, d_navmap_cells->exit5_x, sizeof(int)*xmachine_message_navmap_cell_MAX));
	h_tex_xmachine_message_navmap_cell_exit5_x_offset = (int)tex_xmachine_message_navmap_cell_exit5_x_byte_offset / sizeof(float);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_navmap_cell_exit5_x_offset), &h_tex_xmachine_message_navmap_cell_exit5_x_offset, sizeof(int)));
	size_t tex_xmachine_message_navmap_cell_exit5_y_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_navmap_cell_exit5_y_byte_offset, tex_xmachine_message_navmap_cell_exit5_y, d_navmap_cells->exit5_y, sizeof(int)*xmachine_message_navmap_cell_MAX));
	h_tex_xmachine_message_navmap_cell_exit5_y_offset = (int)tex_xmachine_message_navmap_cell_exit5_y_byte_offset / sizeof(float);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_navmap_cell_exit5_y_offset), &h_tex_xmachine_message_navmap_cell_exit5_y_offset, sizeof(int)));
	size_t tex_xmachine_message_navmap_cell_exit6_x_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_navmap_cell_exit6_x_byte_offset, tex_xmachine_message_navmap_cell_exit6_x, d_navmap_cells->exit6_x, sizeof(int)*xmachine_message_navmap_cell_MAX));
	h_tex_xmachine_message_navmap_cell_exit6_x_offset = (int)tex_xmachine_message_navmap_cell_exit6_x_byte_offset / sizeof(float);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_navmap_cell_exit6_x_offset), &h_tex_xmachine_message_navmap_cell_exit6_x_offset, sizeof(int)));
	size_t tex_xmachine_message_navmap_cell_exit6_y_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_navmap_cell_exit6_y_byte_offset, tex_xmachine_message_navmap_cell_exit6_y, d_navmap_cells->exit6_y, sizeof(int)*xmachine_message_navmap_cell_MAX));
	h_tex_xmachine_message_navmap_cell_exit6_y_offset = (int)tex_xmachine_message_navmap_cell_exit6_y_byte_offset / sizeof(float);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_navmap_cell_exit6_y_offset), &h_tex_xmachine_message_navmap_cell_exit6_y_offset, sizeof(int)));
	
	//IF CONTINUOUS AGENT CAN REALLOCATE (process dead agents) THEN RESET AGENT SWAPS	
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, reset_agent_scan_input, no_sm, state_list_size); 
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	reset_agent_scan_input<<<gridSize, blockSize, 0, stream>>>(d_agents);
	gpuErrchkLaunch();
	
	
	//MAIN XMACHINE FUNCTION CALL (force_flow)
	//Reallocate   : true
	//Input        : navmap_cell
	//Output       : 
	//Agent Output : 
	GPUFLAME_force_flow<<<g, b, sm_size, stream>>>(d_agents, d_navmap_cells, d_rand48);
	gpuErrchkLaunch();
	
	
	//UNBIND MESSAGE INPUT VARIABLE TEXTURES
	//continuous agent with discrete or partitioned message input uses texture caching
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_navmap_cell_x));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_navmap_cell_y));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_navmap_cell_exit_no));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_navmap_cell_height));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_navmap_cell_collision_x));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_navmap_cell_collision_y));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_navmap_cell_exit0_x));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_navmap_cell_exit0_y));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_navmap_cell_exit1_x));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_navmap_cell_exit1_y));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_navmap_cell_exit2_x));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_navmap_cell_exit2_y));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_navmap_cell_exit3_x));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_navmap_cell_exit3_y));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_navmap_cell_exit4_x));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_navmap_cell_exit4_y));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_navmap_cell_exit5_x));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_navmap_cell_exit5_y));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_navmap_cell_exit6_x));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_navmap_cell_exit6_y));
	
	//FOR CONTINUOUS AGENTS WITH REALLOCATION REMOVE POSSIBLE DEAD AGENTS	
    thrust::exclusive_scan(thrust::cuda::par.on(stream), thrust::device_pointer_cast(d_agents->_scan_input), thrust::device_pointer_cast(d_agents->_scan_input) + h_xmachine_memory_agent_count, thrust::device_pointer_cast(d_agents->_position));
	//Scatter into swap
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, scatter_agent_Agents, no_sm, state_list_size); 
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	scatter_agent_Agents<<<gridSize, blockSize, 0, stream>>>(d_agents_swap, d_agents, 0, h_xmachine_memory_agent_count);
	gpuErrchkLaunch();
	//use a temp pointer to make swap default
	xmachine_memory_agent_list* force_flow_agents_temp = d_agents;
	d_agents = d_agents_swap;
	d_agents_swap = force_flow_agents_temp;
	//reset agent count
	gpuErrchk( hipMemcpy( &scan_last_sum, &d_agents_swap->_position[h_xmachine_memory_agent_count-1], sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk( hipMemcpy( &scan_last_included, &d_agents_swap->_scan_input[h_xmachine_memory_agent_count-1], sizeof(int), hipMemcpyDeviceToHost));
	if (scan_last_included == 1)
		h_xmachine_memory_agent_count = scan_last_sum+1;
	else
		h_xmachine_memory_agent_count = scan_last_sum;
	//Copy count to device
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_agent_count), &h_xmachine_memory_agent_count, sizeof(int)));	
	
	
	//************************ MOVE AGENTS TO NEXT STATE ****************************
    
	//check the working agents wont exceed the buffer size in the new state list
	if (h_xmachine_memory_agent_default_count+h_xmachine_memory_agent_count > xmachine_memory_agent_MAX){
		printf("Error: Buffer size of force_flow agents in state default will be exceeded moving working agents to next state in function force_flow\n");
		exit(0);
	}
	//append agents to next state list
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, append_agent_Agents, no_sm, state_list_size); 
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	append_agent_Agents<<<gridSize, blockSize, 0, stream>>>(d_agents_default, d_agents, h_xmachine_memory_agent_default_count, h_xmachine_memory_agent_count);
	gpuErrchkLaunch();
	//update new state agent size
	h_xmachine_memory_agent_default_count += h_xmachine_memory_agent_count;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_agent_default_count), &h_xmachine_memory_agent_default_count, sizeof(int)));	
	
	
}



	
/* Shared memory size calculator for agent function */
int agent_move_sm_size(int blockSize){
	int sm_size;
	sm_size = SM_START;
  
	return sm_size;
}

/** agent_move
 * Agent function prototype for move function of agent agent
 */
void agent_move(hipStream_t &stream){

	int sm_size;
	int blockSize;
	int minGridSize;
	int gridSize;
	int state_list_size;
	dim3 g; //grid for agent func
	dim3 b; //block for agent func

	
	//CHECK THE CURRENT STATE LIST COUNT IS NOT EQUAL TO 0
	
	if (h_xmachine_memory_agent_default_count == 0)
	{
		return;
	}
	
	
	//SET SM size to 0 and save state list size for occupancy calculations
	sm_size = SM_START;
	state_list_size = h_xmachine_memory_agent_default_count;

	

	//******************************** AGENT FUNCTION CONDITION *********************
	//THERE IS NOT A FUNCTION CONDITION
	//currentState maps to working list
	xmachine_memory_agent_list* agents_default_temp = d_agents;
	d_agents = d_agents_default;
	d_agents_default = agents_default_temp;
	//set working count to current state count
	h_xmachine_memory_agent_count = h_xmachine_memory_agent_default_count;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_agent_count), &h_xmachine_memory_agent_count, sizeof(int)));	
	//set current state count to 0
	h_xmachine_memory_agent_default_count = 0;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_agent_default_count), &h_xmachine_memory_agent_default_count, sizeof(int)));	
	
 

	//******************************** AGENT FUNCTION *******************************

	
	
	//calculate the grid block size for main agent function
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, GPUFLAME_move, agent_move_sm_size, state_list_size);
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	b.x = blockSize;
	g.x = gridSize;
	
	sm_size = agent_move_sm_size(blockSize);
	
	
	
	
	//MAIN XMACHINE FUNCTION CALL (move)
	//Reallocate   : false
	//Input        : 
	//Output       : 
	//Agent Output : 
	GPUFLAME_move<<<g, b, sm_size, stream>>>(d_agents);
	gpuErrchkLaunch();
	
	
	
	//************************ MOVE AGENTS TO NEXT STATE ****************************
    
	//check the working agents wont exceed the buffer size in the new state list
	if (h_xmachine_memory_agent_default_count+h_xmachine_memory_agent_count > xmachine_memory_agent_MAX){
		printf("Error: Buffer size of move agents in state default will be exceeded moving working agents to next state in function move\n");
		exit(0);
	}
	//append agents to next state list
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, append_agent_Agents, no_sm, state_list_size); 
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	append_agent_Agents<<<gridSize, blockSize, 0, stream>>>(d_agents_default, d_agents, h_xmachine_memory_agent_default_count, h_xmachine_memory_agent_count);
	gpuErrchkLaunch();
	//update new state agent size
	h_xmachine_memory_agent_default_count += h_xmachine_memory_agent_count;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_agent_default_count), &h_xmachine_memory_agent_default_count, sizeof(int)));	
	
	
}



	
/* Shared memory size calculator for agent function */
int navmap_output_navmap_cells_sm_size(int blockSize){
	int sm_size;
	sm_size = SM_START;
  
	return sm_size;
}

/** navmap_output_navmap_cells
 * Agent function prototype for output_navmap_cells function of navmap agent
 */
void navmap_output_navmap_cells(hipStream_t &stream){

	int sm_size;
	int blockSize;
	int minGridSize;
	int gridSize;
	int state_list_size;
	dim3 g; //grid for agent func
	dim3 b; //block for agent func

	
	//CHECK THE CURRENT STATE LIST COUNT IS NOT EQUAL TO 0
	
	
	//SET SM size to 0 and save state list size for occupancy calculations
	sm_size = SM_START;
	state_list_size = h_xmachine_memory_navmap_static_count;

	

	//******************************** AGENT FUNCTION CONDITION *********************
	//THERE IS NOT A FUNCTION CONDITION
	//currentState maps to working list
	xmachine_memory_navmap_list* navmaps_static_temp = d_navmaps;
	d_navmaps = d_navmaps_static;
	d_navmaps_static = navmaps_static_temp;
	//set working count to current state count
	h_xmachine_memory_navmap_count = h_xmachine_memory_navmap_static_count;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_navmap_count), &h_xmachine_memory_navmap_count, sizeof(int)));	
	//set current state count to 0
	h_xmachine_memory_navmap_static_count = 0;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_navmap_static_count), &h_xmachine_memory_navmap_static_count, sizeof(int)));	
	
 

	//******************************** AGENT FUNCTION *******************************

	
	
	//calculate the grid block size for main agent function
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, GPUFLAME_output_navmap_cells, navmap_output_navmap_cells_sm_size, state_list_size);
	blockSize = closest_sqr_pow2(blockSize); //For discrete agents the block size must be a square power of 2
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	b.x = (int)sqrt(blockSize);
	b.y = b.x;
	g.x = (int)sqrt(gridSize);
	g.y = g.x;
	sm_size = navmap_output_navmap_cells_sm_size(blockSize);
	
	
	
	//SET THE OUTPUT MESSAGE TYPE FOR CONTINUOUS AGENTS
	
	
	//MAIN XMACHINE FUNCTION CALL (output_navmap_cells)
	//Reallocate   : false
	//Input        : 
	//Output       : navmap_cell
	//Agent Output : 
	GPUFLAME_output_navmap_cells<<<g, b, sm_size, stream>>>(d_navmaps, d_navmap_cells);
	gpuErrchkLaunch();
	
	
	//CONTINUOUS AGENTS SCATTER NON PARTITIONED OPTIONAL OUTPUT MESSAGES
	
	//UPDATE MESSAGE COUNTS FOR CONTINUOUS AGENTS WITH NON PARTITIONED MESSAGE OUTPUT 
	
	//************************ MOVE AGENTS TO NEXT STATE ****************************
    
    //currentState maps to working list
	navmaps_static_temp = d_navmaps_static;
	d_navmaps_static = d_navmaps;
	d_navmaps = navmaps_static_temp;
    //set current state count
	h_xmachine_memory_navmap_static_count = h_xmachine_memory_navmap_count;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_navmap_static_count), &h_xmachine_memory_navmap_static_count, sizeof(int)));	
	
	
}



	
/* Shared memory size calculator for agent function */
int navmap_generate_pedestrians_sm_size(int blockSize){
	int sm_size;
	sm_size = SM_START;
  
	return sm_size;
}

/** navmap_generate_pedestrians
 * Agent function prototype for generate_pedestrians function of navmap agent
 */
void navmap_generate_pedestrians(hipStream_t &stream){

	int sm_size;
	int blockSize;
	int minGridSize;
	int gridSize;
	int state_list_size;
	dim3 g; //grid for agent func
	dim3 b; //block for agent func

	
	//CHECK THE CURRENT STATE LIST COUNT IS NOT EQUAL TO 0
	
	
	//SET SM size to 0 and save state list size for occupancy calculations
	sm_size = SM_START;
	state_list_size = h_xmachine_memory_navmap_static_count;

	
	//FOR agent AGENT OUTPUT, RESET THE AGENT NEW LIST SCAN INPUT
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, reset_agent_scan_input, no_sm, state_list_size); 
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	reset_agent_scan_input<<<gridSize, blockSize, 0, stream>>>(d_agents_new);
	gpuErrchkLaunch();
	

	//******************************** AGENT FUNCTION CONDITION *********************
	//THERE IS NOT A FUNCTION CONDITION
	//currentState maps to working list
	xmachine_memory_navmap_list* navmaps_static_temp = d_navmaps;
	d_navmaps = d_navmaps_static;
	d_navmaps_static = navmaps_static_temp;
	//set working count to current state count
	h_xmachine_memory_navmap_count = h_xmachine_memory_navmap_static_count;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_navmap_count), &h_xmachine_memory_navmap_count, sizeof(int)));	
	//set current state count to 0
	h_xmachine_memory_navmap_static_count = 0;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_navmap_static_count), &h_xmachine_memory_navmap_static_count, sizeof(int)));	
	
 

	//******************************** AGENT FUNCTION *******************************

	
	
	//calculate the grid block size for main agent function
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, GPUFLAME_generate_pedestrians, navmap_generate_pedestrians_sm_size, state_list_size);
	blockSize = closest_sqr_pow2(blockSize); //For discrete agents the block size must be a square power of 2
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	b.x = (int)sqrt(blockSize);
	b.y = b.x;
	g.x = (int)sqrt(gridSize);
	g.y = g.x;
	sm_size = navmap_generate_pedestrians_sm_size(blockSize);
	
	
	
	
	//MAIN XMACHINE FUNCTION CALL (generate_pedestrians)
	//Reallocate   : false
	//Input        : 
	//Output       : 
	//Agent Output : agent
	GPUFLAME_generate_pedestrians<<<g, b, sm_size, stream>>>(d_navmaps, d_agents_new, d_rand48);
	gpuErrchkLaunch();
	
	
    //COPY ANY AGENT COUNT BEFORE navmap AGENTS ARE KILLED (needed for scatter)
	int navmaps_pre_death_count = h_xmachine_memory_navmap_count;
	
	//FOR agent AGENT OUTPUT SCATTER AGENTS 
    thrust::exclusive_scan(thrust::cuda::par.on(stream), thrust::device_pointer_cast(d_agents_new->_scan_input), thrust::device_pointer_cast(d_agents_new->_scan_input) + navmaps_pre_death_count, thrust::device_pointer_cast(d_agents_new->_position));
	//reset agent count
	int agent_after_birth_count;
	gpuErrchk( hipMemcpy( &scan_last_sum, &d_agents_new->_position[navmaps_pre_death_count-1], sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk( hipMemcpy( &scan_last_included, &d_agents_new->_scan_input[navmaps_pre_death_count-1], sizeof(int), hipMemcpyDeviceToHost));
	if (scan_last_included == 1)
		agent_after_birth_count = h_xmachine_memory_agent_default_count + scan_last_sum+1;
	else
		agent_after_birth_count = h_xmachine_memory_agent_default_count + scan_last_sum;
	//check buffer is not exceeded
	if (agent_after_birth_count > xmachine_memory_agent_MAX){
		printf("Error: Buffer size of agent agents in state default will be exceeded writing new agents in function generate_pedestrians\n");
		exit(0);
	}
	//Scatter into swap
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, scatter_agent_Agents, no_sm, state_list_size); 
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	scatter_agent_Agents<<<gridSize, blockSize, 0, stream>>>(d_agents_default, d_agents_new, h_xmachine_memory_agent_default_count, navmaps_pre_death_count);
	gpuErrchkLaunch();
	//Copy count to device
	h_xmachine_memory_agent_default_count = agent_after_birth_count;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_agent_default_count), &h_xmachine_memory_agent_default_count, sizeof(int)));	
	
	
	//************************ MOVE AGENTS TO NEXT STATE ****************************
    
    //currentState maps to working list
	navmaps_static_temp = d_navmaps_static;
	d_navmaps_static = d_navmaps;
	d_navmaps = navmaps_static_temp;
    //set current state count
	h_xmachine_memory_navmap_static_count = h_xmachine_memory_navmap_count;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_navmap_static_count), &h_xmachine_memory_navmap_static_count, sizeof(int)));	
	
	
}


 
extern "C" void reset_agent_default_count()
{
    h_xmachine_memory_agent_default_count = 0;
}
 
extern "C" void reset_navmap_static_count()
{
    h_xmachine_memory_navmap_static_count = 0;
}
