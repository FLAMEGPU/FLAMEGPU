#include "hip/hip_runtime.h"


/*
* FLAME GPU v 1.4.0 for CUDA 6
* Copyright 2015 University of Sheffield.
* Author: Dr Paul Richmond
* Contact: p.richmond@sheffield.ac.uk (http://www.paulrichmond.staff.shef.ac.uk)
*
* University of Sheffield retain all intellectual property and
* proprietary rights in and to this software and related documentation.
* Any use, reproduction, disclosure, or distribution of this software
* and related documentation without an express license agreement from
* University of Sheffield is strictly prohibited.
*
* For terms of licence agreement please attached licence or view licence
* on www.flamegpu.com website.
*
*/

#ifndef _FLAMEGPU_KERNELS_H_
#define _FLAMEGPU_KERNELS_H_

#include "header.h"


/* Agent count constants */

__constant__ int d_xmachine_memory_agent_count;

__constant__ int d_xmachine_memory_navmap_count;

/* Agent state count constants */

__constant__ int d_xmachine_memory_agent_default_count;

__constant__ int d_xmachine_memory_navmap_static_count;


/* Message constants */

/* pedestrian_location Message variables */
/* Non partitioned and spatial partitioned message variables  */
__constant__ int d_message_pedestrian_location_count;         /**< message list counter*/
__constant__ int d_message_pedestrian_location_output_type;   /**< message output type (single or optional)*/
//Spatial Partitioning Variables
__constant__ float3 d_message_pedestrian_location_min_bounds;           /**< min bounds (x,y,z) of partitioning environment */
__constant__ float3 d_message_pedestrian_location_max_bounds;           /**< max bounds (x,y,z) of partitioning environment */
__constant__ int3 d_message_pedestrian_location_partitionDim;           /**< partition dimensions (x,y,z) of partitioning environment */
__constant__ float d_message_pedestrian_location_radius;                 /**< partition radius (used to determin the size of the partitions) */

/* navmap_cell Message variables */
//Discrete Partitioning Variables
__constant__ int d_message_navmap_cell_range;     /**< range of the discrete message*/
__constant__ int d_message_navmap_cell_width;     /**< with of the message grid*/

	
    
//include each function file

#include "functions.c"
    
/* Texture bindings */
/* pedestrian_location Message Bindings */texture<float, 1, hipReadModeElementType> tex_xmachine_message_pedestrian_location_x;
__constant__ int d_tex_xmachine_message_pedestrian_location_x_offset;texture<float, 1, hipReadModeElementType> tex_xmachine_message_pedestrian_location_y;
__constant__ int d_tex_xmachine_message_pedestrian_location_y_offset;texture<float, 1, hipReadModeElementType> tex_xmachine_message_pedestrian_location_z;
__constant__ int d_tex_xmachine_message_pedestrian_location_z_offset;
texture<int, 1, hipReadModeElementType> tex_xmachine_message_pedestrian_location_pbm_start;
__constant__ int d_tex_xmachine_message_pedestrian_location_pbm_start_offset;
texture<int, 1, hipReadModeElementType> tex_xmachine_message_pedestrian_location_pbm_end_or_count;
__constant__ int d_tex_xmachine_message_pedestrian_location_pbm_end_or_count_offset;


/* navmap_cell Message Bindings */texture<int, 1, hipReadModeElementType> tex_xmachine_message_navmap_cell_x;
__constant__ int d_tex_xmachine_message_navmap_cell_x_offset;texture<int, 1, hipReadModeElementType> tex_xmachine_message_navmap_cell_y;
__constant__ int d_tex_xmachine_message_navmap_cell_y_offset;texture<int, 1, hipReadModeElementType> tex_xmachine_message_navmap_cell_exit_no;
__constant__ int d_tex_xmachine_message_navmap_cell_exit_no_offset;texture<float, 1, hipReadModeElementType> tex_xmachine_message_navmap_cell_height;
__constant__ int d_tex_xmachine_message_navmap_cell_height_offset;texture<float, 1, hipReadModeElementType> tex_xmachine_message_navmap_cell_collision_x;
__constant__ int d_tex_xmachine_message_navmap_cell_collision_x_offset;texture<float, 1, hipReadModeElementType> tex_xmachine_message_navmap_cell_collision_y;
__constant__ int d_tex_xmachine_message_navmap_cell_collision_y_offset;texture<float, 1, hipReadModeElementType> tex_xmachine_message_navmap_cell_exit0_x;
__constant__ int d_tex_xmachine_message_navmap_cell_exit0_x_offset;texture<float, 1, hipReadModeElementType> tex_xmachine_message_navmap_cell_exit0_y;
__constant__ int d_tex_xmachine_message_navmap_cell_exit0_y_offset;texture<float, 1, hipReadModeElementType> tex_xmachine_message_navmap_cell_exit1_x;
__constant__ int d_tex_xmachine_message_navmap_cell_exit1_x_offset;texture<float, 1, hipReadModeElementType> tex_xmachine_message_navmap_cell_exit1_y;
__constant__ int d_tex_xmachine_message_navmap_cell_exit1_y_offset;texture<float, 1, hipReadModeElementType> tex_xmachine_message_navmap_cell_exit2_x;
__constant__ int d_tex_xmachine_message_navmap_cell_exit2_x_offset;texture<float, 1, hipReadModeElementType> tex_xmachine_message_navmap_cell_exit2_y;
__constant__ int d_tex_xmachine_message_navmap_cell_exit2_y_offset;texture<float, 1, hipReadModeElementType> tex_xmachine_message_navmap_cell_exit3_x;
__constant__ int d_tex_xmachine_message_navmap_cell_exit3_x_offset;texture<float, 1, hipReadModeElementType> tex_xmachine_message_navmap_cell_exit3_y;
__constant__ int d_tex_xmachine_message_navmap_cell_exit3_y_offset;texture<float, 1, hipReadModeElementType> tex_xmachine_message_navmap_cell_exit4_x;
__constant__ int d_tex_xmachine_message_navmap_cell_exit4_x_offset;texture<float, 1, hipReadModeElementType> tex_xmachine_message_navmap_cell_exit4_y;
__constant__ int d_tex_xmachine_message_navmap_cell_exit4_y_offset;texture<float, 1, hipReadModeElementType> tex_xmachine_message_navmap_cell_exit5_x;
__constant__ int d_tex_xmachine_message_navmap_cell_exit5_x_offset;texture<float, 1, hipReadModeElementType> tex_xmachine_message_navmap_cell_exit5_y;
__constant__ int d_tex_xmachine_message_navmap_cell_exit5_y_offset;texture<float, 1, hipReadModeElementType> tex_xmachine_message_navmap_cell_exit6_x;
__constant__ int d_tex_xmachine_message_navmap_cell_exit6_x_offset;texture<float, 1, hipReadModeElementType> tex_xmachine_message_navmap_cell_exit6_y;
__constant__ int d_tex_xmachine_message_navmap_cell_exit6_y_offset;

    
#define WRAP(x,m) (((x)<m)?(x):(x%m)) /**< Simple wrap */
#define sWRAP(x,m) (((x)<m)?(((x)<0)?(m+(x)):(x)):(m-(x))) /**<signed integer wrap (no modulus) for negatives where 2m > |x| > m */

//PADDING WILL ONLY AVOID SM CONFLICTS FOR 32BIT
//SM_OFFSET REQUIRED AS FERMI STARTS INDEXING MEMORY FROM LOCATION 0 (i.e. NULL)??
__constant__ int d_SM_START;
__constant__ int d_PADDING;

//SM addressing macro to avoid conflicts (32 bit only)
#define SHARE_INDEX(i, s) (((s + d_PADDING)* i)+d_SM_START) /**<offset struct size by padding to avoid bank conflicts */

//if doubel support is needed then define the following function which requires sm_13 or later
#ifdef _DOUBLE_SUPPORT_REQUIRED_
__inline__ __device__ double tex1DfetchDouble(texture<int2, 1, hipReadModeElementType> tex, int i)
{
	int2 v = tex1Dfetch(tex, i);
  //IF YOU HAVE AN ERROR HERE THEN YOU ARE USING DOUBLE VALUES IN AGENT MEMORY AND NOT COMPILING FOR DOUBLE SUPPORTED HARDWARE
  //To compile for double supported hardware change the CUDA Build rule property "Use sm_13 Architecture (double support)" on the CUDA-Specific Propert Page of the CUDA Build Rule for simulation.cu
	return __hiloint2double(v.y, v.x);
}
#endif

/* Helper functions */
/** next_cell
 * Function used for finding the next cell when using spatial partitioning
 * Upddates the relative cell variable which can have value of -1, 0 or +1
 * @param relative_cell pointer to the relative cell position
 * @return boolean if there is a next cell. True unless relative_Cell value was 1,1,1
 */
__device__ int next_cell3D(int3* relative_cell)
{
	if (relative_cell->x < 1)
	{
		relative_cell->x++;
		return true;
	}
	relative_cell->x = -1;

	if (relative_cell->y < 1)
	{
		relative_cell->y++;
		return true;
	}
	relative_cell->y = -1;
	
	if (relative_cell->z < 1)
	{
		relative_cell->z++;
		return true;
	}
	relative_cell->z = -1;
	
	return false;
}

/** next_cell2D
 * Function used for finding the next cell when using spatial partitioning. Z component is ignored
 * Upddates the relative cell variable which can have value of -1, 0 or +1
 * @param relative_cell pointer to the relative cell position
 * @return boolean if there is a next cell. True unless relative_Cell value was 1,1
 */
__device__ int next_cell2D(int3* relative_cell)
{
	if (relative_cell->x < 1)
	{
		relative_cell->x++;
		return true;
	}
	relative_cell->x = -1;

	if (relative_cell->y < 1)
	{
		relative_cell->y++;
		return true;
	}
	relative_cell->y = -1;
	
	return false;
}


////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dyanamically created agent agent functions */

/** reset_agent_scan_input
 * agent agent reset scan input function
 * @param agents The xmachine_memory_agent_list agent list
 */
__global__ void reset_agent_scan_input(xmachine_memory_agent_list* agents){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	agents->_position[index] = 0;
	agents->_scan_input[index] = 0;
}



/** scatter_agent_Agents
 * agent scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_agent_list agent list destination
 * @param agents_src xmachine_memory_agent_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void scatter_agent_Agents(xmachine_memory_agent_list* agents_dst, xmachine_memory_agent_list* agents_src, int dst_agent_count, int number_to_scatter){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = agents_src->_scan_input[index];

	//if optional message is to be written. 
	//must check agent is within number to scatter as unused threads may have scan input = 1
	if ((_scan_input == 1)&&(index < number_to_scatter)){
		int output_index = agents_src->_position[index] + dst_agent_count;

		//AoS - xmachine_message_location Un-Coalesced scattered memory write     
        agents_dst->_position[output_index] = output_index;        
		agents_dst->x[output_index] = agents_src->x[index];        
		agents_dst->y[output_index] = agents_src->y[index];        
		agents_dst->velx[output_index] = agents_src->velx[index];        
		agents_dst->vely[output_index] = agents_src->vely[index];        
		agents_dst->steer_x[output_index] = agents_src->steer_x[index];        
		agents_dst->steer_y[output_index] = agents_src->steer_y[index];        
		agents_dst->height[output_index] = agents_src->height[index];        
		agents_dst->exit_no[output_index] = agents_src->exit_no[index];        
		agents_dst->speed[output_index] = agents_src->speed[index];        
		agents_dst->lod[output_index] = agents_src->lod[index];        
		agents_dst->animate[output_index] = agents_src->animate[index];        
		agents_dst->animate_dir[output_index] = agents_src->animate_dir[index];
	}
}

/** append_agent_Agents
 * agent scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_agent_list agent list destination
 * @param agents_src xmachine_memory_agent_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void append_agent_Agents(xmachine_memory_agent_list* agents_dst, xmachine_memory_agent_list* agents_src, int dst_agent_count, int number_to_append){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	//must check agent is within number to append as unused threads may have scan input = 1
    if (index < number_to_append){
	    int output_index = index + dst_agent_count;

	    //AoS - xmachine_message_location Un-Coalesced scattered memory write
	    agents_dst->_position[output_index] = output_index;
	    agents_dst->x[output_index] = agents_src->x[index];
	    agents_dst->y[output_index] = agents_src->y[index];
	    agents_dst->velx[output_index] = agents_src->velx[index];
	    agents_dst->vely[output_index] = agents_src->vely[index];
	    agents_dst->steer_x[output_index] = agents_src->steer_x[index];
	    agents_dst->steer_y[output_index] = agents_src->steer_y[index];
	    agents_dst->height[output_index] = agents_src->height[index];
	    agents_dst->exit_no[output_index] = agents_src->exit_no[index];
	    agents_dst->speed[output_index] = agents_src->speed[index];
	    agents_dst->lod[output_index] = agents_src->lod[index];
	    agents_dst->animate[output_index] = agents_src->animate[index];
	    agents_dst->animate_dir[output_index] = agents_src->animate_dir[index];
    }
}

/** add_agent_agent
 * Continuous agent agent add agent function writes agent data to agent swap
 * @param agents xmachine_memory_agent_list to add agents to 
 * @param x agent variable of type float
 * @param y agent variable of type float
 * @param velx agent variable of type float
 * @param vely agent variable of type float
 * @param steer_x agent variable of type float
 * @param steer_y agent variable of type float
 * @param height agent variable of type float
 * @param exit_no agent variable of type int
 * @param speed agent variable of type float
 * @param lod agent variable of type int
 * @param animate agent variable of type float
 * @param animate_dir agent variable of type int
 */
template <int AGENT_TYPE>
__device__ void add_agent_agent(xmachine_memory_agent_list* agents, float x, float y, float velx, float vely, float steer_x, float steer_y, float height, int exit_no, float speed, int lod, float animate, int animate_dir){
	
	int index;
    
    //calculate the agents index in global agent list (depends on agent type)
	if (AGENT_TYPE == DISCRETE_2D){
		int width = (blockDim.x* gridDim.x);
		int2 global_position;
		global_position.x = (blockIdx.x*blockDim.x) + threadIdx.x;
		global_position.y = (blockIdx.y*blockDim.y) + threadIdx.y;
		index = global_position.x + (global_position.y* width);
	}else//AGENT_TYPE == CONTINOUS
		index = threadIdx.x + blockIdx.x*blockDim.x;

	//for prefix sum
	agents->_position[index] = 0;
	agents->_scan_input[index] = 1;

	//write data to new buffer
	agents->x[index] = x;
	agents->y[index] = y;
	agents->velx[index] = velx;
	agents->vely[index] = vely;
	agents->steer_x[index] = steer_x;
	agents->steer_y[index] = steer_y;
	agents->height[index] = height;
	agents->exit_no[index] = exit_no;
	agents->speed[index] = speed;
	agents->lod[index] = lod;
	agents->animate[index] = animate;
	agents->animate_dir[index] = animate_dir;

}

//non templated version assumes DISCRETE_2D but works also for CONTINUOUS
__device__ void add_agent_agent(xmachine_memory_agent_list* agents, float x, float y, float velx, float vely, float steer_x, float steer_y, float height, int exit_no, float speed, int lod, float animate, int animate_dir){
    add_agent_agent<DISCRETE_2D>(agents, x, y, velx, vely, steer_x, steer_y, height, exit_no, speed, lod, animate, animate_dir);
}

/** reorder_agent_agents
 * Continuous agent agent areorder function used after key value pairs have been sorted
 * @param values sorted index values
 * @param unordered_agents list of unordered agents
 * @ param ordered_agents list used to output ordered agents
 */
__global__ void reorder_agent_agents(unsigned int* values, xmachine_memory_agent_list* unordered_agents, xmachine_memory_agent_list* ordered_agents)
{
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	uint old_pos = values[index];

	//reorder agent data
	ordered_agents->x[index] = unordered_agents->x[old_pos];
	ordered_agents->y[index] = unordered_agents->y[old_pos];
	ordered_agents->velx[index] = unordered_agents->velx[old_pos];
	ordered_agents->vely[index] = unordered_agents->vely[old_pos];
	ordered_agents->steer_x[index] = unordered_agents->steer_x[old_pos];
	ordered_agents->steer_y[index] = unordered_agents->steer_y[old_pos];
	ordered_agents->height[index] = unordered_agents->height[old_pos];
	ordered_agents->exit_no[index] = unordered_agents->exit_no[old_pos];
	ordered_agents->speed[index] = unordered_agents->speed[old_pos];
	ordered_agents->lod[index] = unordered_agents->lod[old_pos];
	ordered_agents->animate[index] = unordered_agents->animate[old_pos];
	ordered_agents->animate_dir[index] = unordered_agents->animate_dir[old_pos];
}

////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dyanamically created navmap agent functions */

/** reset_navmap_scan_input
 * navmap agent reset scan input function
 * @param agents The xmachine_memory_navmap_list agent list
 */
__global__ void reset_navmap_scan_input(xmachine_memory_navmap_list* agents){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	agents->_position[index] = 0;
	agents->_scan_input[index] = 0;
}


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dyanamically created pedestrian_location message functions */


/** add_pedestrian_location_message
 * Add non partitioned or spatially partitioned pedestrian_location message
 * @param messages xmachine_message_pedestrian_location_list message list to add too
 * @param x agent variable of type float
 * @param y agent variable of type float
 * @param z agent variable of type float
 */
__device__ void add_pedestrian_location_message(xmachine_message_pedestrian_location_list* messages, float x, float y, float z){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x + d_message_pedestrian_location_count;

	int _position;
	int _scan_input;

	//decide output position
	if(d_message_pedestrian_location_output_type == single_message){
		_position = index; //same as agent position
		_scan_input = 0;
	}else if (d_message_pedestrian_location_output_type == optional_message){
		_position = 0;	   //to be calculated using Prefix sum
		_scan_input = 1;
	}

	//AoS - xmachine_message_pedestrian_location Coalesced memory write
	messages->_scan_input[index] = _scan_input;	
	messages->_position[index] = _position;
	messages->x[index] = x;
	messages->y[index] = y;
	messages->z[index] = z;

}

/**
 * Scatter non partitioned or spatially partitioned pedestrian_location message (for optional messages)
 * @param messages scatter_optional_pedestrian_location_messages Sparse xmachine_message_pedestrian_location_list message list
 * @param message_swap temp xmachine_message_pedestrian_location_list message list to scatter sparse messages to
 */
__global__ void scatter_optional_pedestrian_location_messages(xmachine_message_pedestrian_location_list* messages, xmachine_message_pedestrian_location_list* messages_swap){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = messages_swap->_scan_input[index];

	//if optional message is to be written
	if (_scan_input == 1){
		int output_index = messages_swap->_position[index] + d_message_pedestrian_location_count;

		//AoS - xmachine_message_pedestrian_location Un-Coalesced scattered memory write
		messages->_position[output_index] = output_index;
		messages->x[output_index] = messages_swap->x[index];
		messages->y[output_index] = messages_swap->y[index];
		messages->z[output_index] = messages_swap->z[index];				
	}
}

/** reset_pedestrian_location_swaps
 * Reset non partitioned or spatially partitioned pedestrian_location message swaps (for scattering optional messages)
 * @param message_swap message list to reset _position and _scan_input values back to 0
 */
__global__ void reset_pedestrian_location_swaps(xmachine_message_pedestrian_location_list* messages_swap){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	messages_swap->_position[index] = 0;
	messages_swap->_scan_input[index] = 0;
}

/* Message functions */

/** message_pedestrian_location_grid_position
 * Calculates the grid cell position given an float3 vector
 * @param position float3 vector representing a position
 */
__device__ int3 message_pedestrian_location_grid_position(float3 position)
{
    int3 gridPos;
    gridPos.x = floor((position.x - d_message_pedestrian_location_min_bounds.x) * (float)d_message_pedestrian_location_partitionDim.x / (d_message_pedestrian_location_max_bounds.x - d_message_pedestrian_location_min_bounds.x));
    gridPos.y = floor((position.y - d_message_pedestrian_location_min_bounds.y) * (float)d_message_pedestrian_location_partitionDim.y / (d_message_pedestrian_location_max_bounds.y - d_message_pedestrian_location_min_bounds.y));
    gridPos.z = floor((position.z - d_message_pedestrian_location_min_bounds.z) * (float)d_message_pedestrian_location_partitionDim.z / (d_message_pedestrian_location_max_bounds.z - d_message_pedestrian_location_min_bounds.z));

	//do wrapping or bounding
	

    return gridPos;
}

/** message_pedestrian_location_hash
 * Given the grid position in partition space this function calculates a hash value
 * @param gridPos The position in partition space
 */
__device__ unsigned int message_pedestrian_location_hash(int3 gridPos)
{
	//cheap bounding without mod (within range +- partition dimension)
	gridPos.x = (gridPos.x<0)? d_message_pedestrian_location_partitionDim.x-1: gridPos.x; 
	gridPos.x = (gridPos.x>=d_message_pedestrian_location_partitionDim.x)? 0 : gridPos.x; 
	gridPos.y = (gridPos.y<0)? d_message_pedestrian_location_partitionDim.y-1 : gridPos.y; 
	gridPos.y = (gridPos.y>=d_message_pedestrian_location_partitionDim.y)? 0 : gridPos.y; 
	gridPos.z = (gridPos.z<0)? d_message_pedestrian_location_partitionDim.z-1: gridPos.z; 
	gridPos.z = (gridPos.z>=d_message_pedestrian_location_partitionDim.z)? 0 : gridPos.z; 

	//unique id
	return ((gridPos.z * d_message_pedestrian_location_partitionDim.y) * d_message_pedestrian_location_partitionDim.x) + (gridPos.y * d_message_pedestrian_location_partitionDim.x) + gridPos.x;
}

#ifdef FAST_ATOMIC_SORTING
	/** hist_pedestrian_location_messages
		 * Kernal function for performing a histogram (count) on each partition bin and saving the hash and index of a message within that bin
		 * @param local_bin_index output index of the message within the calculated bin
		 * @param unsorted_index output bin index (hash) value
		 * @param messages the message list used to generate the hash value outputs
		 */
	__global__ void hist_pedestrian_location_messages(uint* local_bin_index, uint* unsorted_index, int* global_bin_count, xmachine_message_pedestrian_location_list* messages)
	{
		unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;

		float3 position = make_float3(messages->x[index], messages->y[index], messages->z[index]);
		int3 grid_position = message_pedestrian_location_grid_position(position);
		unsigned int hash = message_pedestrian_location_hash(grid_position);
		unsigned int bin_idx = atomicInc((unsigned int*) &global_bin_count[hash], 0xFFFFFFFF);
		local_bin_index[index] = bin_idx;
		unsorted_index[index] = hash;
	}
	
	/** reorder_pedestrian_location_messages
	 * Reorders the messages accoring to the partition boundary matrix start indices of each bin
	 * @param local_bin_index index of the message within the desired bin
	 * @param unsorted_index bin index (hash) value
	 * @param pbm_start_index the start indices of the partition boundary matrix
	 * @param unordered_messages the original unordered message data
	 * @param ordered_messages buffer used to scatter messages into the correct order
	 */
	 __global__ void reorder_pedestrian_location_messages(uint* local_bin_index, uint* unsorted_index, int* pbm_start_index, xmachine_message_pedestrian_location_list* unordered_messages, xmachine_message_pedestrian_location_list* ordered_messages)
	{
		int index = (blockIdx.x *blockDim.x) + threadIdx.x;

		uint i = unsorted_index[index];
		unsigned int sorted_index = local_bin_index[index] + pbm_start_index[i];

		//finally reorder agent data
		ordered_messages->x[sorted_index] = unordered_messages->x[index];
		ordered_messages->y[sorted_index] = unordered_messages->y[index];
		ordered_messages->z[sorted_index] = unordered_messages->z[index];
	}
	 
#else

	/** hash_pedestrian_location_messages
	 * Kernal function for calculating a hash value for each messahe depending on its position
	 * @param keys output for the hash key
	 * @param values output for the index value
	 * @param messages the message list used to generate the hash value outputs
	 */
	__global__ void hash_pedestrian_location_messages(uint* keys, uint* values, xmachine_message_pedestrian_location_list* messages)
	{
		unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;

		float3 position = make_float3(messages->x[index], messages->y[index], messages->z[index]);
		int3 grid_position = message_pedestrian_location_grid_position(position);
		unsigned int hash = message_pedestrian_location_hash(grid_position);

		keys[index] = hash;
		values[index] = index;
	}

	/** reorder_pedestrian_location_messages
	 * Reorders the messages accoring to the ordered sort identifiers and builds a Partition Boundary Matrix by looking at the previosu threads sort id.
	 * @param keys the sorted hash keys
	 * @param values the sorted index values
	 * @param matrix the PBM
	 * @param unordered_messages the original unordered message data
	 * @param ordered_messages buffer used to scatter messages into the correct order
	 */
	__global__ void reorder_pedestrian_location_messages(uint* keys, uint* values, xmachine_message_pedestrian_location_PBM* matrix, xmachine_message_pedestrian_location_list* unordered_messages, xmachine_message_pedestrian_location_list* ordered_messages)
	{
		extern __shared__ int sm_data [];

		int index = (blockIdx.x * blockDim.x) + threadIdx.x;

		//load threads sort key into sm
		uint key = keys[index];
		uint old_pos = values[index];

		sm_data[threadIdx.x] = key;
		__syncthreads();
	
		unsigned int prev_key;

		//if first thread then no prev sm value so get prev from global memory 
		if (threadIdx.x == 0)
		{
			//first thread has no prev value so ignore
			if (index != 0)
				prev_key = keys[index-1];
		}
		//get previous ident from sm
		else	
		{
			prev_key = sm_data[threadIdx.x-1];
		}

		//TODO: Check key is not out of bounds

		//set partition boundaries
		if (index < d_message_pedestrian_location_count)
		{
			//if first thread then set first partition cell start
			if (index == 0)
			{
				matrix->start[key] = index;
			}

			//if edge of a boundr update start and end of partition
			else if (prev_key != key)
			{
				//set start for key
				matrix->start[key] = index;

				//set end for key -1
				matrix->end_or_count[prev_key] = index;
			}

			//if last thread then set final partition cell end
			if (index == d_message_pedestrian_location_count-1)
			{
				matrix->end_or_count[key] = index+1;
			}
		}
	
		//finally reorder agent data
		ordered_messages->x[index] = unordered_messages->x[old_pos];
		ordered_messages->y[index] = unordered_messages->y[old_pos];
		ordered_messages->z[index] = unordered_messages->z[old_pos];
	}

#endif

/** load_next_pedestrian_location_message
 * Used to load the next message data to shared memory
 * Idea is check the current cell index to see if we can simpley get a message from the current cell
 * If we are at the end of the current cell then loop till we find the next cell with messages (this way we ignore cells with no messages)
 * @param messages the message list
 * @param partition_matrix the PBM
 * @param relative_cell the relative partition cell position from the agent position
 * @param cell_index_max the maximum index of the currnt partition cell
 * @param agent_grid_cell the agents partition cell position
 * @param cell_index the current cell index in agent_grid_cell+relative_cell
 * @return true if a messag has been loaded into sm false otherwise
 */
__device__ int load_next_pedestrian_location_message(xmachine_message_pedestrian_location_list* messages, xmachine_message_pedestrian_location_PBM* partition_matrix, int3 relative_cell, int cell_index_max, int3 agent_grid_cell, int cell_index)
{
	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];

	int move_cell = true;
	cell_index ++;

	//see if we need to move to a new partition cell
	if(cell_index < cell_index_max)
		move_cell = false;

	while(move_cell)
	{
		//get the next relative grid position 
        if (next_cell2D(&relative_cell))
		{
			//calculate the next cells grid position and hash
			int3 next_cell_position = agent_grid_cell + relative_cell;
			int next_cell_hash = message_pedestrian_location_hash(next_cell_position);
			//use the hash to calculate the start index
			int cell_index_min = tex1Dfetch(tex_xmachine_message_pedestrian_location_pbm_start, next_cell_hash + d_tex_xmachine_message_pedestrian_location_pbm_start_offset);

			//check for messages in the cell (empty cells with have a start index of baadf00d
			if (cell_index_min != 0xffffffff)
			{
				//if there are messages in the cell then update the cell index max value
				cell_index_max = tex1Dfetch(tex_xmachine_message_pedestrian_location_pbm_end_or_count, next_cell_hash + d_tex_xmachine_message_pedestrian_location_pbm_end_or_count_offset);
#ifdef FAST_ATOMIC_SORTING
				cell_index_max += cell_index_min; //when using fast atomics value represents bin count not last index!
#endif
				//start from the cell index min
				cell_index = cell_index_min;
				//exit the loop as we have found a valid cell with message data
				move_cell = false;
			}
		}
		else
		{
			//we have exhausted all the neightbouring cells so there are no more messages
			return false;
		}
	}
	
	//get message data using texture fetch
	xmachine_message_pedestrian_location temp_message;
	temp_message._relative_cell = relative_cell;
	temp_message._cell_index_max = cell_index_max;
	temp_message._cell_index = cell_index;
	temp_message._agent_grid_cell = agent_grid_cell;

	//Using texture cache
  temp_message.x = tex1Dfetch(tex_xmachine_message_pedestrian_location_x, cell_index + d_tex_xmachine_message_pedestrian_location_x_offset); temp_message.y = tex1Dfetch(tex_xmachine_message_pedestrian_location_y, cell_index + d_tex_xmachine_message_pedestrian_location_y_offset); temp_message.z = tex1Dfetch(tex_xmachine_message_pedestrian_location_z, cell_index + d_tex_xmachine_message_pedestrian_location_z_offset); 

	//load it into shared memory (no sync as no sharing between threads)
	int message_index = SHARE_INDEX(threadIdx.x, sizeof(xmachine_message_pedestrian_location));
	xmachine_message_pedestrian_location* sm_message = ((xmachine_message_pedestrian_location*)&message_share[message_index]);
	sm_message[0] = temp_message;

	return true;
}

/*
 * get first non partitioned pedestrian_location message (first batch load into shared memory)
 */
__device__ xmachine_message_pedestrian_location* get_first_pedestrian_location_message(xmachine_message_pedestrian_location_list* messages, xmachine_message_pedestrian_location_PBM* partition_matrix, float x, float y, float z){

	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];

	int3 relative_cell = make_int3(-2, -1, -1);
	int cell_index_max = 0;
	int cell_index = 0;
	float3 position = make_float3(x, y, z);
	int3 agent_grid_cell = message_pedestrian_location_grid_position(position);
	
	if (load_next_pedestrian_location_message(messages, partition_matrix, relative_cell, cell_index_max, agent_grid_cell, cell_index))
	{
		int message_index = SHARE_INDEX(threadIdx.x, sizeof(xmachine_message_pedestrian_location));
		return ((xmachine_message_pedestrian_location*)&message_share[message_index]);
	}
	else
	{
		return false;
	}
}

/*
 * get next non partitioned pedestrian_location message (either from SM or next batch load)
 */
__device__ xmachine_message_pedestrian_location* get_next_pedestrian_location_message(xmachine_message_pedestrian_location* message, xmachine_message_pedestrian_location_list* messages, xmachine_message_pedestrian_location_PBM* partition_matrix){
	
	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];
	
	//TODO: check message count
	
	if (load_next_pedestrian_location_message(messages, partition_matrix, message->_relative_cell, message->_cell_index_max, message->_agent_grid_cell, message->_cell_index))
	{
		//get conflict free address of 
		int message_index = SHARE_INDEX(threadIdx.x, sizeof(xmachine_message_pedestrian_location));
		return ((xmachine_message_pedestrian_location*)&message_share[message_index]);
	}
	else
		return false;
	
}


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dyanamically created navmap_cell message functions */


/* Message functions */

template <int AGENT_TYPE>
__device__ void add_navmap_cell_message(xmachine_message_navmap_cell_list* messages, int x, int y, int exit_no, float height, float collision_x, float collision_y, float exit0_x, float exit0_y, float exit1_x, float exit1_y, float exit2_x, float exit2_y, float exit3_x, float exit3_y, float exit4_x, float exit4_y, float exit5_x, float exit5_y, float exit6_x, float exit6_y){
	if (AGENT_TYPE == DISCRETE_2D){
		int width = (blockDim.x * gridDim.x);
		int2 global_position;
		global_position.x = (blockIdx.x * blockDim.x) + threadIdx.x;
		global_position.y = (blockIdx.y * blockDim.y) + threadIdx.y;

		int index = global_position.x + (global_position.y * width);

		
		messages->x[index] = x;			
		messages->y[index] = y;			
		messages->exit_no[index] = exit_no;			
		messages->height[index] = height;			
		messages->collision_x[index] = collision_x;			
		messages->collision_y[index] = collision_y;			
		messages->exit0_x[index] = exit0_x;			
		messages->exit0_y[index] = exit0_y;			
		messages->exit1_x[index] = exit1_x;			
		messages->exit1_y[index] = exit1_y;			
		messages->exit2_x[index] = exit2_x;			
		messages->exit2_y[index] = exit2_y;			
		messages->exit3_x[index] = exit3_x;			
		messages->exit3_y[index] = exit3_y;			
		messages->exit4_x[index] = exit4_x;			
		messages->exit4_y[index] = exit4_y;			
		messages->exit5_x[index] = exit5_x;			
		messages->exit5_y[index] = exit5_y;			
		messages->exit6_x[index] = exit6_x;			
		messages->exit6_y[index] = exit6_y;			
	}
	//else CONTINUOUS agents can not write to discrete space
}

//Used by continuous agents this accesses messages with texture cache. agent_x and agent_y are discrete positions in the message space
__device__ xmachine_message_navmap_cell* get_first_navmap_cell_message_continuous(xmachine_message_navmap_cell_list* messages,  int agent_x, int agent_y){

	//shared memory get from offset dependant on sm usage in function
	extern __shared__ int sm_data [];

	xmachine_message_navmap_cell* message_share = (xmachine_message_navmap_cell*)&sm_data[0];
	
	int range = d_message_navmap_cell_range;
	int width = d_message_navmap_cell_width;
	
	int2 global_position;
	global_position.x = sWRAP(agent_x-range , width);
	global_position.y = sWRAP(agent_y-range , width);
	

	int index = ((global_position.y)* width) + global_position.x;
	
	xmachine_message_navmap_cell temp_message;
	temp_message._position = make_int2(agent_x, agent_y);
	temp_message._relative = make_int2(-range, -range);

	temp_message.x = tex1Dfetch(tex_xmachine_message_navmap_cell_x, index + d_tex_xmachine_message_navmap_cell_x_offset);temp_message.y = tex1Dfetch(tex_xmachine_message_navmap_cell_y, index + d_tex_xmachine_message_navmap_cell_y_offset);temp_message.exit_no = tex1Dfetch(tex_xmachine_message_navmap_cell_exit_no, index + d_tex_xmachine_message_navmap_cell_exit_no_offset);temp_message.height = tex1Dfetch(tex_xmachine_message_navmap_cell_height, index + d_tex_xmachine_message_navmap_cell_height_offset);temp_message.collision_x = tex1Dfetch(tex_xmachine_message_navmap_cell_collision_x, index + d_tex_xmachine_message_navmap_cell_collision_x_offset);temp_message.collision_y = tex1Dfetch(tex_xmachine_message_navmap_cell_collision_y, index + d_tex_xmachine_message_navmap_cell_collision_y_offset);temp_message.exit0_x = tex1Dfetch(tex_xmachine_message_navmap_cell_exit0_x, index + d_tex_xmachine_message_navmap_cell_exit0_x_offset);temp_message.exit0_y = tex1Dfetch(tex_xmachine_message_navmap_cell_exit0_y, index + d_tex_xmachine_message_navmap_cell_exit0_y_offset);temp_message.exit1_x = tex1Dfetch(tex_xmachine_message_navmap_cell_exit1_x, index + d_tex_xmachine_message_navmap_cell_exit1_x_offset);temp_message.exit1_y = tex1Dfetch(tex_xmachine_message_navmap_cell_exit1_y, index + d_tex_xmachine_message_navmap_cell_exit1_y_offset);temp_message.exit2_x = tex1Dfetch(tex_xmachine_message_navmap_cell_exit2_x, index + d_tex_xmachine_message_navmap_cell_exit2_x_offset);temp_message.exit2_y = tex1Dfetch(tex_xmachine_message_navmap_cell_exit2_y, index + d_tex_xmachine_message_navmap_cell_exit2_y_offset);temp_message.exit3_x = tex1Dfetch(tex_xmachine_message_navmap_cell_exit3_x, index + d_tex_xmachine_message_navmap_cell_exit3_x_offset);temp_message.exit3_y = tex1Dfetch(tex_xmachine_message_navmap_cell_exit3_y, index + d_tex_xmachine_message_navmap_cell_exit3_y_offset);temp_message.exit4_x = tex1Dfetch(tex_xmachine_message_navmap_cell_exit4_x, index + d_tex_xmachine_message_navmap_cell_exit4_x_offset);temp_message.exit4_y = tex1Dfetch(tex_xmachine_message_navmap_cell_exit4_y, index + d_tex_xmachine_message_navmap_cell_exit4_y_offset);temp_message.exit5_x = tex1Dfetch(tex_xmachine_message_navmap_cell_exit5_x, index + d_tex_xmachine_message_navmap_cell_exit5_x_offset);temp_message.exit5_y = tex1Dfetch(tex_xmachine_message_navmap_cell_exit5_y, index + d_tex_xmachine_message_navmap_cell_exit5_y_offset);temp_message.exit6_x = tex1Dfetch(tex_xmachine_message_navmap_cell_exit6_x, index + d_tex_xmachine_message_navmap_cell_exit6_x_offset);temp_message.exit6_y = tex1Dfetch(tex_xmachine_message_navmap_cell_exit6_y, index + d_tex_xmachine_message_navmap_cell_exit6_y_offset);
	
	message_share[threadIdx.x] = temp_message;

	//return top left of messages
	return &message_share[threadIdx.x];
}

//Get next navmap_cell message  continuous
//Used by continuous agents this accesses messages with texture cache (agent position in discrete space was set when accessing first message)
__device__ xmachine_message_navmap_cell* get_next_navmap_cell_message_continuous(xmachine_message_navmap_cell* message, xmachine_message_navmap_cell_list* messages){

	//shared memory get from offset dependant on sm usage in function
	extern __shared__ int sm_data [];

	xmachine_message_navmap_cell* message_share = (xmachine_message_navmap_cell*)&sm_data[0];
	
	int range = d_message_navmap_cell_range;
	int width = d_message_navmap_cell_width;

	//Get previous position
	int2 previous_relative = message->_relative;

	//exit if at (range, range)
	if (previous_relative.x == (range))
        if (previous_relative.y == (range))
		    return false;

	//calculate next message relative position
	int2 next_relative = previous_relative;
	next_relative.x += 1;
	if ((next_relative.x)>range){
		next_relative.x = -range;
		next_relative.y = previous_relative.y + 1;
	}

	//skip own message
	if (next_relative.x == 0)
        if (next_relative.y == 0)
		    next_relative.x += 1;

	int2 global_position;
	global_position.x =	sWRAP(message->_position.x + next_relative.x, width);
	global_position.y = sWRAP(message->_position.y + next_relative.y, width);

	int index = ((global_position.y)* width) + (global_position.x);
	
	xmachine_message_navmap_cell temp_message;
	temp_message._position = message->_position;
	temp_message._relative = next_relative;

	temp_message.x = tex1Dfetch(tex_xmachine_message_navmap_cell_x, index + d_tex_xmachine_message_navmap_cell_x_offset);	temp_message.y = tex1Dfetch(tex_xmachine_message_navmap_cell_y, index + d_tex_xmachine_message_navmap_cell_y_offset);	temp_message.exit_no = tex1Dfetch(tex_xmachine_message_navmap_cell_exit_no, index + d_tex_xmachine_message_navmap_cell_exit_no_offset);	temp_message.height = tex1Dfetch(tex_xmachine_message_navmap_cell_height, index + d_tex_xmachine_message_navmap_cell_height_offset);	temp_message.collision_x = tex1Dfetch(tex_xmachine_message_navmap_cell_collision_x, index + d_tex_xmachine_message_navmap_cell_collision_x_offset);	temp_message.collision_y = tex1Dfetch(tex_xmachine_message_navmap_cell_collision_y, index + d_tex_xmachine_message_navmap_cell_collision_y_offset);	temp_message.exit0_x = tex1Dfetch(tex_xmachine_message_navmap_cell_exit0_x, index + d_tex_xmachine_message_navmap_cell_exit0_x_offset);	temp_message.exit0_y = tex1Dfetch(tex_xmachine_message_navmap_cell_exit0_y, index + d_tex_xmachine_message_navmap_cell_exit0_y_offset);	temp_message.exit1_x = tex1Dfetch(tex_xmachine_message_navmap_cell_exit1_x, index + d_tex_xmachine_message_navmap_cell_exit1_x_offset);	temp_message.exit1_y = tex1Dfetch(tex_xmachine_message_navmap_cell_exit1_y, index + d_tex_xmachine_message_navmap_cell_exit1_y_offset);	temp_message.exit2_x = tex1Dfetch(tex_xmachine_message_navmap_cell_exit2_x, index + d_tex_xmachine_message_navmap_cell_exit2_x_offset);	temp_message.exit2_y = tex1Dfetch(tex_xmachine_message_navmap_cell_exit2_y, index + d_tex_xmachine_message_navmap_cell_exit2_y_offset);	temp_message.exit3_x = tex1Dfetch(tex_xmachine_message_navmap_cell_exit3_x, index + d_tex_xmachine_message_navmap_cell_exit3_x_offset);	temp_message.exit3_y = tex1Dfetch(tex_xmachine_message_navmap_cell_exit3_y, index + d_tex_xmachine_message_navmap_cell_exit3_y_offset);	temp_message.exit4_x = tex1Dfetch(tex_xmachine_message_navmap_cell_exit4_x, index + d_tex_xmachine_message_navmap_cell_exit4_x_offset);	temp_message.exit4_y = tex1Dfetch(tex_xmachine_message_navmap_cell_exit4_y, index + d_tex_xmachine_message_navmap_cell_exit4_y_offset);	temp_message.exit5_x = tex1Dfetch(tex_xmachine_message_navmap_cell_exit5_x, index + d_tex_xmachine_message_navmap_cell_exit5_x_offset);	temp_message.exit5_y = tex1Dfetch(tex_xmachine_message_navmap_cell_exit5_y, index + d_tex_xmachine_message_navmap_cell_exit5_y_offset);	temp_message.exit6_x = tex1Dfetch(tex_xmachine_message_navmap_cell_exit6_x, index + d_tex_xmachine_message_navmap_cell_exit6_x_offset);	temp_message.exit6_y = tex1Dfetch(tex_xmachine_message_navmap_cell_exit6_y, index + d_tex_xmachine_message_navmap_cell_exit6_y_offset);	

	message_share[threadIdx.x] = temp_message;

	return &message_share[threadIdx.x];
}

//method used by discrete agents accessing discrete messages to load messages into shared memory
__device__ void navmap_cell_message_to_sm(xmachine_message_navmap_cell_list* messages, char* message_share, int sm_index, int global_index){
		xmachine_message_navmap_cell temp_message;
		
		temp_message.x = messages->x[global_index];		
		temp_message.y = messages->y[global_index];		
		temp_message.exit_no = messages->exit_no[global_index];		
		temp_message.height = messages->height[global_index];		
		temp_message.collision_x = messages->collision_x[global_index];		
		temp_message.collision_y = messages->collision_y[global_index];		
		temp_message.exit0_x = messages->exit0_x[global_index];		
		temp_message.exit0_y = messages->exit0_y[global_index];		
		temp_message.exit1_x = messages->exit1_x[global_index];		
		temp_message.exit1_y = messages->exit1_y[global_index];		
		temp_message.exit2_x = messages->exit2_x[global_index];		
		temp_message.exit2_y = messages->exit2_y[global_index];		
		temp_message.exit3_x = messages->exit3_x[global_index];		
		temp_message.exit3_y = messages->exit3_y[global_index];		
		temp_message.exit4_x = messages->exit4_x[global_index];		
		temp_message.exit4_y = messages->exit4_y[global_index];		
		temp_message.exit5_x = messages->exit5_x[global_index];		
		temp_message.exit5_y = messages->exit5_y[global_index];		
		temp_message.exit6_x = messages->exit6_x[global_index];		
		temp_message.exit6_y = messages->exit6_y[global_index];		

	  int message_index = SHARE_INDEX(sm_index, sizeof(xmachine_message_navmap_cell));
	  xmachine_message_navmap_cell* sm_message = ((xmachine_message_navmap_cell*)&message_share[message_index]);
	  sm_message[0] = temp_message;
}

//Get first navmap_cell message 
//Used by discrete agents this accesses messages with texture cache. Agent position is determined by position in the grid/block
//Possibility of upto 8 thread divergances
__device__ xmachine_message_navmap_cell* get_first_navmap_cell_message_discrete(xmachine_message_navmap_cell_list* messages){

	//shared memory get from offset dependant on sm usage in function
	extern __shared__ int sm_data [];

	char* message_share = (char*)&sm_data[0];
  
	__syncthreads();

	int range = d_message_navmap_cell_range;
	int width = d_message_navmap_cell_width;
	int sm_grid_width = blockDim.x + (range* 2);
	
	
	int2 global_position;
	global_position.x = (blockIdx.x * blockDim.x) + threadIdx.x;
	global_position.y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = global_position.x + (global_position.y * width);
	

	//calculate the position in shared memeory of first load
	int2 sm_pos;
	sm_pos.x = threadIdx.x + range;
	sm_pos.y = threadIdx.y + range;
	int sm_index = (sm_pos.y * sm_grid_width) + sm_pos.x;

	//each thread loads to shared memeory (coalesced read)
	navmap_cell_message_to_sm(messages, message_share, sm_index, index);

	//check for edge conditions
	int left_border = (threadIdx.x < range);
	int right_border = (threadIdx.x >= (blockDim.x-range));
	int top_border = (threadIdx.y < range);
	int bottom_border = (threadIdx.y >= (blockDim.y-range));

	
	int  border_index;
	int  sm_border_index;

	//left
	if (left_border){	
		int2 border_index_2d = global_position;
		border_index_2d.x = sWRAP(border_index_2d.x - range, width);
		border_index = (border_index_2d.y * width) + border_index_2d.x;
		sm_border_index = (sm_pos.y * sm_grid_width) + threadIdx.x;
		
		navmap_cell_message_to_sm(messages, message_share, sm_border_index, border_index);
	}

	//right
	if (right_border){
		int2 border_index_2d = global_position;
		border_index_2d.x = sWRAP(border_index_2d.x + range, width);
		border_index = (border_index_2d.y * width) + border_index_2d.x;
		sm_border_index = (sm_pos.y * sm_grid_width) + (sm_pos.x + range);

		navmap_cell_message_to_sm(messages, message_share, sm_border_index, border_index);
	}

	//top
	if (top_border){
		int2 border_index_2d = global_position;
		border_index_2d.y = sWRAP(border_index_2d.y - range, width);
		border_index = (border_index_2d.y * width) + border_index_2d.x;
		sm_border_index = (threadIdx.y * sm_grid_width) + sm_pos.x;

		navmap_cell_message_to_sm(messages, message_share, sm_border_index, border_index);
	}

	//bottom
	if (bottom_border){
		int2 border_index_2d = global_position;
		border_index_2d.y = sWRAP(border_index_2d.y + range, width);
		border_index = (border_index_2d.y * width) + border_index_2d.x;
		sm_border_index = ((sm_pos.y + range) * sm_grid_width) + sm_pos.x;

		navmap_cell_message_to_sm(messages, message_share, sm_border_index, border_index);
	}

	//top left
	if ((top_border)&&(left_border)){	
		int2 border_index_2d = global_position;
		border_index_2d.x = sWRAP(border_index_2d.x - range, width);
		border_index_2d.y = sWRAP(border_index_2d.y - range, width);
		border_index = (border_index_2d.y * width) + border_index_2d.x;
		sm_border_index = (threadIdx.y * sm_grid_width) + threadIdx.x;
		
		navmap_cell_message_to_sm(messages, message_share, sm_border_index, border_index);
	}

	//top right
	if ((top_border)&&(right_border)){	
		int2 border_index_2d = global_position;
		border_index_2d.x = sWRAP(border_index_2d.x + range, width);
		border_index_2d.y = sWRAP(border_index_2d.y - range, width);
		border_index = (border_index_2d.y * width) + border_index_2d.x;
		sm_border_index = (threadIdx.y * sm_grid_width) + (sm_pos.x + range);
		
		navmap_cell_message_to_sm(messages, message_share, sm_border_index, border_index);
	}

	//bottom right
	if ((bottom_border)&&(right_border)){	
		int2 border_index_2d = global_position;
		border_index_2d.x = sWRAP(border_index_2d.x + range, width);
		border_index_2d.y = sWRAP(border_index_2d.y + range, width);
		border_index = (border_index_2d.y * width) + border_index_2d.x;
		sm_border_index = ((sm_pos.y + range) * sm_grid_width) + (sm_pos.x + range);
		
		navmap_cell_message_to_sm(messages, message_share, sm_border_index, border_index);
	}

	//bottom left
	if ((bottom_border)&&(left_border)){	
		int2 border_index_2d = global_position;
		border_index_2d.x = sWRAP(border_index_2d.x - range, width);
		border_index_2d.y = sWRAP(border_index_2d.y + range, width);
		border_index = (border_index_2d.y * width) + border_index_2d.x;
		sm_border_index = ((sm_pos.y + range) * sm_grid_width) + threadIdx.x;
		
		navmap_cell_message_to_sm(messages, message_share, sm_border_index, border_index);
	}

	__syncthreads();
	
  
	//top left of block position sm index
	sm_index = (threadIdx.y * sm_grid_width) + threadIdx.x;
	
	int message_index = SHARE_INDEX(sm_index, sizeof(xmachine_message_navmap_cell));
	xmachine_message_navmap_cell* temp = ((xmachine_message_navmap_cell*)&message_share[message_index]);
	temp->_relative = make_int2(-range, -range); //this is the relative position
	return temp;
}

//Get next navmap_cell message 
//Used by discrete agents this accesses messages through shared memeory which were all loaded on first message retrieval call.
__device__ xmachine_message_navmap_cell* get_next_navmap_cell_message_discrete(xmachine_message_navmap_cell* message, xmachine_message_navmap_cell_list* messages){

	//shared memory get from offset dependant on sm usage in function
	extern __shared__ int sm_data [];

	char* message_share = (char*)&sm_data[0];
  
	__syncthreads();
	
	int range = d_message_navmap_cell_range;
	int sm_grid_width = blockDim.x+(range*2);


	//Get previous position
	int2 previous_relative = message->_relative;

	//exit if at (range, range)
	if (previous_relative.x == range)
        if (previous_relative.y == range)
		    return false;

	//calculate next message relative position
	int2 next_relative = previous_relative;
	next_relative.x += 1;
	if ((next_relative.x)>range){
		next_relative.x = -range;
		next_relative.y = previous_relative.y + 1;
	}

	//skip own message
	if (next_relative.x == 0)
        if (next_relative.y == 0)
		    next_relative.x += 1;


	//calculate the next message position
	int2 next_position;// = block_position+next_relative;
	//offset next position by the sm border size
	next_position.x = threadIdx.x + next_relative.x + range;
	next_position.y = threadIdx.y + next_relative.y + range;

	int sm_index = next_position.x + (next_position.y * sm_grid_width);
	
	__syncthreads();
  
	int message_index = SHARE_INDEX(sm_index, sizeof(xmachine_message_navmap_cell));
	xmachine_message_navmap_cell* temp = ((xmachine_message_navmap_cell*)&message_share[message_index]);
	temp->_relative = next_relative; //this is the relative position
	return temp;
}

//Get first navmap_cell message
template <int AGENT_TYPE>
__device__ xmachine_message_navmap_cell* get_first_navmap_cell_message(xmachine_message_navmap_cell_list* messages, int agent_x, int agent_y){

	if (AGENT_TYPE == DISCRETE_2D)	//use shared memory method
		return get_first_navmap_cell_message_discrete(messages);
	else	//use texture fetching method
		return get_first_navmap_cell_message_continuous(messages, agent_x, agent_y);

}

//Get next navmap_cell message
template <int AGENT_TYPE>
__device__ xmachine_message_navmap_cell* get_next_navmap_cell_message(xmachine_message_navmap_cell* message, xmachine_message_navmap_cell_list* messages){

	if (AGENT_TYPE == DISCRETE_2D)	//use shared memory method
		return get_next_navmap_cell_message_discrete(message, messages);
	else	//use texture fetching method
		return get_next_navmap_cell_message_continuous(message, messages);

}


	
/////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dyanamically created GPU kernals  */



/**
 *
 */
__global__ void GPUFLAME_output_pedestrian_location(xmachine_memory_agent_list* agents, xmachine_message_pedestrian_location_list* pedestrian_location_messages){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    //For agents not using non partitioned message input check the agent bounds
    if (index >= d_xmachine_memory_agent_count)
        return;
    

	//SoA to AoS - xmachine_memory_output_pedestrian_location Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_agent agent;
	agent.x = agents->x[index];
	agent.y = agents->y[index];
	agent.velx = agents->velx[index];
	agent.vely = agents->vely[index];
	agent.steer_x = agents->steer_x[index];
	agent.steer_y = agents->steer_y[index];
	agent.height = agents->height[index];
	agent.exit_no = agents->exit_no[index];
	agent.speed = agents->speed[index];
	agent.lod = agents->lod[index];
	agent.animate = agents->animate[index];
	agent.animate_dir = agents->animate_dir[index];

	//FLAME function call
	int dead = !output_pedestrian_location(&agent, pedestrian_location_messages	);
	
	//continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_output_pedestrian_location Coalesced memory write (ignore arrays)
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
	agents->velx[index] = agent.velx;
	agents->vely[index] = agent.vely;
	agents->steer_x[index] = agent.steer_x;
	agents->steer_y[index] = agent.steer_y;
	agents->height[index] = agent.height;
	agents->exit_no[index] = agent.exit_no;
	agents->speed[index] = agent.speed;
	agents->lod[index] = agent.lod;
	agents->animate[index] = agent.animate;
	agents->animate_dir[index] = agent.animate_dir;
}

/**
 *
 */
__global__ void GPUFLAME_avoid_pedestrians(xmachine_memory_agent_list* agents, xmachine_message_pedestrian_location_list* pedestrian_location_messages, xmachine_message_pedestrian_location_PBM* partition_matrix, RNG_rand48* rand48){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    //For agents not using non partitioned message input check the agent bounds
    if (index >= d_xmachine_memory_agent_count)
        return;
    

	//SoA to AoS - xmachine_memory_avoid_pedestrians Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_agent agent;
	agent.x = agents->x[index];
	agent.y = agents->y[index];
	agent.velx = agents->velx[index];
	agent.vely = agents->vely[index];
	agent.steer_x = agents->steer_x[index];
	agent.steer_y = agents->steer_y[index];
	agent.height = agents->height[index];
	agent.exit_no = agents->exit_no[index];
	agent.speed = agents->speed[index];
	agent.lod = agents->lod[index];
	agent.animate = agents->animate[index];
	agent.animate_dir = agents->animate_dir[index];

	//FLAME function call
	int dead = !avoid_pedestrians(&agent, pedestrian_location_messages, partition_matrix, rand48);
	
	//continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_avoid_pedestrians Coalesced memory write (ignore arrays)
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
	agents->velx[index] = agent.velx;
	agents->vely[index] = agent.vely;
	agents->steer_x[index] = agent.steer_x;
	agents->steer_y[index] = agent.steer_y;
	agents->height[index] = agent.height;
	agents->exit_no[index] = agent.exit_no;
	agents->speed[index] = agent.speed;
	agents->lod[index] = agent.lod;
	agents->animate[index] = agent.animate;
	agents->animate_dir[index] = agent.animate_dir;
}

/**
 *
 */
__global__ void GPUFLAME_force_flow(xmachine_memory_agent_list* agents, xmachine_message_navmap_cell_list* navmap_cell_messages, RNG_rand48* rand48){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    //For agents not using non partitioned message input check the agent bounds
    if (index >= d_xmachine_memory_agent_count)
        return;
    

	//SoA to AoS - xmachine_memory_force_flow Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_agent agent;
	agent.x = agents->x[index];
	agent.y = agents->y[index];
	agent.velx = agents->velx[index];
	agent.vely = agents->vely[index];
	agent.steer_x = agents->steer_x[index];
	agent.steer_y = agents->steer_y[index];
	agent.height = agents->height[index];
	agent.exit_no = agents->exit_no[index];
	agent.speed = agents->speed[index];
	agent.lod = agents->lod[index];
	agent.animate = agents->animate[index];
	agent.animate_dir = agents->animate_dir[index];

	//FLAME function call
	int dead = !force_flow(&agent, navmap_cell_messages, rand48);
	
	//continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_force_flow Coalesced memory write (ignore arrays)
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
	agents->velx[index] = agent.velx;
	agents->vely[index] = agent.vely;
	agents->steer_x[index] = agent.steer_x;
	agents->steer_y[index] = agent.steer_y;
	agents->height[index] = agent.height;
	agents->exit_no[index] = agent.exit_no;
	agents->speed[index] = agent.speed;
	agents->lod[index] = agent.lod;
	agents->animate[index] = agent.animate;
	agents->animate_dir[index] = agent.animate_dir;
}

/**
 *
 */
__global__ void GPUFLAME_move(xmachine_memory_agent_list* agents){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    //For agents not using non partitioned message input check the agent bounds
    if (index >= d_xmachine_memory_agent_count)
        return;
    

	//SoA to AoS - xmachine_memory_move Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_agent agent;
	agent.x = agents->x[index];
	agent.y = agents->y[index];
	agent.velx = agents->velx[index];
	agent.vely = agents->vely[index];
	agent.steer_x = agents->steer_x[index];
	agent.steer_y = agents->steer_y[index];
	agent.height = agents->height[index];
	agent.exit_no = agents->exit_no[index];
	agent.speed = agents->speed[index];
	agent.lod = agents->lod[index];
	agent.animate = agents->animate[index];
	agent.animate_dir = agents->animate_dir[index];

	//FLAME function call
	int dead = !move(&agent);
	
	//continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_move Coalesced memory write (ignore arrays)
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
	agents->velx[index] = agent.velx;
	agents->vely[index] = agent.vely;
	agents->steer_x[index] = agent.steer_x;
	agents->steer_y[index] = agent.steer_y;
	agents->height[index] = agent.height;
	agents->exit_no[index] = agent.exit_no;
	agents->speed[index] = agent.speed;
	agents->lod[index] = agent.lod;
	agents->animate[index] = agent.animate;
	agents->animate_dir[index] = agent.animate_dir;
}

/**
 *
 */
__global__ void GPUFLAME_output_navmap_cells(xmachine_memory_navmap_list* agents, xmachine_message_navmap_cell_list* navmap_cell_messages){
	
	
	//discrete agent: index is position in 2D agent grid
	int width = (blockDim.x * gridDim.x);
	int2 global_position;
	global_position.x = (blockIdx.x * blockDim.x) + threadIdx.x;
	global_position.y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = global_position.x + (global_position.y * width);
	

	//SoA to AoS - xmachine_memory_output_navmap_cells Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_navmap agent;
	agent.x = agents->x[index];
	agent.y = agents->y[index];
	agent.exit_no = agents->exit_no[index];
	agent.height = agents->height[index];
	agent.collision_x = agents->collision_x[index];
	agent.collision_y = agents->collision_y[index];
	agent.exit0_x = agents->exit0_x[index];
	agent.exit0_y = agents->exit0_y[index];
	agent.exit1_x = agents->exit1_x[index];
	agent.exit1_y = agents->exit1_y[index];
	agent.exit2_x = agents->exit2_x[index];
	agent.exit2_y = agents->exit2_y[index];
	agent.exit3_x = agents->exit3_x[index];
	agent.exit3_y = agents->exit3_y[index];
	agent.exit4_x = agents->exit4_x[index];
	agent.exit4_y = agents->exit4_y[index];
	agent.exit5_x = agents->exit5_x[index];
	agent.exit5_y = agents->exit5_y[index];
	agent.exit6_x = agents->exit6_x[index];
	agent.exit6_y = agents->exit6_y[index];

	//FLAME function call
	output_navmap_cells(&agent, navmap_cell_messages	);
	
	

	//AoS to SoA - xmachine_memory_output_navmap_cells Coalesced memory write (ignore arrays)
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
	agents->exit_no[index] = agent.exit_no;
	agents->height[index] = agent.height;
	agents->collision_x[index] = agent.collision_x;
	agents->collision_y[index] = agent.collision_y;
	agents->exit0_x[index] = agent.exit0_x;
	agents->exit0_y[index] = agent.exit0_y;
	agents->exit1_x[index] = agent.exit1_x;
	agents->exit1_y[index] = agent.exit1_y;
	agents->exit2_x[index] = agent.exit2_x;
	agents->exit2_y[index] = agent.exit2_y;
	agents->exit3_x[index] = agent.exit3_x;
	agents->exit3_y[index] = agent.exit3_y;
	agents->exit4_x[index] = agent.exit4_x;
	agents->exit4_y[index] = agent.exit4_y;
	agents->exit5_x[index] = agent.exit5_x;
	agents->exit5_y[index] = agent.exit5_y;
	agents->exit6_x[index] = agent.exit6_x;
	agents->exit6_y[index] = agent.exit6_y;
}

/**
 *
 */
__global__ void GPUFLAME_generate_pedestrians(xmachine_memory_navmap_list* agents, xmachine_memory_agent_list* agent_agents, RNG_rand48* rand48){
	
	
	//discrete agent: index is position in 2D agent grid
	int width = (blockDim.x * gridDim.x);
	int2 global_position;
	global_position.x = (blockIdx.x * blockDim.x) + threadIdx.x;
	global_position.y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = global_position.x + (global_position.y * width);
	

	//SoA to AoS - xmachine_memory_generate_pedestrians Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_navmap agent;
	agent.x = agents->x[index];
	agent.y = agents->y[index];
	agent.exit_no = agents->exit_no[index];
	agent.height = agents->height[index];
	agent.collision_x = agents->collision_x[index];
	agent.collision_y = agents->collision_y[index];
	agent.exit0_x = agents->exit0_x[index];
	agent.exit0_y = agents->exit0_y[index];
	agent.exit1_x = agents->exit1_x[index];
	agent.exit1_y = agents->exit1_y[index];
	agent.exit2_x = agents->exit2_x[index];
	agent.exit2_y = agents->exit2_y[index];
	agent.exit3_x = agents->exit3_x[index];
	agent.exit3_y = agents->exit3_y[index];
	agent.exit4_x = agents->exit4_x[index];
	agent.exit4_y = agents->exit4_y[index];
	agent.exit5_x = agents->exit5_x[index];
	agent.exit5_y = agents->exit5_y[index];
	agent.exit6_x = agents->exit6_x[index];
	agent.exit6_y = agents->exit6_y[index];

	//FLAME function call
	generate_pedestrians(&agent, agent_agents, rand48);
	
	

	//AoS to SoA - xmachine_memory_generate_pedestrians Coalesced memory write (ignore arrays)
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
	agents->exit_no[index] = agent.exit_no;
	agents->height[index] = agent.height;
	agents->collision_x[index] = agent.collision_x;
	agents->collision_y[index] = agent.collision_y;
	agents->exit0_x[index] = agent.exit0_x;
	agents->exit0_y[index] = agent.exit0_y;
	agents->exit1_x[index] = agent.exit1_x;
	agents->exit1_y[index] = agent.exit1_y;
	agents->exit2_x[index] = agent.exit2_x;
	agents->exit2_y[index] = agent.exit2_y;
	agents->exit3_x[index] = agent.exit3_x;
	agents->exit3_y[index] = agent.exit3_y;
	agents->exit4_x[index] = agent.exit4_x;
	agents->exit4_y[index] = agent.exit4_y;
	agents->exit5_x[index] = agent.exit5_x;
	agents->exit5_y[index] = agent.exit5_y;
	agents->exit6_x[index] = agent.exit6_x;
	agents->exit6_y[index] = agent.exit6_y;
}

	
	
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Rand48 functions */

__device__ static uint2 RNG_rand48_iterate_single(uint2 Xn, uint2 A, uint2 C)
{
	unsigned int R0, R1;

	// low 24-bit multiplication
	const unsigned int lo00 = __umul24(Xn.x, A.x);
	const unsigned int hi00 = __umulhi(Xn.x, A.x);

	// 24bit distribution of 32bit multiplication results
	R0 = (lo00 & 0xFFFFFF);
	R1 = (lo00 >> 24) | (hi00 << 8);

	R0 += C.x; R1 += C.y;

	// transfer overflows
	R1 += (R0 >> 24);
	R0 &= 0xFFFFFF;

	// cross-terms, low/hi 24-bit multiplication
	R1 += __umul24(Xn.y, A.x);
	R1 += __umul24(Xn.x, A.y);

	R1 &= 0xFFFFFF;

	return make_uint2(R0, R1);
}

//Templated function
template <int AGENT_TYPE>
__device__ float rnd(RNG_rand48* rand48){

	int index;
	
	//calculate the agents index in global agent list
	if (AGENT_TYPE == DISCRETE_2D){
		int width = (blockDim.x * gridDim.x);
		int2 global_position;
		global_position.x = (blockIdx.x * blockDim.x) + threadIdx.x;
		global_position.y = (blockIdx.y * blockDim.y) + threadIdx.y;
		index = global_position.x + (global_position.y * width);
	}else//AGENT_TYPE == CONTINOUS
		index = threadIdx.x + blockIdx.x*blockDim.x;

	uint2 state = rand48->seeds[index];
	uint2 A = rand48->A;
	uint2 C = rand48->C;

	int rand = ( state.x >> 17 ) | ( state.y << 7);

	// this actually iterates the RNG
	state = RNG_rand48_iterate_single(state, A, C);

	rand48->seeds[index] = state;

	return (float)rand/2147483647;
}

__device__ float rnd(RNG_rand48* rand48){
	return rnd<DISCRETE_2D>(rand48);
}

#endif //_FLAMEGPU_KERNELS_H_
