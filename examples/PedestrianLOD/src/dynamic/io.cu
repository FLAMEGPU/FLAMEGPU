#include "hip/hip_runtime.h"

/*
* FLAME GPU v 1.4.0 for CUDA 6
* Copyright 2015 University of Sheffield.
* Author: Dr Paul Richmond
* Contact: p.richmond@sheffield.ac.uk (http://www.paulrichmond.staff.shef.ac.uk)
*
* University of Sheffield retain all intellectual property and
* proprietary rights in and to this software and related documentation.
* Any use, reproduction, disclosure, or distribution of this software
* and related documentation without an express license agreement from
* University of Sheffield is strictly prohibited.
*
* For terms of licence agreement please attached licence or view licence
* on www.flamegpu.com website.
*
*/

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <limits.h>
	

// include header
#include "header.h"

float3 agent_maximum;
float3 agent_minimum;

void readIntArrayInput(char* buffer, int *array, unsigned int expected_items){
    unsigned int i = 0;
    const char s[2] = ",";
    char * token;

    token = strtok(buffer, s);
    while (token != NULL){
        if (i>=expected_items){
            printf("Error: Agent Memeory array has too many items, expected %d!\n", expected_items);
            exit(0);
        }
        
        array[i++] = atoi(token);
        
        token = strtok(NULL, s);
    }
    if (i != expected_items){
        printf("Error: Agent Memeory array has %d items, expected %d!\n", i, expected_items);
        exit(0);
    }
}

void readFloatArrayInput(char* buffer, float *array, unsigned int expected_items){
    unsigned int i = 0;
    const char s[2] = ",";
    char * token;

    token = strtok(buffer, s);
    while (token != NULL){
        if (i>=expected_items){
            printf("Error: Agent Memeory array has too many items, expected %d!\n", expected_items);
            exit(0);
        }
        
        array[i++] = (float)atof(token);
        
        token = strtok(NULL, s);
    }
    if (i != expected_items){
        printf("Error: Agent Memeory array has %d items, expected %d!\n", i, expected_items);
        exit(0);
    }
}

void saveIterationData(char* outputpath, int iteration_number, xmachine_memory_agent_list* h_agents_default, xmachine_memory_agent_list* d_agents_default, int h_xmachine_memory_agent_default_count)
{
	hipError_t cudaStatus;
	
	//Device to host memory transfer
	
	cudaStatus = hipMemcpy( h_agents_default, d_agents_default, sizeof(xmachine_memory_agent_list), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr,"Error Copying agent Agent default State Memory from GPU: %s\n", hipGetErrorString(cudaStatus));
		exit(cudaStatus);
	}
	
	/* Pointer to file */
	FILE *file;
	char data[100];

	sprintf(data, "%s%i.xml", outputpath, iteration_number);
	//printf("Writing itteration %i data to %s\n", iteration_number, data);
	file = fopen(data, "w");
	fputs("<states>\n<itno>", file);
	sprintf(data, "%i", iteration_number);
	fputs(data, file);
	fputs("</itno>\n", file);
	fputs("<environment>\n" , file);
	fputs("</environment>\n" , file);

	//Write each agent agent to xml
	for (int i=0; i<h_xmachine_memory_agent_default_count; i++){
		fputs("<xagent>\n" , file);
		fputs("<name>agent</name>\n", file);
        
		fputs("<x>", file);
        sprintf(data, "%f", h_agents_default->x[i]);
		fputs(data, file);
		fputs("</x>\n", file);
        
		fputs("<y>", file);
        sprintf(data, "%f", h_agents_default->y[i]);
		fputs(data, file);
		fputs("</y>\n", file);
        
		fputs("<velx>", file);
        sprintf(data, "%f", h_agents_default->velx[i]);
		fputs(data, file);
		fputs("</velx>\n", file);
        
		fputs("<vely>", file);
        sprintf(data, "%f", h_agents_default->vely[i]);
		fputs(data, file);
		fputs("</vely>\n", file);
        
		fputs("<steer_x>", file);
        sprintf(data, "%f", h_agents_default->steer_x[i]);
		fputs(data, file);
		fputs("</steer_x>\n", file);
        
		fputs("<steer_y>", file);
        sprintf(data, "%f", h_agents_default->steer_y[i]);
		fputs(data, file);
		fputs("</steer_y>\n", file);
        
		fputs("<height>", file);
        sprintf(data, "%f", h_agents_default->height[i]);
		fputs(data, file);
		fputs("</height>\n", file);
        
		fputs("<exit_no>", file);
        sprintf(data, "%i", h_agents_default->exit_no[i]);
		fputs(data, file);
		fputs("</exit_no>\n", file);
        
		fputs("<speed>", file);
        sprintf(data, "%f", h_agents_default->speed[i]);
		fputs(data, file);
		fputs("</speed>\n", file);
        
		fputs("<lod>", file);
        sprintf(data, "%i", h_agents_default->lod[i]);
		fputs(data, file);
		fputs("</lod>\n", file);
        
		fputs("<animate>", file);
        sprintf(data, "%f", h_agents_default->animate[i]);
		fputs(data, file);
		fputs("</animate>\n", file);
        
		fputs("<animate_dir>", file);
        sprintf(data, "%i", h_agents_default->animate_dir[i]);
		fputs(data, file);
		fputs("</animate_dir>\n", file);
        
		fputs("</xagent>\n", file);
	}
	
	

	fputs("</states>\n" , file);
	
	/* Close the file */
	fclose(file);
}

void readInitialStates(char* inputpath, xmachine_memory_agent_list* h_agents, int* h_xmachine_memory_agent_count)
{

	int temp = 0;
	int* itno = &temp;

	/* Pointer to file */
	FILE *file;
	/* Char and char buffer for reading file to */
	char c = ' ';
	char buffer[10000];
	char agentname[1000];

	/* Pointer to x-memory for initial state data */
	/*xmachine * current_xmachine;*/
	/* Variables for checking tags */
	int reading, i;
	int in_tag, in_itno, in_name;
    int in_agent_x;
    int in_agent_y;
    int in_agent_velx;
    int in_agent_vely;
    int in_agent_steer_x;
    int in_agent_steer_y;
    int in_agent_height;
    int in_agent_exit_no;
    int in_agent_speed;
    int in_agent_lod;
    int in_agent_animate;
    int in_agent_animate_dir;

	/* for continuous agents: set agent count to zero */	
	*h_xmachine_memory_agent_count = 0;
	
	/* Variables for initial state data */
	float agent_x;
	float agent_y;
	float agent_velx;
	float agent_vely;
	float agent_steer_x;
	float agent_steer_y;
	float agent_height;
	int agent_exit_no;
	float agent_speed;
	int agent_lod;
	float agent_animate;
	int agent_animate_dir;
	
	/* Open config file to read-only */
	if((file = fopen(inputpath, "r"))==NULL)
	{
		printf("error opening initial states\n");
		exit(0);
	}
	
	/* Initialise variables */
    agent_maximum.x = 0;
    agent_maximum.y = 0;
    agent_maximum.z = 0;
    agent_minimum.x = 0;
    agent_minimum.y = 0;
    agent_minimum.z = 0;
	reading = 1;
	in_tag = 0;
	in_itno = 0;
	in_name = 0;
	in_agent_x = 0;
	in_agent_y = 0;
	in_agent_velx = 0;
	in_agent_vely = 0;
	in_agent_steer_x = 0;
	in_agent_steer_y = 0;
	in_agent_height = 0;
	in_agent_exit_no = 0;
	in_agent_speed = 0;
	in_agent_lod = 0;
	in_agent_animate = 0;
	in_agent_animate_dir = 0;
	//set all agent values to 0
	//If this is not done then it will cause errors in emu mode where undefined memory is not 0
	for (int k=0; k<xmachine_memory_agent_MAX; k++)
	{	
		h_agents->x[k] = 0;
		h_agents->y[k] = 0;
		h_agents->velx[k] = 0;
		h_agents->vely[k] = 0;
		h_agents->steer_x[k] = 0;
		h_agents->steer_y[k] = 0;
		h_agents->height[k] = 0;
		h_agents->exit_no[k] = 0;
		h_agents->speed[k] = 0;
		h_agents->lod[k] = 0;
		h_agents->animate[k] = 0;
		h_agents->animate_dir[k] = 0;
	}
	

	/* Default variables for memory */
    agent_x = 0;
    agent_y = 0;
    agent_velx = 0;
    agent_vely = 0;
    agent_steer_x = 0;
    agent_steer_y = 0;
    agent_height = 0;
    agent_exit_no = 0;
    agent_speed = 0;
    agent_lod = 0;
    agent_animate = 0;
    agent_animate_dir = 0;

	/* Read file until end of xml */
    i = 0;
	while(reading==1)
	{
		/* Get the next char from the file */
		c = (char)fgetc(file);
		
		/* If the end of a tag */
		if(c == '>')
		{
			/* Place 0 at end of buffer to make chars a string */
			buffer[i] = 0;
			
			if(strcmp(buffer, "states") == 0) reading = 1;
			if(strcmp(buffer, "/states") == 0) reading = 0;
			if(strcmp(buffer, "itno") == 0) in_itno = 1;
			if(strcmp(buffer, "/itno") == 0) in_itno = 0;
			if(strcmp(buffer, "name") == 0) in_name = 1;
			if(strcmp(buffer, "/name") == 0) in_name = 0;
			if(strcmp(buffer, "/xagent") == 0)
			{
				if(strcmp(agentname, "agent") == 0)
				{		
					if (*h_xmachine_memory_agent_count > xmachine_memory_agent_MAX){
						printf("ERROR: MAX Buffer size (%i) for agent agent exceeded whilst reading data\n", xmachine_memory_agent_MAX);
						// Close the file and stop reading
						fclose(file);
						exit(0);
					}
                    
					h_agents->x[*h_xmachine_memory_agent_count] = agent_x;//Check maximum x value
                    if(agent_maximum.x < agent_x)
                        agent_maximum.x = (float)agent_x;
                    //Check minimum x value
                    if(agent_minimum.x > agent_x)
                        agent_minimum.x = (float)agent_x;
                    
					h_agents->y[*h_xmachine_memory_agent_count] = agent_y;//Check maximum y value
                    if(agent_maximum.y < agent_y)
                        agent_maximum.y = (float)agent_y;
                    //Check minimum y value
                    if(agent_minimum.y > agent_y)
                        agent_minimum.y = (float)agent_y;
                    
					h_agents->velx[*h_xmachine_memory_agent_count] = agent_velx;
					h_agents->vely[*h_xmachine_memory_agent_count] = agent_vely;
					h_agents->steer_x[*h_xmachine_memory_agent_count] = agent_steer_x;
					h_agents->steer_y[*h_xmachine_memory_agent_count] = agent_steer_y;
					h_agents->height[*h_xmachine_memory_agent_count] = agent_height;
					h_agents->exit_no[*h_xmachine_memory_agent_count] = agent_exit_no;
					h_agents->speed[*h_xmachine_memory_agent_count] = agent_speed;
					h_agents->lod[*h_xmachine_memory_agent_count] = agent_lod;
					h_agents->animate[*h_xmachine_memory_agent_count] = agent_animate;
					h_agents->animate_dir[*h_xmachine_memory_agent_count] = agent_animate_dir;
					(*h_xmachine_memory_agent_count) ++;	
				}
				else
				{
					printf("Warning: agent name undefined - '%s'\n", agentname);
				}
				

				
				/* Reset xagent variables */
                agent_x = 0;
                agent_y = 0;
                agent_velx = 0;
                agent_vely = 0;
                agent_steer_x = 0;
                agent_steer_y = 0;
                agent_height = 0;
                agent_exit_no = 0;
                agent_speed = 0;
                agent_lod = 0;
                agent_animate = 0;
                agent_animate_dir = 0;

			}
			if(strcmp(buffer, "x") == 0) in_agent_x = 1;
			if(strcmp(buffer, "/x") == 0) in_agent_x = 0;
			if(strcmp(buffer, "y") == 0) in_agent_y = 1;
			if(strcmp(buffer, "/y") == 0) in_agent_y = 0;
			if(strcmp(buffer, "velx") == 0) in_agent_velx = 1;
			if(strcmp(buffer, "/velx") == 0) in_agent_velx = 0;
			if(strcmp(buffer, "vely") == 0) in_agent_vely = 1;
			if(strcmp(buffer, "/vely") == 0) in_agent_vely = 0;
			if(strcmp(buffer, "steer_x") == 0) in_agent_steer_x = 1;
			if(strcmp(buffer, "/steer_x") == 0) in_agent_steer_x = 0;
			if(strcmp(buffer, "steer_y") == 0) in_agent_steer_y = 1;
			if(strcmp(buffer, "/steer_y") == 0) in_agent_steer_y = 0;
			if(strcmp(buffer, "height") == 0) in_agent_height = 1;
			if(strcmp(buffer, "/height") == 0) in_agent_height = 0;
			if(strcmp(buffer, "exit_no") == 0) in_agent_exit_no = 1;
			if(strcmp(buffer, "/exit_no") == 0) in_agent_exit_no = 0;
			if(strcmp(buffer, "speed") == 0) in_agent_speed = 1;
			if(strcmp(buffer, "/speed") == 0) in_agent_speed = 0;
			if(strcmp(buffer, "lod") == 0) in_agent_lod = 1;
			if(strcmp(buffer, "/lod") == 0) in_agent_lod = 0;
			if(strcmp(buffer, "animate") == 0) in_agent_animate = 1;
			if(strcmp(buffer, "/animate") == 0) in_agent_animate = 0;
			if(strcmp(buffer, "animate_dir") == 0) in_agent_animate_dir = 1;
			if(strcmp(buffer, "/animate_dir") == 0) in_agent_animate_dir = 0;
			
			
			/* End of tag and reset buffer */
			in_tag = 0;
			i = 0;
		}
		/* If start of tag */
		else if(c == '<')
		{
			/* Place /0 at end of buffer to end numbers */
			buffer[i] = 0;
			/* Flag in tag */
			in_tag = 1;
			
			if(in_itno) *itno = atoi(buffer);
			if(in_name) strcpy(agentname, buffer);
			else
			{
				if(in_agent_x){ 
                    agent_x = (float) atof(buffer);    
                }
				if(in_agent_y){ 
                    agent_y = (float) atof(buffer);    
                }
				if(in_agent_velx){ 
                    agent_velx = (float) atof(buffer);    
                }
				if(in_agent_vely){ 
                    agent_vely = (float) atof(buffer);    
                }
				if(in_agent_steer_x){ 
                    agent_steer_x = (float) atof(buffer);    
                }
				if(in_agent_steer_y){ 
                    agent_steer_y = (float) atof(buffer);    
                }
				if(in_agent_height){ 
                    agent_height = (float) atof(buffer);    
                }
				if(in_agent_exit_no){ 
                    agent_exit_no = (int) atoi(buffer);    
                }
				if(in_agent_speed){ 
                    agent_speed = (float) atof(buffer);    
                }
				if(in_agent_lod){ 
                    agent_lod = (int) atoi(buffer);    
                }
				if(in_agent_animate){ 
                    agent_animate = (float) atof(buffer);    
                }
				if(in_agent_animate_dir){ 
                    agent_animate_dir = (int) atoi(buffer);    
                }
				
			}
			
			/* Reset buffer */
			i = 0;
		}
		/* If in tag put read char into buffer */
		else if(in_tag)
		{
			buffer[i] = c;
			i++;
		}
		/* If in data read char into buffer */
		else
		{
			buffer[i] = c;
			i++;
		}
	}
	/* Close the file */
	fclose(file);
}

float3 getMaximumBounds(){
    return agent_maximum;
}

float3 getMinimumBounds(){
    return agent_minimum;
}

