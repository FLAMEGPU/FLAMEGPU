#include "hip/hip_runtime.h"


/*
* FLAME GPU v 1.4.0 for CUDA 6
* Copyright 2015 University of Sheffield.
* Author: Dr Paul Richmond
* Contact: p.richmond@sheffield.ac.uk (http://www.paulrichmond.staff.shef.ac.uk)
*
* University of Sheffield retain all intellectual property and
* proprietary rights in and to this software and related documentation.
* Any use, reproduction, disclosure, or distribution of this software
* and related documentation without an express license agreement from
* University of Sheffield is strictly prohibited.
*
* For terms of licence agreement please attached licence or view licence
* on www.flamegpu.com website.
*
*/

#ifndef _FLAMEGPU_KERNELS_H_
#define _FLAMEGPU_KERNELS_H_

#include "header.h"


/* Agent count constants */

__constant__ int d_xmachine_memory_cell_count;

/* Agent state count constants */

__constant__ int d_xmachine_memory_cell_default_count;


/* Message constants */

/* state Message variables */
//Discrete Partitioning Variables
__constant__ int d_message_state_range;     /**< range of the discrete message*/
__constant__ int d_message_state_width;     /**< with of the message grid*/

	
    
//include each function file

#include "functions.c"
    
/* Texture bindings */
/* state Message Bindings */texture<int, 1, hipReadModeElementType> tex_xmachine_message_state_state;
__constant__ int d_tex_xmachine_message_state_state_offset;texture<int, 1, hipReadModeElementType> tex_xmachine_message_state_x;
__constant__ int d_tex_xmachine_message_state_x_offset;texture<int, 1, hipReadModeElementType> tex_xmachine_message_state_y;
__constant__ int d_tex_xmachine_message_state_y_offset;

    
#define WRAP(x,m) (((x)<m)?(x):(x%m)) /**< Simple wrap */
#define sWRAP(x,m) (((x)<m)?(((x)<0)?(m+(x)):(x)):(m-(x))) /**<signed integer wrap (no modulus) for negatives where 2m > |x| > m */

//PADDING WILL ONLY AVOID SM CONFLICTS FOR 32BIT
//SM_OFFSET REQUIRED AS FERMI STARTS INDEXING MEMORY FROM LOCATION 0 (i.e. NULL)??
__constant__ int d_SM_START;
__constant__ int d_PADDING;

//SM addressing macro to avoid conflicts (32 bit only)
#define SHARE_INDEX(i, s) (((s + d_PADDING)* i)+d_SM_START) /**<offset struct size by padding to avoid bank conflicts */

//if doubel support is needed then define the following function which requires sm_13 or later
#ifdef _DOUBLE_SUPPORT_REQUIRED_
__inline__ __device__ double tex1DfetchDouble(texture<int2, 1, hipReadModeElementType> tex, int i)
{
	int2 v = tex1Dfetch(tex, i);
  //IF YOU HAVE AN ERROR HERE THEN YOU ARE USING DOUBLE VALUES IN AGENT MEMORY AND NOT COMPILING FOR DOUBLE SUPPORTED HARDWARE
  //To compile for double supported hardware change the CUDA Build rule property "Use sm_13 Architecture (double support)" on the CUDA-Specific Propert Page of the CUDA Build Rule for simulation.cu
	return __hiloint2double(v.y, v.x);
}
#endif

/* Helper functions */
/** next_cell
 * Function used for finding the next cell when using spatial partitioning
 * Upddates the relative cell variable which can have value of -1, 0 or +1
 * @param relative_cell pointer to the relative cell position
 * @return boolean if there is a next cell. True unless relative_Cell value was 1,1,1
 */
__device__ int next_cell3D(int3* relative_cell)
{
	if (relative_cell->x < 1)
	{
		relative_cell->x++;
		return true;
	}
	relative_cell->x = -1;

	if (relative_cell->y < 1)
	{
		relative_cell->y++;
		return true;
	}
	relative_cell->y = -1;
	
	if (relative_cell->z < 1)
	{
		relative_cell->z++;
		return true;
	}
	relative_cell->z = -1;
	
	return false;
}

/** next_cell2D
 * Function used for finding the next cell when using spatial partitioning. Z component is ignored
 * Upddates the relative cell variable which can have value of -1, 0 or +1
 * @param relative_cell pointer to the relative cell position
 * @return boolean if there is a next cell. True unless relative_Cell value was 1,1
 */
__device__ int next_cell2D(int3* relative_cell)
{
	if (relative_cell->x < 1)
	{
		relative_cell->x++;
		return true;
	}
	relative_cell->x = -1;

	if (relative_cell->y < 1)
	{
		relative_cell->y++;
		return true;
	}
	relative_cell->y = -1;
	
	return false;
}


////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dyanamically created cell agent functions */

/** reset_cell_scan_input
 * cell agent reset scan input function
 * @param agents The xmachine_memory_cell_list agent list
 */
__global__ void reset_cell_scan_input(xmachine_memory_cell_list* agents){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	agents->_position[index] = 0;
	agents->_scan_input[index] = 0;
}


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dyanamically created state message functions */


/* Message functions */

template <int AGENT_TYPE>
__device__ void add_state_message(xmachine_message_state_list* messages, int state, int x, int y){
	if (AGENT_TYPE == DISCRETE_2D){
		int width = (blockDim.x * gridDim.x);
		int2 global_position;
		global_position.x = (blockIdx.x * blockDim.x) + threadIdx.x;
		global_position.y = (blockIdx.y * blockDim.y) + threadIdx.y;

		int index = global_position.x + (global_position.y * width);

		
		messages->state[index] = state;			
		messages->x[index] = x;			
		messages->y[index] = y;			
	}
	//else CONTINUOUS agents can not write to discrete space
}

//Used by continuous agents this accesses messages with texture cache. agent_x and agent_y are discrete positions in the message space
__device__ xmachine_message_state* get_first_state_message_continuous(xmachine_message_state_list* messages,  int agent_x, int agent_y){

	//shared memory get from offset dependant on sm usage in function
	extern __shared__ int sm_data [];

	xmachine_message_state* message_share = (xmachine_message_state*)&sm_data[0];
	
	int range = d_message_state_range;
	int width = d_message_state_width;
	
	int2 global_position;
	global_position.x = sWRAP(agent_x-range , width);
	global_position.y = sWRAP(agent_y-range , width);
	

	int index = ((global_position.y)* width) + global_position.x;
	
	xmachine_message_state temp_message;
	temp_message._position = make_int2(agent_x, agent_y);
	temp_message._relative = make_int2(-range, -range);

	temp_message.state = tex1Dfetch(tex_xmachine_message_state_state, index + d_tex_xmachine_message_state_state_offset);temp_message.x = tex1Dfetch(tex_xmachine_message_state_x, index + d_tex_xmachine_message_state_x_offset);temp_message.y = tex1Dfetch(tex_xmachine_message_state_y, index + d_tex_xmachine_message_state_y_offset);
	
	message_share[threadIdx.x] = temp_message;

	//return top left of messages
	return &message_share[threadIdx.x];
}

//Get next state message  continuous
//Used by continuous agents this accesses messages with texture cache (agent position in discrete space was set when accessing first message)
__device__ xmachine_message_state* get_next_state_message_continuous(xmachine_message_state* message, xmachine_message_state_list* messages){

	//shared memory get from offset dependant on sm usage in function
	extern __shared__ int sm_data [];

	xmachine_message_state* message_share = (xmachine_message_state*)&sm_data[0];
	
	int range = d_message_state_range;
	int width = d_message_state_width;

	//Get previous position
	int2 previous_relative = message->_relative;

	//exit if at (range, range)
	if (previous_relative.x == (range))
        if (previous_relative.y == (range))
		    return false;

	//calculate next message relative position
	int2 next_relative = previous_relative;
	next_relative.x += 1;
	if ((next_relative.x)>range){
		next_relative.x = -range;
		next_relative.y = previous_relative.y + 1;
	}

	//skip own message
	if (next_relative.x == 0)
        if (next_relative.y == 0)
		    next_relative.x += 1;

	int2 global_position;
	global_position.x =	sWRAP(message->_position.x + next_relative.x, width);
	global_position.y = sWRAP(message->_position.y + next_relative.y, width);

	int index = ((global_position.y)* width) + (global_position.x);
	
	xmachine_message_state temp_message;
	temp_message._position = message->_position;
	temp_message._relative = next_relative;

	temp_message.state = tex1Dfetch(tex_xmachine_message_state_state, index + d_tex_xmachine_message_state_state_offset);	temp_message.x = tex1Dfetch(tex_xmachine_message_state_x, index + d_tex_xmachine_message_state_x_offset);	temp_message.y = tex1Dfetch(tex_xmachine_message_state_y, index + d_tex_xmachine_message_state_y_offset);	

	message_share[threadIdx.x] = temp_message;

	return &message_share[threadIdx.x];
}

//method used by discrete agents accessing discrete messages to load messages into shared memory
__device__ void state_message_to_sm(xmachine_message_state_list* messages, char* message_share, int sm_index, int global_index){
		xmachine_message_state temp_message;
		
		temp_message.state = messages->state[global_index];		
		temp_message.x = messages->x[global_index];		
		temp_message.y = messages->y[global_index];		

	  int message_index = SHARE_INDEX(sm_index, sizeof(xmachine_message_state));
	  xmachine_message_state* sm_message = ((xmachine_message_state*)&message_share[message_index]);
	  sm_message[0] = temp_message;
}

//Get first state message 
//Used by discrete agents this accesses messages with texture cache. Agent position is determined by position in the grid/block
//Possibility of upto 8 thread divergances
__device__ xmachine_message_state* get_first_state_message_discrete(xmachine_message_state_list* messages){

	//shared memory get from offset dependant on sm usage in function
	extern __shared__ int sm_data [];

	char* message_share = (char*)&sm_data[0];
  
	__syncthreads();

	int range = d_message_state_range;
	int width = d_message_state_width;
	int sm_grid_width = blockDim.x + (range* 2);
	
	
	int2 global_position;
	global_position.x = (blockIdx.x * blockDim.x) + threadIdx.x;
	global_position.y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = global_position.x + (global_position.y * width);
	

	//calculate the position in shared memeory of first load
	int2 sm_pos;
	sm_pos.x = threadIdx.x + range;
	sm_pos.y = threadIdx.y + range;
	int sm_index = (sm_pos.y * sm_grid_width) + sm_pos.x;

	//each thread loads to shared memeory (coalesced read)
	state_message_to_sm(messages, message_share, sm_index, index);

	//check for edge conditions
	int left_border = (threadIdx.x < range);
	int right_border = (threadIdx.x >= (blockDim.x-range));
	int top_border = (threadIdx.y < range);
	int bottom_border = (threadIdx.y >= (blockDim.y-range));

	
	int  border_index;
	int  sm_border_index;

	//left
	if (left_border){	
		int2 border_index_2d = global_position;
		border_index_2d.x = sWRAP(border_index_2d.x - range, width);
		border_index = (border_index_2d.y * width) + border_index_2d.x;
		sm_border_index = (sm_pos.y * sm_grid_width) + threadIdx.x;
		
		state_message_to_sm(messages, message_share, sm_border_index, border_index);
	}

	//right
	if (right_border){
		int2 border_index_2d = global_position;
		border_index_2d.x = sWRAP(border_index_2d.x + range, width);
		border_index = (border_index_2d.y * width) + border_index_2d.x;
		sm_border_index = (sm_pos.y * sm_grid_width) + (sm_pos.x + range);

		state_message_to_sm(messages, message_share, sm_border_index, border_index);
	}

	//top
	if (top_border){
		int2 border_index_2d = global_position;
		border_index_2d.y = sWRAP(border_index_2d.y - range, width);
		border_index = (border_index_2d.y * width) + border_index_2d.x;
		sm_border_index = (threadIdx.y * sm_grid_width) + sm_pos.x;

		state_message_to_sm(messages, message_share, sm_border_index, border_index);
	}

	//bottom
	if (bottom_border){
		int2 border_index_2d = global_position;
		border_index_2d.y = sWRAP(border_index_2d.y + range, width);
		border_index = (border_index_2d.y * width) + border_index_2d.x;
		sm_border_index = ((sm_pos.y + range) * sm_grid_width) + sm_pos.x;

		state_message_to_sm(messages, message_share, sm_border_index, border_index);
	}

	//top left
	if ((top_border)&&(left_border)){	
		int2 border_index_2d = global_position;
		border_index_2d.x = sWRAP(border_index_2d.x - range, width);
		border_index_2d.y = sWRAP(border_index_2d.y - range, width);
		border_index = (border_index_2d.y * width) + border_index_2d.x;
		sm_border_index = (threadIdx.y * sm_grid_width) + threadIdx.x;
		
		state_message_to_sm(messages, message_share, sm_border_index, border_index);
	}

	//top right
	if ((top_border)&&(right_border)){	
		int2 border_index_2d = global_position;
		border_index_2d.x = sWRAP(border_index_2d.x + range, width);
		border_index_2d.y = sWRAP(border_index_2d.y - range, width);
		border_index = (border_index_2d.y * width) + border_index_2d.x;
		sm_border_index = (threadIdx.y * sm_grid_width) + (sm_pos.x + range);
		
		state_message_to_sm(messages, message_share, sm_border_index, border_index);
	}

	//bottom right
	if ((bottom_border)&&(right_border)){	
		int2 border_index_2d = global_position;
		border_index_2d.x = sWRAP(border_index_2d.x + range, width);
		border_index_2d.y = sWRAP(border_index_2d.y + range, width);
		border_index = (border_index_2d.y * width) + border_index_2d.x;
		sm_border_index = ((sm_pos.y + range) * sm_grid_width) + (sm_pos.x + range);
		
		state_message_to_sm(messages, message_share, sm_border_index, border_index);
	}

	//bottom left
	if ((bottom_border)&&(left_border)){	
		int2 border_index_2d = global_position;
		border_index_2d.x = sWRAP(border_index_2d.x - range, width);
		border_index_2d.y = sWRAP(border_index_2d.y + range, width);
		border_index = (border_index_2d.y * width) + border_index_2d.x;
		sm_border_index = ((sm_pos.y + range) * sm_grid_width) + threadIdx.x;
		
		state_message_to_sm(messages, message_share, sm_border_index, border_index);
	}

	__syncthreads();
	
  
	//top left of block position sm index
	sm_index = (threadIdx.y * sm_grid_width) + threadIdx.x;
	
	int message_index = SHARE_INDEX(sm_index, sizeof(xmachine_message_state));
	xmachine_message_state* temp = ((xmachine_message_state*)&message_share[message_index]);
	temp->_relative = make_int2(-range, -range); //this is the relative position
	return temp;
}

//Get next state message 
//Used by discrete agents this accesses messages through shared memeory which were all loaded on first message retrieval call.
__device__ xmachine_message_state* get_next_state_message_discrete(xmachine_message_state* message, xmachine_message_state_list* messages){

	//shared memory get from offset dependant on sm usage in function
	extern __shared__ int sm_data [];

	char* message_share = (char*)&sm_data[0];
  
	__syncthreads();
	
	int range = d_message_state_range;
	int sm_grid_width = blockDim.x+(range*2);


	//Get previous position
	int2 previous_relative = message->_relative;

	//exit if at (range, range)
	if (previous_relative.x == range)
        if (previous_relative.y == range)
		    return false;

	//calculate next message relative position
	int2 next_relative = previous_relative;
	next_relative.x += 1;
	if ((next_relative.x)>range){
		next_relative.x = -range;
		next_relative.y = previous_relative.y + 1;
	}

	//skip own message
	if (next_relative.x == 0)
        if (next_relative.y == 0)
		    next_relative.x += 1;


	//calculate the next message position
	int2 next_position;// = block_position+next_relative;
	//offset next position by the sm border size
	next_position.x = threadIdx.x + next_relative.x + range;
	next_position.y = threadIdx.y + next_relative.y + range;

	int sm_index = next_position.x + (next_position.y * sm_grid_width);
	
	__syncthreads();
  
	int message_index = SHARE_INDEX(sm_index, sizeof(xmachine_message_state));
	xmachine_message_state* temp = ((xmachine_message_state*)&message_share[message_index]);
	temp->_relative = next_relative; //this is the relative position
	return temp;
}

//Get first state message
template <int AGENT_TYPE>
__device__ xmachine_message_state* get_first_state_message(xmachine_message_state_list* messages, int agent_x, int agent_y){

	if (AGENT_TYPE == DISCRETE_2D)	//use shared memory method
		return get_first_state_message_discrete(messages);
	else	//use texture fetching method
		return get_first_state_message_continuous(messages, agent_x, agent_y);

}

//Get next state message
template <int AGENT_TYPE>
__device__ xmachine_message_state* get_next_state_message(xmachine_message_state* message, xmachine_message_state_list* messages){

	if (AGENT_TYPE == DISCRETE_2D)	//use shared memory method
		return get_next_state_message_discrete(message, messages);
	else	//use texture fetching method
		return get_next_state_message_continuous(message, messages);

}


	
/////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dyanamically created GPU kernals  */



/**
 *
 */
__global__ void GPUFLAME_output_state(xmachine_memory_cell_list* agents, xmachine_message_state_list* state_messages){
	
	
	//discrete agent: index is position in 2D agent grid
	int width = (blockDim.x * gridDim.x);
	int2 global_position;
	global_position.x = (blockIdx.x * blockDim.x) + threadIdx.x;
	global_position.y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = global_position.x + (global_position.y * width);
	

	//SoA to AoS - xmachine_memory_output_state Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_cell agent;
	agent.state = agents->state[index];
	agent.x = agents->x[index];
	agent.y = agents->y[index];

	//FLAME function call
	output_state(&agent, state_messages	);
	
	

	//AoS to SoA - xmachine_memory_output_state Coalesced memory write (ignore arrays)
	agents->state[index] = agent.state;
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
}

/**
 *
 */
__global__ void GPUFLAME_update_state(xmachine_memory_cell_list* agents, xmachine_message_state_list* state_messages){
	
	
	//discrete agent: index is position in 2D agent grid
	int width = (blockDim.x * gridDim.x);
	int2 global_position;
	global_position.x = (blockIdx.x * blockDim.x) + threadIdx.x;
	global_position.y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = global_position.x + (global_position.y * width);
	

	//SoA to AoS - xmachine_memory_update_state Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_cell agent;
	agent.state = agents->state[index];
	agent.x = agents->x[index];
	agent.y = agents->y[index];

	//FLAME function call
	update_state(&agent, state_messages);
	
	

	//AoS to SoA - xmachine_memory_update_state Coalesced memory write (ignore arrays)
	agents->state[index] = agent.state;
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
}

	
	
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Rand48 functions */

__device__ static uint2 RNG_rand48_iterate_single(uint2 Xn, uint2 A, uint2 C)
{
	unsigned int R0, R1;

	// low 24-bit multiplication
	const unsigned int lo00 = __umul24(Xn.x, A.x);
	const unsigned int hi00 = __umulhi(Xn.x, A.x);

	// 24bit distribution of 32bit multiplication results
	R0 = (lo00 & 0xFFFFFF);
	R1 = (lo00 >> 24) | (hi00 << 8);

	R0 += C.x; R1 += C.y;

	// transfer overflows
	R1 += (R0 >> 24);
	R0 &= 0xFFFFFF;

	// cross-terms, low/hi 24-bit multiplication
	R1 += __umul24(Xn.y, A.x);
	R1 += __umul24(Xn.x, A.y);

	R1 &= 0xFFFFFF;

	return make_uint2(R0, R1);
}

//Templated function
template <int AGENT_TYPE>
__device__ float rnd(RNG_rand48* rand48){

	int index;
	
	//calculate the agents index in global agent list
	if (AGENT_TYPE == DISCRETE_2D){
		int width = (blockDim.x * gridDim.x);
		int2 global_position;
		global_position.x = (blockIdx.x * blockDim.x) + threadIdx.x;
		global_position.y = (blockIdx.y * blockDim.y) + threadIdx.y;
		index = global_position.x + (global_position.y * width);
	}else//AGENT_TYPE == CONTINOUS
		index = threadIdx.x + blockIdx.x*blockDim.x;

	uint2 state = rand48->seeds[index];
	uint2 A = rand48->A;
	uint2 C = rand48->C;

	int rand = ( state.x >> 17 ) | ( state.y << 7);

	// this actually iterates the RNG
	state = RNG_rand48_iterate_single(state, A, C);

	rand48->seeds[index] = state;

	return (float)rand/2147483647;
}

__device__ float rnd(RNG_rand48* rand48){
	return rnd<DISCRETE_2D>(rand48);
}

#endif //_FLAMEGPU_KERNELS_H_
