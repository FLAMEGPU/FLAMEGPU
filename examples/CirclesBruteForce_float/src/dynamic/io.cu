#include "hip/hip_runtime.h"

/*
* FLAME GPU v 1.4.0 for CUDA 6
* Copyright 2015 University of Sheffield.
* Author: Dr Paul Richmond
* Contact: p.richmond@sheffield.ac.uk (http://www.paulrichmond.staff.shef.ac.uk)
*
* University of Sheffield retain all intellectual property and
* proprietary rights in and to this software and related documentation.
* Any use, reproduction, disclosure, or distribution of this software
* and related documentation without an express license agreement from
* University of Sheffield is strictly prohibited.
*
* For terms of licence agreement please attached licence or view licence
* on www.flamegpu.com website.
*
*/

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <limits.h>
	

// include header
#include "header.h"

float3 agent_maximum;
float3 agent_minimum;

void readIntArrayInput(char* buffer, int *array, unsigned int expected_items){
    unsigned int i = 0;
    const char s[2] = ",";
    char * token;

    token = strtok(buffer, s);
    while (token != NULL){
        if (i>=expected_items){
            printf("Error: Agent Memeory array has too many items, expected %d!\n", expected_items);
            exit(0);
        }
        
        array[i++] = atoi(token);
        
        token = strtok(NULL, s);
    }
    if (i != expected_items){
        printf("Error: Agent Memeory array has %d items, expected %d!\n", i, expected_items);
        exit(0);
    }
}

void readFloatArrayInput(char* buffer, float *array, unsigned int expected_items){
    unsigned int i = 0;
    const char s[2] = ",";
    char * token;

    token = strtok(buffer, s);
    while (token != NULL){
        if (i>=expected_items){
            printf("Error: Agent Memeory array has too many items, expected %d!\n", expected_items);
            exit(0);
        }
        
        array[i++] = (float)atof(token);
        
        token = strtok(NULL, s);
    }
    if (i != expected_items){
        printf("Error: Agent Memeory array has %d items, expected %d!\n", i, expected_items);
        exit(0);
    }
}

void saveIterationData(char* outputpath, int iteration_number, xmachine_memory_Circle_list* h_Circles_default, xmachine_memory_Circle_list* d_Circles_default, int h_xmachine_memory_Circle_default_count)
{
	hipError_t cudaStatus;
	
	//Device to host memory transfer
	
	cudaStatus = hipMemcpy( h_Circles_default, d_Circles_default, sizeof(xmachine_memory_Circle_list), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr,"Error Copying Circle Agent default State Memory from GPU: %s\n", hipGetErrorString(cudaStatus));
		exit(cudaStatus);
	}
	
	/* Pointer to file */
	FILE *file;
	char data[100];

	sprintf(data, "%s%i.xml", outputpath, iteration_number);
	//printf("Writing itteration %i data to %s\n", iteration_number, data);
	file = fopen(data, "w");
	fputs("<states>\n<itno>", file);
	sprintf(data, "%i", iteration_number);
	fputs(data, file);
	fputs("</itno>\n", file);
	fputs("<environment>\n" , file);
	fputs("</environment>\n" , file);

	//Write each Circle agent to xml
	for (int i=0; i<h_xmachine_memory_Circle_default_count; i++){
		fputs("<xagent>\n" , file);
		fputs("<name>Circle</name>\n", file);
        
		fputs("<id>", file);
        sprintf(data, "%i", h_Circles_default->id[i]);
		fputs(data, file);
		fputs("</id>\n", file);
        
		fputs("<x>", file);
        sprintf(data, "%f", h_Circles_default->x[i]);
		fputs(data, file);
		fputs("</x>\n", file);
        
		fputs("<y>", file);
        sprintf(data, "%f", h_Circles_default->y[i]);
		fputs(data, file);
		fputs("</y>\n", file);
        
		fputs("<z>", file);
        sprintf(data, "%f", h_Circles_default->z[i]);
		fputs(data, file);
		fputs("</z>\n", file);
        
		fputs("<fx>", file);
        sprintf(data, "%f", h_Circles_default->fx[i]);
		fputs(data, file);
		fputs("</fx>\n", file);
        
		fputs("<fy>", file);
        sprintf(data, "%f", h_Circles_default->fy[i]);
		fputs(data, file);
		fputs("</fy>\n", file);
        
		fputs("</xagent>\n", file);
	}
	
	

	fputs("</states>\n" , file);
	
	/* Close the file */
	fclose(file);
}

void readInitialStates(char* inputpath, xmachine_memory_Circle_list* h_Circles, int* h_xmachine_memory_Circle_count)
{

	int temp = 0;
	int* itno = &temp;

	/* Pointer to file */
	FILE *file;
	/* Char and char buffer for reading file to */
	char c = ' ';
	char buffer[10000];
	char agentname[1000];

	/* Pointer to x-memory for initial state data */
	/*xmachine * current_xmachine;*/
	/* Variables for checking tags */
	int reading, i;
	int in_tag, in_itno, in_name;
    int in_Circle_id;
    int in_Circle_x;
    int in_Circle_y;
    int in_Circle_z;
    int in_Circle_fx;
    int in_Circle_fy;

	/* for continuous agents: set agent count to zero */	
	*h_xmachine_memory_Circle_count = 0;
	
	/* Variables for initial state data */
	int Circle_id;
	float Circle_x;
	float Circle_y;
	float Circle_z;
	float Circle_fx;
	float Circle_fy;
	
	/* Open config file to read-only */
	if((file = fopen(inputpath, "r"))==NULL)
	{
		printf("error opening initial states\n");
		exit(0);
	}
	
	/* Initialise variables */
    agent_maximum.x = 0;
    agent_maximum.y = 0;
    agent_maximum.z = 0;
    agent_minimum.x = 0;
    agent_minimum.y = 0;
    agent_minimum.z = 0;
	reading = 1;
	in_tag = 0;
	in_itno = 0;
	in_name = 0;
	in_Circle_id = 0;
	in_Circle_x = 0;
	in_Circle_y = 0;
	in_Circle_z = 0;
	in_Circle_fx = 0;
	in_Circle_fy = 0;
	//set all Circle values to 0
	//If this is not done then it will cause errors in emu mode where undefined memory is not 0
	for (int k=0; k<xmachine_memory_Circle_MAX; k++)
	{	
		h_Circles->id[k] = 0;
		h_Circles->x[k] = 0;
		h_Circles->y[k] = 0;
		h_Circles->z[k] = 0;
		h_Circles->fx[k] = 0;
		h_Circles->fy[k] = 0;
	}
	

	/* Default variables for memory */
    Circle_id = 0;
    Circle_x = 0;
    Circle_y = 0;
    Circle_z = 0;
    Circle_fx = 0;
    Circle_fy = 0;

	/* Read file until end of xml */
    i = 0;
	while(reading==1)
	{
		/* Get the next char from the file */
		c = (char)fgetc(file);
		
		/* If the end of a tag */
		if(c == '>')
		{
			/* Place 0 at end of buffer to make chars a string */
			buffer[i] = 0;
			
			if(strcmp(buffer, "states") == 0) reading = 1;
			if(strcmp(buffer, "/states") == 0) reading = 0;
			if(strcmp(buffer, "itno") == 0) in_itno = 1;
			if(strcmp(buffer, "/itno") == 0) in_itno = 0;
			if(strcmp(buffer, "name") == 0) in_name = 1;
			if(strcmp(buffer, "/name") == 0) in_name = 0;
			if(strcmp(buffer, "/xagent") == 0)
			{
				if(strcmp(agentname, "Circle") == 0)
				{		
					if (*h_xmachine_memory_Circle_count > xmachine_memory_Circle_MAX){
						printf("ERROR: MAX Buffer size (%i) for agent Circle exceeded whilst reading data\n", xmachine_memory_Circle_MAX);
						// Close the file and stop reading
						fclose(file);
						exit(0);
					}
                    
					h_Circles->id[*h_xmachine_memory_Circle_count] = Circle_id;
					h_Circles->x[*h_xmachine_memory_Circle_count] = Circle_x;//Check maximum x value
                    if(agent_maximum.x < Circle_x)
                        agent_maximum.x = (float)Circle_x;
                    //Check minimum x value
                    if(agent_minimum.x > Circle_x)
                        agent_minimum.x = (float)Circle_x;
                    
					h_Circles->y[*h_xmachine_memory_Circle_count] = Circle_y;//Check maximum y value
                    if(agent_maximum.y < Circle_y)
                        agent_maximum.y = (float)Circle_y;
                    //Check minimum y value
                    if(agent_minimum.y > Circle_y)
                        agent_minimum.y = (float)Circle_y;
                    
					h_Circles->z[*h_xmachine_memory_Circle_count] = Circle_z;//Check maximum z value
                    if(agent_maximum.z < Circle_z)
                        agent_maximum.z = (float)Circle_z;
                    //Check minimum z value
                    if(agent_minimum.z > Circle_z)
                        agent_minimum.z = (float)Circle_z;
                    
					h_Circles->fx[*h_xmachine_memory_Circle_count] = Circle_fx;
					h_Circles->fy[*h_xmachine_memory_Circle_count] = Circle_fy;
					(*h_xmachine_memory_Circle_count) ++;	
				}
				else
				{
					printf("Warning: agent name undefined - '%s'\n", agentname);
				}
				

				
				/* Reset xagent variables */
                Circle_id = 0;
                Circle_x = 0;
                Circle_y = 0;
                Circle_z = 0;
                Circle_fx = 0;
                Circle_fy = 0;

			}
			if(strcmp(buffer, "id") == 0) in_Circle_id = 1;
			if(strcmp(buffer, "/id") == 0) in_Circle_id = 0;
			if(strcmp(buffer, "x") == 0) in_Circle_x = 1;
			if(strcmp(buffer, "/x") == 0) in_Circle_x = 0;
			if(strcmp(buffer, "y") == 0) in_Circle_y = 1;
			if(strcmp(buffer, "/y") == 0) in_Circle_y = 0;
			if(strcmp(buffer, "z") == 0) in_Circle_z = 1;
			if(strcmp(buffer, "/z") == 0) in_Circle_z = 0;
			if(strcmp(buffer, "fx") == 0) in_Circle_fx = 1;
			if(strcmp(buffer, "/fx") == 0) in_Circle_fx = 0;
			if(strcmp(buffer, "fy") == 0) in_Circle_fy = 1;
			if(strcmp(buffer, "/fy") == 0) in_Circle_fy = 0;
			
			
			/* End of tag and reset buffer */
			in_tag = 0;
			i = 0;
		}
		/* If start of tag */
		else if(c == '<')
		{
			/* Place /0 at end of buffer to end numbers */
			buffer[i] = 0;
			/* Flag in tag */
			in_tag = 1;
			
			if(in_itno) *itno = atoi(buffer);
			if(in_name) strcpy(agentname, buffer);
			else
			{
				if(in_Circle_id){ 
                    Circle_id = (int) atoi(buffer);    
                }
				if(in_Circle_x){ 
                    Circle_x = (float) atof(buffer);    
                }
				if(in_Circle_y){ 
                    Circle_y = (float) atof(buffer);    
                }
				if(in_Circle_z){ 
                    Circle_z = (float) atof(buffer);    
                }
				if(in_Circle_fx){ 
                    Circle_fx = (float) atof(buffer);    
                }
				if(in_Circle_fy){ 
                    Circle_fy = (float) atof(buffer);    
                }
				
			}
			
			/* Reset buffer */
			i = 0;
		}
		/* If in tag put read char into buffer */
		else if(in_tag)
		{
			buffer[i] = c;
			i++;
		}
		/* If in data read char into buffer */
		else
		{
			buffer[i] = c;
			i++;
		}
	}
	/* Close the file */
	fclose(file);
}

float3 getMaximumBounds(){
    return agent_maximum;
}

float3 getMinimumBounds(){
    return agent_minimum;
}

