#include "hip/hip_runtime.h"


/*
* FLAME GPU v 1.4.0 for CUDA 6
* Copyright 2015 University of Sheffield.
* Author: Dr Paul Richmond
* Contact: p.richmond@sheffield.ac.uk (http://www.paulrichmond.staff.shef.ac.uk)
*
* University of Sheffield retain all intellectual property and
* proprietary rights in and to this software and related documentation.
* Any use, reproduction, disclosure, or distribution of this software
* and related documentation without an express license agreement from
* University of Sheffield is strictly prohibited.
*
* For terms of licence agreement please attached licence or view licence
* on www.flamegpu.com website.
*
*/

#ifndef _FLAMEGPU_KERNELS_H_
#define _FLAMEGPU_KERNELS_H_

#include "header.h"


/* Agent count constants */

__constant__ int d_xmachine_memory_Boid_count;

/* Agent state count constants */

__constant__ int d_xmachine_memory_Boid_default_count;


/* Message constants */

/* location Message variables */
/* Non partitioned and spatial partitioned message variables  */
__constant__ int d_message_location_count;         /**< message list counter*/
__constant__ int d_message_location_output_type;   /**< message output type (single or optional)*/

	
    
//include each function file

#include "functions.c"
    
/* Texture bindings */

    
#define WRAP(x,m) (((x)<m)?(x):(x%m)) /**< Simple wrap */
#define sWRAP(x,m) (((x)<m)?(((x)<0)?(m+(x)):(x)):(m-(x))) /**<signed integer wrap (no modulus) for negatives where 2m > |x| > m */

//PADDING WILL ONLY AVOID SM CONFLICTS FOR 32BIT
//SM_OFFSET REQUIRED AS FERMI STARTS INDEXING MEMORY FROM LOCATION 0 (i.e. NULL)??
__constant__ int d_SM_START;
__constant__ int d_PADDING;

//SM addressing macro to avoid conflicts (32 bit only)
#define SHARE_INDEX(i, s) (((s + d_PADDING)* i)+d_SM_START) /**<offset struct size by padding to avoid bank conflicts */

//if doubel support is needed then define the following function which requires sm_13 or later
#ifdef _DOUBLE_SUPPORT_REQUIRED_
__inline__ __device__ double tex1DfetchDouble(texture<int2, 1, hipReadModeElementType> tex, int i)
{
	int2 v = tex1Dfetch(tex, i);
  //IF YOU HAVE AN ERROR HERE THEN YOU ARE USING DOUBLE VALUES IN AGENT MEMORY AND NOT COMPILING FOR DOUBLE SUPPORTED HARDWARE
  //To compile for double supported hardware change the CUDA Build rule property "Use sm_13 Architecture (double support)" on the CUDA-Specific Propert Page of the CUDA Build Rule for simulation.cu
	return __hiloint2double(v.y, v.x);
}
#endif

/* Helper functions */
/** next_cell
 * Function used for finding the next cell when using spatial partitioning
 * Upddates the relative cell variable which can have value of -1, 0 or +1
 * @param relative_cell pointer to the relative cell position
 * @return boolean if there is a next cell. True unless relative_Cell value was 1,1,1
 */
__device__ int next_cell3D(int3* relative_cell)
{
	if (relative_cell->x < 1)
	{
		relative_cell->x++;
		return true;
	}
	relative_cell->x = -1;

	if (relative_cell->y < 1)
	{
		relative_cell->y++;
		return true;
	}
	relative_cell->y = -1;
	
	if (relative_cell->z < 1)
	{
		relative_cell->z++;
		return true;
	}
	relative_cell->z = -1;
	
	return false;
}

/** next_cell2D
 * Function used for finding the next cell when using spatial partitioning. Z component is ignored
 * Upddates the relative cell variable which can have value of -1, 0 or +1
 * @param relative_cell pointer to the relative cell position
 * @return boolean if there is a next cell. True unless relative_Cell value was 1,1
 */
__device__ int next_cell2D(int3* relative_cell)
{
	if (relative_cell->x < 1)
	{
		relative_cell->x++;
		return true;
	}
	relative_cell->x = -1;

	if (relative_cell->y < 1)
	{
		relative_cell->y++;
		return true;
	}
	relative_cell->y = -1;
	
	return false;
}


////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dyanamically created Boid agent functions */

/** reset_Boid_scan_input
 * Boid agent reset scan input function
 * @param agents The xmachine_memory_Boid_list agent list
 */
__global__ void reset_Boid_scan_input(xmachine_memory_Boid_list* agents){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	agents->_position[index] = 0;
	agents->_scan_input[index] = 0;
}



/** scatter_Boid_Agents
 * Boid scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_Boid_list agent list destination
 * @param agents_src xmachine_memory_Boid_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void scatter_Boid_Agents(xmachine_memory_Boid_list* agents_dst, xmachine_memory_Boid_list* agents_src, int dst_agent_count, int number_to_scatter){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = agents_src->_scan_input[index];

	//if optional message is to be written. 
	//must check agent is within number to scatter as unused threads may have scan input = 1
	if ((_scan_input == 1)&&(index < number_to_scatter)){
		int output_index = agents_src->_position[index] + dst_agent_count;

		//AoS - xmachine_message_location Un-Coalesced scattered memory write     
        agents_dst->_position[output_index] = output_index;        
		agents_dst->id[output_index] = agents_src->id[index];        
		agents_dst->x[output_index] = agents_src->x[index];        
		agents_dst->y[output_index] = agents_src->y[index];        
		agents_dst->z[output_index] = agents_src->z[index];        
		agents_dst->fx[output_index] = agents_src->fx[index];        
		agents_dst->fy[output_index] = agents_src->fy[index];        
		agents_dst->fz[output_index] = agents_src->fz[index];
	}
}

/** append_Boid_Agents
 * Boid scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_Boid_list agent list destination
 * @param agents_src xmachine_memory_Boid_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void append_Boid_Agents(xmachine_memory_Boid_list* agents_dst, xmachine_memory_Boid_list* agents_src, int dst_agent_count, int number_to_append){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	//must check agent is within number to append as unused threads may have scan input = 1
    if (index < number_to_append){
	    int output_index = index + dst_agent_count;

	    //AoS - xmachine_message_location Un-Coalesced scattered memory write
	    agents_dst->_position[output_index] = output_index;
	    agents_dst->id[output_index] = agents_src->id[index];
	    agents_dst->x[output_index] = agents_src->x[index];
	    agents_dst->y[output_index] = agents_src->y[index];
	    agents_dst->z[output_index] = agents_src->z[index];
	    agents_dst->fx[output_index] = agents_src->fx[index];
	    agents_dst->fy[output_index] = agents_src->fy[index];
	    agents_dst->fz[output_index] = agents_src->fz[index];
    }
}

/** add_Boid_agent
 * Continuous Boid agent add agent function writes agent data to agent swap
 * @param agents xmachine_memory_Boid_list to add agents to 
 * @param id agent variable of type int
 * @param x agent variable of type float
 * @param y agent variable of type float
 * @param z agent variable of type float
 * @param fx agent variable of type float
 * @param fy agent variable of type float
 * @param fz agent variable of type float
 */
template <int AGENT_TYPE>
__device__ void add_Boid_agent(xmachine_memory_Boid_list* agents, int id, float x, float y, float z, float fx, float fy, float fz){
	
	int index;
    
    //calculate the agents index in global agent list (depends on agent type)
	if (AGENT_TYPE == DISCRETE_2D){
		int width = (blockDim.x* gridDim.x);
		int2 global_position;
		global_position.x = (blockIdx.x*blockDim.x) + threadIdx.x;
		global_position.y = (blockIdx.y*blockDim.y) + threadIdx.y;
		index = global_position.x + (global_position.y* width);
	}else//AGENT_TYPE == CONTINOUS
		index = threadIdx.x + blockIdx.x*blockDim.x;

	//for prefix sum
	agents->_position[index] = 0;
	agents->_scan_input[index] = 1;

	//write data to new buffer
	agents->id[index] = id;
	agents->x[index] = x;
	agents->y[index] = y;
	agents->z[index] = z;
	agents->fx[index] = fx;
	agents->fy[index] = fy;
	agents->fz[index] = fz;

}

//non templated version assumes DISCRETE_2D but works also for CONTINUOUS
__device__ void add_Boid_agent(xmachine_memory_Boid_list* agents, int id, float x, float y, float z, float fx, float fy, float fz){
    add_Boid_agent<DISCRETE_2D>(agents, id, x, y, z, fx, fy, fz);
}

/** reorder_Boid_agents
 * Continuous Boid agent areorder function used after key value pairs have been sorted
 * @param values sorted index values
 * @param unordered_agents list of unordered agents
 * @ param ordered_agents list used to output ordered agents
 */
__global__ void reorder_Boid_agents(unsigned int* values, xmachine_memory_Boid_list* unordered_agents, xmachine_memory_Boid_list* ordered_agents)
{
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	uint old_pos = values[index];

	//reorder agent data
	ordered_agents->id[index] = unordered_agents->id[old_pos];
	ordered_agents->x[index] = unordered_agents->x[old_pos];
	ordered_agents->y[index] = unordered_agents->y[old_pos];
	ordered_agents->z[index] = unordered_agents->z[old_pos];
	ordered_agents->fx[index] = unordered_agents->fx[old_pos];
	ordered_agents->fy[index] = unordered_agents->fy[old_pos];
	ordered_agents->fz[index] = unordered_agents->fz[old_pos];
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dyanamically created location message functions */


/** add_location_message
 * Add non partitioned or spatially partitioned location message
 * @param messages xmachine_message_location_list message list to add too
 * @param id agent variable of type int
 * @param x agent variable of type float
 * @param y agent variable of type float
 * @param z agent variable of type float
 * @param fx agent variable of type float
 * @param fy agent variable of type float
 * @param fz agent variable of type float
 */
__device__ void add_location_message(xmachine_message_location_list* messages, int id, float x, float y, float z, float fx, float fy, float fz){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x + d_message_location_count;

	int _position;
	int _scan_input;

	//decide output position
	if(d_message_location_output_type == single_message){
		_position = index; //same as agent position
		_scan_input = 0;
	}else if (d_message_location_output_type == optional_message){
		_position = 0;	   //to be calculated using Prefix sum
		_scan_input = 1;
	}

	//AoS - xmachine_message_location Coalesced memory write
	messages->_scan_input[index] = _scan_input;	
	messages->_position[index] = _position;
	messages->id[index] = id;
	messages->x[index] = x;
	messages->y[index] = y;
	messages->z[index] = z;
	messages->fx[index] = fx;
	messages->fy[index] = fy;
	messages->fz[index] = fz;

}

/**
 * Scatter non partitioned or spatially partitioned location message (for optional messages)
 * @param messages scatter_optional_location_messages Sparse xmachine_message_location_list message list
 * @param message_swap temp xmachine_message_location_list message list to scatter sparse messages to
 */
__global__ void scatter_optional_location_messages(xmachine_message_location_list* messages, xmachine_message_location_list* messages_swap){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = messages_swap->_scan_input[index];

	//if optional message is to be written
	if (_scan_input == 1){
		int output_index = messages_swap->_position[index] + d_message_location_count;

		//AoS - xmachine_message_location Un-Coalesced scattered memory write
		messages->_position[output_index] = output_index;
		messages->id[output_index] = messages_swap->id[index];
		messages->x[output_index] = messages_swap->x[index];
		messages->y[output_index] = messages_swap->y[index];
		messages->z[output_index] = messages_swap->z[index];
		messages->fx[output_index] = messages_swap->fx[index];
		messages->fy[output_index] = messages_swap->fy[index];
		messages->fz[output_index] = messages_swap->fz[index];				
	}
}

/** reset_location_swaps
 * Reset non partitioned or spatially partitioned location message swaps (for scattering optional messages)
 * @param message_swap message list to reset _position and _scan_input values back to 0
 */
__global__ void reset_location_swaps(xmachine_message_location_list* messages_swap){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	messages_swap->_position[index] = 0;
	messages_swap->_scan_input[index] = 0;
}

/* Message functions */

__device__ xmachine_message_location* get_first_location_message(xmachine_message_location_list* messages){

	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];
	
	//wrap size is the number of tiles required to load all messages
	int wrap_size = (ceil((float)d_message_location_count/ blockDim.x)* blockDim.x);

	//if no messages then return false
	if (wrap_size == 0)
		return false;

	//global thread index
	int global_index = (blockIdx.x*blockDim.x) + threadIdx.x;

	//global thread index
	int index = WRAP(global_index, wrap_size);

	//SoA to AoS - xmachine_message_location Coalesced memory read
	xmachine_message_location temp_message;
	temp_message._position = messages->_position[index];
	temp_message.id = messages->id[index];
	temp_message.x = messages->x[index];
	temp_message.y = messages->y[index];
	temp_message.z = messages->z[index];
	temp_message.fx = messages->fx[index];
	temp_message.fy = messages->fy[index];
	temp_message.fz = messages->fz[index];

	//AoS to shared memory
	int message_index = SHARE_INDEX(threadIdx.x, sizeof(xmachine_message_location));
	xmachine_message_location* sm_message = ((xmachine_message_location*)&message_share[message_index]);
	sm_message[0] = temp_message;

	__syncthreads();

  //HACK FOR 64 bit addressing issue in sm
	return ((xmachine_message_location*)&message_share[d_SM_START]);
}

__device__ xmachine_message_location* get_next_location_message(xmachine_message_location* message, xmachine_message_location_list* messages){

	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];
	
	//wrap size is the number of tiles required to load all messages
	int wrap_size = ceil((float)d_message_location_count/ blockDim.x)*blockDim.x;

	int i = WRAP((message->_position + 1),wrap_size);

	//If end of messages (last message not multiple of gridsize) go to 0 index
	if (i >= d_message_location_count)
		i = 0;

	//Check if back to start position of first message
	if (i == WRAP((blockDim.x* blockIdx.x), wrap_size))
		return false;

	int tile = floor((float)i/(blockDim.x)); //tile is round down position over blockDim
	i = i % blockDim.x;						 //mod i for shared memory index

	//if count == Block Size load next tile int shared memory values
	if (i == 0){
		__syncthreads();					//make sure we dont change shared memeory until all threads are here (important for emu-debug mode)
		
		//SoA to AoS - xmachine_message_location Coalesced memory read
		int index = (tile* blockDim.x) + threadIdx.x;
		xmachine_message_location temp_message;
		temp_message._position = messages->_position[index];
		temp_message.id = messages->id[index];
		temp_message.x = messages->x[index];
		temp_message.y = messages->y[index];
		temp_message.z = messages->z[index];
		temp_message.fx = messages->fx[index];
		temp_message.fy = messages->fy[index];
		temp_message.fz = messages->fz[index];

		//AoS to shared memory
		int message_index = SHARE_INDEX(threadIdx.x, sizeof(xmachine_message_location));
		xmachine_message_location* sm_message = ((xmachine_message_location*)&message_share[message_index]);
		sm_message[0] = temp_message;

		__syncthreads();					//make sure we dont start returning messages untill all threads have updated shared memory
	}

	int message_index = SHARE_INDEX(i, sizeof(xmachine_message_location));
	return ((xmachine_message_location*)&message_share[message_index]);
}


	
/////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dyanamically created GPU kernals  */



/**
 *
 */
__global__ void GPUFLAME_outputdata(xmachine_memory_Boid_list* agents, xmachine_message_location_list* location_messages){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    //For agents not using non partitioned message input check the agent bounds
    if (index >= d_xmachine_memory_Boid_count)
        return;
    

	//SoA to AoS - xmachine_memory_outputdata Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_Boid agent;
	agent.id = agents->id[index];
	agent.x = agents->x[index];
	agent.y = agents->y[index];
	agent.z = agents->z[index];
	agent.fx = agents->fx[index];
	agent.fy = agents->fy[index];
	agent.fz = agents->fz[index];

	//FLAME function call
	int dead = !outputdata(&agent, location_messages	);
	
	//continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_outputdata Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
	agents->z[index] = agent.z;
	agents->fx[index] = agent.fx;
	agents->fy[index] = agent.fy;
	agents->fz[index] = agent.fz;
}

/**
 *
 */
__global__ void GPUFLAME_inputdata(xmachine_memory_Boid_list* agents, xmachine_message_location_list* location_messages){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    
    //No partitioned input requires threads to be launched beyond the agent count to ensure full block sizes
    

	//SoA to AoS - xmachine_memory_inputdata Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_Boid agent;
	agent.id = agents->id[index];
	agent.x = agents->x[index];
	agent.y = agents->y[index];
	agent.z = agents->z[index];
	agent.fx = agents->fx[index];
	agent.fy = agents->fy[index];
	agent.fz = agents->fz[index];

	//FLAME function call
	int dead = !inputdata(&agent, location_messages);
	
	//continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_inputdata Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
	agents->z[index] = agent.z;
	agents->fx[index] = agent.fx;
	agents->fy[index] = agent.fy;
	agents->fz[index] = agent.fz;
}

	
	
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Rand48 functions */

__device__ static uint2 RNG_rand48_iterate_single(uint2 Xn, uint2 A, uint2 C)
{
	unsigned int R0, R1;

	// low 24-bit multiplication
	const unsigned int lo00 = __umul24(Xn.x, A.x);
	const unsigned int hi00 = __umulhi(Xn.x, A.x);

	// 24bit distribution of 32bit multiplication results
	R0 = (lo00 & 0xFFFFFF);
	R1 = (lo00 >> 24) | (hi00 << 8);

	R0 += C.x; R1 += C.y;

	// transfer overflows
	R1 += (R0 >> 24);
	R0 &= 0xFFFFFF;

	// cross-terms, low/hi 24-bit multiplication
	R1 += __umul24(Xn.y, A.x);
	R1 += __umul24(Xn.x, A.y);

	R1 &= 0xFFFFFF;

	return make_uint2(R0, R1);
}

//Templated function
template <int AGENT_TYPE>
__device__ float rnd(RNG_rand48* rand48){

	int index;
	
	//calculate the agents index in global agent list
	if (AGENT_TYPE == DISCRETE_2D){
		int width = (blockDim.x * gridDim.x);
		int2 global_position;
		global_position.x = (blockIdx.x * blockDim.x) + threadIdx.x;
		global_position.y = (blockIdx.y * blockDim.y) + threadIdx.y;
		index = global_position.x + (global_position.y * width);
	}else//AGENT_TYPE == CONTINOUS
		index = threadIdx.x + blockIdx.x*blockDim.x;

	uint2 state = rand48->seeds[index];
	uint2 A = rand48->A;
	uint2 C = rand48->C;

	int rand = ( state.x >> 17 ) | ( state.y << 7);

	// this actually iterates the RNG
	state = RNG_rand48_iterate_single(state, A, C);

	rand48->seeds[index] = state;

	return (float)rand/2147483647;
}

__device__ float rnd(RNG_rand48* rand48){
	return rnd<DISCRETE_2D>(rand48);
}

#endif //_FLAMEGPU_KERNELS_H_
